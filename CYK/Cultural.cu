#include "hip/hip_runtime.h"
#include "Base.h"
#include "Cultural.cuh"
#ifndef CULTURAL_CU
#define CULTURAL_CU

__forceinline__ __device__  void DeviceCultural::beforeOutput(int threadIndex) {
	__syncthreads();
	if(threadIndex == 0) {
		*_outputAllowed = 0;
		if(_numOfConnections > 1) {		
			int count = 0;
			for(int i = 0; i < _numOfConnections; i++) {
				if(_myVersionArray[0] == _myVersionArray[i] ) {
					count += 1;
				} else {
					break;
				}
			}	
			if(count == _numOfConnections) {
				*_outputAllowed = 1;
			}
		}
			__threadfence_system();
	}
	__syncthreads();
}

__forceinline__ __device__  void DeviceCultural::afterOutput(int threadIndex) {
	__syncthreads();
	if(threadIndex == 0 && *_outputAllowed != 0) {
		_myVersionArray[0] = _myVersionArray[0] + 1;
		*_outputAllowed = 0;
		__threadfence_system();
	}
	__syncthreads();
}

__forceinline__ __device__ void DeviceCultural::beforeInput(int threadIndex) {
	__syncthreads();
	if(threadIndex == 0) {
		*_inputAllowed = 0;
		if(_sbVersionArray[_ID] < _sbVersionArray[0]) {
			*_inputAllowed = 1;
			__threadfence_system();
		}
	}
	__syncthreads();
}

__forceinline__ __device__ void DeviceCultural::afterInput(int threadIndex) {
	__syncthreads();
	if(threadIndex == 0 && *_inputAllowed != 0) {
			_sbVersionArray[_ID] = _sbVersionArray[0];
			*_inputAllowed = 0;
			__threadfence_system();
	}
	__syncthreads();
	
}

__forceinline__ __device__ void DeviceCultural::setStatus(int num) {
	*_myStatus = num;
}

__forceinline__ __device__ void DeviceCultural::setResult(int num) {
	*_result = num;
}

#endif