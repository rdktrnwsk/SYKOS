#include "hip/hip_runtime.h"
#include "CYK.cuh"

template<int action>
__global__ void cykAlgorithm(DeviceCYKData data, hiprandState * randGlobal)
{
	/*if (action == 1 || action == 2) {
		printf("%d", threadIdx.x);
	}*/

	__shared__ int** cykArray;
	__shared__ int inputStringLength;
	__shared__ int** rulesNonTermsArray;
	__shared__ int nonTermsCount;

	if (threadIdx.x == 0) {

		cykArray = data.getCYKArray();
		inputStringLength = data.getInputCount();
		rulesNonTermsArray = data.getRulesNonTermsArray();
		nonTermsCount = data.getNonTermsCount();

		for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)

			for (int j = 0; j < inputStringLength - i; j++) { // every word of given length 5, 4, 3, 2, 1... (2)

				for (int k = 0; k < i; k++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

											  //TODO correct split points!
					int first = cykArray[k][j];
					int second = cykArray[i - k - 1][j + k + 1];

					//decode nonterminals (find out if bits are on a given positions)
					int base = 1;
					for (int l = 0; l < nonTermsCount; l++) {
						int bitMaskFirst = base << l;
						//all possibilities connected with rules
						for (int m = 0; m < nonTermsCount; m++) {
							int bitMaskSecond = base << m;

							// if rule with 'l' index and 'm' index is created and ready to be found if corrrect X ->lm (does X exist in a grammar?)
							if (first & bitMaskFirst && second & bitMaskSecond) {
								//cout << bitMaskFirst << ", " << bitMaskSecond << " | ";

								//rule exists
								if (rulesNonTermsArray[l][m] != -1) {
									int shiftValue = rulesNonTermsArray[l][m];
									int bitValue = base << shiftValue;

									cykArray[i][j] |= bitValue;
								}

							}


						}

					}

					//cout << first << " | " << second << endl;

					//combinations of productions

					// for each production (rulesNonTerminals)

				}

			}

			//break; //only first line

		}
	}
	
	__syncthreads();
	
	if (threadIdx.x == 0) {

		for (int i = 0; i < nonTermsCount; i++) {
			for (int j = 0; j < nonTermsCount; j++) {
				//cout << rulesNonTermsArray[i][j] << " | ";

				printf("%d | ", rulesNonTermsArray[i][j]);
			}
			//cout << endl;

			printf("\n");
		}

		for (int j = 1; j < inputStringLength; j++) {
			for (int i = 0; i < inputStringLength - j; i++) {
				printf("%d | ", cykArray[j][i]);
			}
			printf("\n");
		}

		int* result = data.getResult();
		printf("RESUUUULt: %d | ", result[0]);
		result[0] = 1337;
	}

	__syncthreads();

	return;
}