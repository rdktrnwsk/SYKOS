#include "hip/hip_runtime.h"
﻿#include "CYK.cuh"

template<int action>
__global__ void cykAlgorithm(DeviceCYKData data, hiprandState * randGlobal) {

	__shared__ int** cykArray;
	__shared__ int inputStringLength;
	__shared__ int** rulesNonTermsArray;
	__shared__ int nonTermsCount;

	if (threadIdx.x == 0 && threadIdx.y == 0) {
		cykArray = data.getCYKArray();
		inputStringLength = data.getInputCount();
		rulesNonTermsArray = data.getRulesNonTermsArray();
		nonTermsCount = data.getNonTermsCount();
	}

	__syncthreads();

	if (action == 0) {
		if (threadIdx.x == 0) {
			int idx = threadIdx.x;

			for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)

				for (int j = 0; j < inputStringLength - i; j++) { // every word of given length 5, 4, 3, 2, 1... (2)

					for (int k = 0; k < i; k++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

												  //TODO correct split points!
						int first = cykArray[k][j];
						int second = cykArray[i - k - 1][j + k + 1];

						//decode nonterminals (find out if bits are on a given positions)
						int base = 1;
						for (int l = 0; l < nonTermsCount; l++) {

							int bitMaskFirst = base << l;

							// all possibilities connected with rules
							for (int m = 0; m < nonTermsCount; m++) {
								int bitMaskSecond = base << m;

								// if rule with 'l' index and 'm' index is created and ready to be found if corrrect X ->lm (does X exist in a grammar?)
								if (first & bitMaskFirst && second & bitMaskSecond) {

									//rule exists
									if (rulesNonTermsArray[l][m] != -1) {
										int shiftValue = rulesNonTermsArray[l][m];
										int bitValue = base << shiftValue;

										cykArray[i][j] |= bitValue;
									}

								}

							}

						} // l loop end

					}

				}

				//break; //only first line

			}
		}
	} else if (action == 1) {

		int idx = threadIdx.x;

		if (idx < nonTermsCount) {
			
			for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)
				
				for (int j = 0; j < inputStringLength - i; j++) { // every word of given length 5, 4, 3, 2, 1... (2)

					for (int k = 0; k < i; k++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

												  //TODO correct split points!
						int first = cykArray[k][j];
						int second = cykArray[i - k - 1][j + k + 1];

						//decode nonterminals (find out if bits are on a given positions)
						int base = 1;
						for (int l = 0; l < nonTermsCount; l++) {
						
							int bitMaskFirst = base << l;

							// all possibilities connected with rules
							//for (int m = 0; m < nonTermsCount; m++) {
							int m = idx;

								int bitMaskSecond = base << m;

								// if rule with 'l' index and 'm' index is created and ready to be found if corrrect X ->lm (does X exist in a grammar?)
								if (first & bitMaskFirst && second & bitMaskSecond) {

									//rule exists
									if (rulesNonTermsArray[l][m] != -1) {
										int shiftValue = rulesNonTermsArray[l][m];
										int bitValue = base << shiftValue;

										cykArray[i][j] |= bitValue;
									}

								}

							//}

						} // l loop end

					}

				}

				//break; //only first line

			}
		} 
	} else if (action == 2) { //////////////////////////////////////////////////////////////// each entry 1D threads (last loop 1)

		int idx = threadIdx.x;

		if (idx < nonTermsCount) {
			for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)

				for (int j = 0; j < inputStringLength - i; j++) { // every word of given length 5, 4, 3, 2, 1... (2)

					for (int k = 0; k < i; k++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

												  //TODO correct split points!
						int first = cykArray[k][j];
						int second = cykArray[i - k - 1][j + k + 1];

						//decode nonterminals (find out if bits are on a given positions)
						int base = 1;
						//for (int m = 0; m < nonTermsCount; m++) {
						int m = idx;
						int bitMaskFirst = base << m;
						//all possibilities connected with rules

						for (int n = 0; n < nonTermsCount; n++) {
							int bitMaskSecond = base << n;

							// if rule with 'm' index and 'n' index is created and ready to be found if corrrect X ->mn (does X exist in a grammar?)
							if (first & bitMaskFirst && second & bitMaskSecond) {
								//cout << bitMaskFirst << ", " << bitMaskSecond << " | ";

								//rule exists
								if (rulesNonTermsArray[m][n] != -1) {
									int shiftValue = rulesNonTermsArray[m][n];
									int bitValue = base << shiftValue;

									//TODO - tutaj może być problem
									atomicOr(&cykArray[i][j], bitValue);
									//cykArray[i][j] |= bitValue;
								}

							}

						}

					}

				}

			}
		}

		
	} else if (action == 3) { //////////////////////////////////////////////////////////////// each entry 2D threads (last loop 1)

	 if (threadIdx.x <= nonTermsCount && threadIdx.y <= nonTermsCount) {
		 int idx = threadIdx.x;
		 int idy = threadIdx.y;

		 for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)

			 for (int j = 0; j < inputStringLength - i; j++) { // every word of given length 5, 4, 3, 2, 1... (2)

				 for (int k = 0; k < i; k++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

											   //TODO correct split points!
					 int first = cykArray[k][j];
					 int second = cykArray[i - k - 1][j + k + 1];

					 //decode nonterminals (find out if bits are on a given positions)
					 int base = 1;
					 //for (int l = 0; l < nonTermsCount; l++) {
					 int l = idx;
					 int bitMaskFirst = base << l;
					 //all possibilities connected with rules

					 //for (int m = 0; m < nonTermsCount; m++) {

						 int m = idy;

						 int bitMaskSecond = base << m;

						 // if rule with 'l' index and 'm' index is created and ready to be found if corrrect X ->lm (does X exist in a grammar?)
						 if (first & bitMaskFirst && second & bitMaskSecond) {
							 //cout << bitMaskFirst << ", " << bitMaskSecond << " | ";

							 //rule exists
							 if (rulesNonTermsArray[l][m] != -1) {
								 int shiftValue = rulesNonTermsArray[l][m];
								 int bitValue = base << shiftValue;

								 //TODO - tutaj może być problem
								 atomicOr(&cykArray[i][j], bitValue);

								 //cykArray[i][j] |= bitValue;
							 }

						 }


					// }

					 //} // l loop end

					 //cout << first << " | " << second << endl;

					 //combinations of productions

					 // for each production (rulesNonTerminals)

				 }

			 }

			 //break; //only first line

		 }
	 }
 }	else if (action == 4) { //////////////////////////////////////////////////////////////// magisterka rozwiazanie pierwsze

	 if (threadIdx.x < inputStringLength) {
		 int idx = threadIdx.x;

		 for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)

			 //for (int j = 0; j < inputStringLength - i; j++) { // every word of given length 5, 4, 3, 2, 1... (2)

			 if (threadIdx.x < inputStringLength - i) {
				 int j = threadIdx.x;

				 for (int k = 0; k < i; k++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

											   //TODO correct split points!
					 int first = cykArray[k][j];
					 int second = cykArray[i - k - 1][j + k + 1];

					 //decode nonterminals (find out if bits are on a given positions)
					 int base = 1;
					 for (int l = 0; l < nonTermsCount; l++) {

						 int bitMaskFirst = base << l;

						 // all possibilities connected with rules
						 for (int m = 0; m < nonTermsCount; m++) {
							 int bitMaskSecond = base << m;

							 // if rule with 'l' index and 'm' index is created and ready to be found if corrrect X ->lm (does X exist in a grammar?)
							 if (first & bitMaskFirst && second & bitMaskSecond) {

								 //rule exists
								 if (rulesNonTermsArray[l][m] != -1) {
									 int shiftValue = rulesNonTermsArray[l][m];
									 int bitValue = base << shiftValue;

									 cykArray[i][j] |= bitValue;
								 }

							 }

						 }

					 } // l loop end

				 }

			 }

			 __syncthreads();

			 //break; //only first line

			}
		}
	}
 else if (action == 5) { //////////////////////////////////////////////////////////////// magisterka rozwiazanie drugie

	 int idx = threadIdx.x;
	 int idy = threadIdx.y;

		 for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)

			 /*int pidx = i - idy - 1;
			 int pidy = idx + idy + 1;*/

			 //for (int j = 0; j < inputStringLength - i; j++) { // every word of given length 5, 4, 3, 2, 1... (2)

			 float iter = ceilf((float)(inputStringLength - i) / (float)blockDim.x);
			 //iter = 2.0f;

			 for (int r = 0; r < (int)iter; r++) {

				 int temp_idx = idx + (r * blockDim.x);

				 float iter2 = ceilf((float)(i) / (float)blockDim.y);

				 for (int s = 0; s < (int)iter2; s++) {

					 int temp_idy = idy + (s * blockDim.y);

				 if (temp_idx < inputStringLength - i && temp_idy < i) {
					 int j = temp_idx;
					 int k = temp_idy;

					 //for (int k = 0; k < i; k++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

						 //TODO correct split points!
						 int first = cykArray[k][j];
						 int second = cykArray[i - k - 1][j + k + 1];

						 //decode nonterminals (find out if bits are on a given positions)
						 int base = 1;
						 for (int l = 0; l < nonTermsCount; l++) {

							 int bitMaskFirst = base << l;

							 // all possibilities connected with rules
							 for (int m = 0; m < nonTermsCount; m++) {
								 int bitMaskSecond = base << m;

								 // if rule with 'l' index and 'm' index is created and ready to be found if corrrect X ->lm (does X exist in a grammar?)
								 if (first & bitMaskFirst && second & bitMaskSecond) {

									 //rule exists
									 if (rulesNonTermsArray[l][m] != -1) {
										 int shiftValue = rulesNonTermsArray[l][m];
										 int bitValue = base << shiftValue;

										 cykArray[i][j] |= bitValue;
									 }

								 }

							 }

						 } // l loop end
				 

					 //} // k loop

					 }
				 } // end s loop

		 } // r loop

			 __syncthreads();

			 //break; //only first line

		 }
	}

	
	__syncthreads();
	
	if (threadIdx.x == 0 && threadIdx.y == 0) {

		for (int i = 0; i < nonTermsCount; i++) {
			for (int j = 0; j < nonTermsCount; j++) {
				//cout << rulesNonTermsArray[i][j] << " | ";

				printf("%d | ", rulesNonTermsArray[i][j]);
			}
			//cout << endl;

			printf("\n");
		}

		for (int j = 1; j < inputStringLength; j++) {
			for (int i = 0; i < inputStringLength - j; i++) {
				printf("%d | ", cykArray[j][i]);
			}
			printf("\n");
		}

		int* result = data.getResult();
		printf("RESUUUULt: %d | ", result[0]);
		result[0] = 1337;
	}

	__syncthreads();

	return;
}

/*                                                                                      GLOBAL FUNCTION                                                                     */

__device__ volatile int g_mutex;

template<int action>
__global__ void cykAlgorithmCooperative(DeviceCYKData data, hiprandState * randGlobal, volatile int* arrayIn, volatile int* arrayOut) {
	__shared__ int** cykArray;
	__shared__ int inputStringLength;
	__shared__ int** rulesNonTermsArray;
	__shared__ int nonTermsCount;

	int bidx = blockIdx.x;
	int idx = threadIdx.x;
	int idy = threadIdx.y;

	if (threadIdx.x == 0 && threadIdx.y == 0) {
		g_mutex = 0;
		cykArray = data.getCYKArray();
		inputStringLength = data.getInputCount();
		rulesNonTermsArray = data.getRulesNonTermsArray();
		nonTermsCount = data.getNonTermsCount();
		//printf("Dim %d\n", gridDim.x);
	}


	__syncthreads();

	if (action == 0) { //////////////////////////////////////////////////////////////// entry = block, block used are decremented, threads 2D

		if (threadIdx.x < nonTermsCount && threadIdx.y < nonTermsCount ) {
		
			for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)
				
				//for (int j = 0; j < inputStringLength - i; j++) { // every word of given length 5, 4, 3, 2, 1... (2)

				float iter = ceilf((float)(inputStringLength - i) / (float)gridDim.x);
				//iter = 2.0f;
				//if (iter < 1.0f) iter = 1.0f;
				

				for (int r = 0; r < (int)iter ; r++) {

					int temp_bidx = bidx + (r * gridDim.x);

					
					if (temp_bidx < inputStringLength - i) {
						int j = temp_bidx;

						

						for (int k = 0; k < i; k++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

													  //TODO correct split points!
							int first = cykArray[k][j];
							int second = cykArray[i - k - 1][j + k + 1];

							//decode nonterminals (find out if bits are on a given positions)
							int base = 1;
							//for (int l = 0; l < nonTermsCount; l++) {
							int l = idx;
							int bitMaskFirst = base << l;
							//all possibilities connected with rules

							//for (int m = 0; m < nonTermsCount; m++) {

							int m = idy;

							int bitMaskSecond = base << m;

							// if rule with 'l' index and 'm' index is created and ready to be found if corrrect X ->lm (does X exist in a grammar?)
							if (first & bitMaskFirst && second & bitMaskSecond) {
								//cout << bitMaskFirst << ", " << bitMaskSecond << " | ";

								//rule exists
								if (rulesNonTermsArray[l][m] != -1) {
									int shiftValue = rulesNonTermsArray[l][m];
									int bitValue = base << shiftValue;

									//TODO - tutaj może być problem
									atomicOr(&cykArray[i][j], bitValue);

									//cykArray[i][j] |= bitValue;
								}

							}

						}

					}
				}

				//break; //only first line
				if (idx == 0 && idy == 0) {
					//printf("%d | ", g_mutex);
					atomicAdd((int *)&g_mutex, 1);
					//only when all blocks add 1 to g_mutex
					//will g_mutex equal to goalVal
					while (g_mutex != (gridDim.x * i)) {
						//Do nothing here
					}
					
				}
				__syncthreads();

			}
		}

		// only thread 0 is used for synchronization


	} else if (action == 1) { //////////////////////////////////////////////////////////////// entry = block, block used are decremented, threads 2D + FAST BARIER SYNCHRONISATION

		if (threadIdx.x <= nonTermsCount && threadIdx.y <= nonTermsCount) {


			for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)

				__syncthreads();
														  //for (int j = 0; j < inputStringLength - i; j++) { // every word of given length 5, 4, 3, 2, 1... (2)

				float iter = ceilf((float)(inputStringLength - i) / (float)gridDim.x);
				//iter = 2.0f;

				for (int j = 0; j < (int)iter; j++) {

					int temp_bidx = bidx + (j * gridDim.x);

					if (temp_bidx < inputStringLength - i) {
						int j = temp_bidx;

						for (int k = 0; k < i; k++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

													  //TODO correct split points!
							int first = cykArray[k][j];
							int second = cykArray[i - k - 1][j + k + 1];

							//decode nonterminals (find out if bits are on a given positions)
							int base = 1;
							//for (int l = 0; l < nonTermsCount; l++) {
							int l = idx;
							int bitMaskFirst = base << l;
							//all possibilities connected with rules

							//for (int m = 0; m < nonTermsCount; m++) {

							int m = idy;

							int bitMaskSecond = base << m;

							// if rule with 'l' index and 'm' index is created and ready to be found if corrrect X ->lm (does X exist in a grammar?)
							if (first & bitMaskFirst && second & bitMaskSecond) {
								//cout << bitMaskFirst << ", " << bitMaskSecond << " | ";

								//rule exists
								if (rulesNonTermsArray[l][m] != -1) {
									int shiftValue = rulesNonTermsArray[l][m];
									int bitValue = base << shiftValue;

									//TODO - tutaj może być problem
									atomicOr(&cykArray[i][j], bitValue);

									//cykArray[i][j] |= bitValue;
								}

							}


							// }

							//} // l loop end

							//cout << first << " | " << second << endl;

							//combinations of productions

							// for each production (rulesNonTerminals)

						}

					}
				}

				if (idx == 0 && idy == 0) {
					arrayIn[bidx] = i;
				}

				if (bidx == 1) {

					if (idx < gridDim.x && idy == 0) {
						while (arrayIn[idx] != i) {

						}
					}

					__syncthreads();

					if (idx < gridDim.x && idy == 0) {
						arrayOut[idx] = i;
					}

				}

				//break; //only first line
				if (idx == 0 && idy == 0) {
					while (arrayOut[bidx] != i) {

					}
				}
				__syncthreads();

			}
		}

		// only thread 0 is used for synchronization


	} else if (action == 2) { //////////////////////////////////////////////////////////////// entry = block, block used are decremented, threads 2D + FAST BARIER SYNCHRONISATION

		if (threadIdx.x <= nonTermsCount && threadIdx.y <= nonTermsCount) {

			for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)

														  //for (int j = 0; j < inputStringLength - i; j++) { // every word of given length 5, 4, 3, 2, 1... (2)

				float iter = ceilf((float)(inputStringLength - i) / (float)gridDim.x);
				//iter = 2.0f;

				for (int r = 0; r < (int)iter; r++) {

					int temp_bidx = bidx + (r * gridDim.x);

					if (temp_bidx < inputStringLength - i) {
						int j = temp_bidx;

						int idz = threadIdx.z;

						float iter2 = ceilf((float)(i) / (float)blockDim.z);

						for (int s = 0; s < (int)iter2; s++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

							int temp_idz = idz + (s * blockDim.z);

							if (temp_idz < i) {

								int k = temp_idz;

													  //TODO correct split points!
							int first = cykArray[k][j];
							int second = cykArray[i - k - 1][j + k + 1];

							//decode nonterminals (find out if bits are on a given positions)
							int base = 1;
							//for (int l = 0; l < nonTermsCount; l++) {
							int l = idx;
							int bitMaskFirst = base << l;
							//all possibilities connected with rules

							//for (int m = 0; m < nonTermsCount; m++) {

							int m = idy;

							int bitMaskSecond = base << m;

							// if rule with 'l' index and 'm' index is created and ready to be found if corrrect X ->lm (does X exist in a grammar?)
							if (first & bitMaskFirst && second & bitMaskSecond) {
								//cout << bitMaskFirst << ", " << bitMaskSecond << " | ";

								//rule exists
								if (rulesNonTermsArray[l][m] != -1) {
									int shiftValue = rulesNonTermsArray[l][m];
									int bitValue = base << shiftValue;

									//TODO - tutaj może być problem
									atomicOr(&cykArray[i][j], bitValue);

									//cykArray[i][j] |= bitValue;
								}

							}

						} //end s loop
					}

					}
				}

				//break; //only first line
				if (idx == 0 && idy == 0) {
					//printf("%d | ", g_mutex);
					atomicAdd((int *)&g_mutex, 1);
					//only when all blocks add 1 to g_mutex
					//will g_mutex equal to goalVal
					while (g_mutex != (gridDim.x * i)) {
						//Do nothing here
					}

				}
				__syncthreads();

			}
		}

	 } else if (action == 3) { //////////////////////////////////////////////////////////////// entry = block, block used are decremented, threads 2D + FAST BARIER SYNCHRONISATION

		if (threadIdx.x <= nonTermsCount) {

			for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)

														  //for (int j = 0; j < inputStringLength - i; j++) { // every word of given length 5, 4, 3, 2, 1... (2)

				float iter = ceilf((float)(inputStringLength - i) / (float)gridDim.x);
				//iter = 2.0f;

				for (int r = 0; r < (int)iter; r++) {

					int temp_bidx = bidx + (r * gridDim.x);

					if (temp_bidx < inputStringLength - i) {
						int j = temp_bidx;

						int idy = threadIdx.y;

						float iter2 = ceilf((float)(i) / (float)blockDim.y);

						for (int s = 0; s < (int)iter2; s++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

							int temp_idy = idy + (s * blockDim.y);

							if (temp_idy < i) {

								int k = temp_idy;

								//TODO correct split points!
								int first = cykArray[k][j];
								int second = cykArray[i - k - 1][j + k + 1];

								//decode nonterminals (find out if bits are on a given positions)
								int base = 1;
								//for (int l = 0; l < nonTermsCount; l++) {
								int l = idx;
								int bitMaskFirst = base << l;
								//all possibilities connected with rules

								for (int m = 0; m < nonTermsCount; m++) {
									int bitMaskSecond = base << m;

									// if rule with 'l' index and 'm' index is created and ready to be found if corrrect X ->lm (does X exist in a grammar?)
									if (first & bitMaskFirst && second & bitMaskSecond) {
										//cout << bitMaskFirst << ", " << bitMaskSecond << " | ";

										//rule exists
										if (rulesNonTermsArray[l][m] != -1) {
											int shiftValue = rulesNonTermsArray[l][m];
											int bitValue = base << shiftValue;

											//TODO - tutaj może być problem
											atomicOr(&cykArray[i][j], bitValue);

											//cykArray[i][j] |= bitValue;
										}

									}


								}

							} //end s loop
						}

					}
				}

				//break; //only first line
				if (idx == 0 && idy == 0) {
					//printf("%d | ", g_mutex);
					atomicAdd((int *)&g_mutex, 1);
					//only when all blocks add 1 to g_mutex
					//will g_mutex equal to goalVal
					while (g_mutex != (gridDim.x * i)) {
						//Do nothing here
					}

				}
				__syncthreads();

			}
		}

	} 

	

	__syncthreads();

	if (threadIdx.x == 0 && threadIdx.y == 0 && bidx == 0) {
		
		for (int i = 0; i < nonTermsCount; i++) {
			for (int j = 0; j < nonTermsCount; j++) {
				//cout << rulesNonTermsArray[i][j] << " | ";

				printf("%d | ", rulesNonTermsArray[i][j]);
			}
			//cout << endl;

			printf("\n");
		}

		for (int j = 1; j < inputStringLength; j++) {
			for (int i = 0; i < inputStringLength - j; i++) {
				printf("%d | ", cykArray[j][i]);
			}
			printf("\n");
		}

		int* result = data.getResult();
		printf("RESUUUULt: %d | ", result[0]);
		result[0] = 1337;
	}

	__syncthreads();

	return;
}



template<int action>
__global__ void cykAlgorithmRules(DeviceCYKData data, hiprandState * randGlobal, volatile int * arrayIn, volatile int * arrayOut, int** rulesArray, int rulesCount)
{
	__shared__ int** cykArray;
	__shared__ int inputStringLength;
	__shared__ int** rulesNonTermsArray;
	__shared__ int nonTermsCount;

	int bidx = blockIdx.x;
	int bidy = blockIdx.y;
	int idx = threadIdx.x;
	int idy = threadIdx.y;

	if (threadIdx.x == 0 && threadIdx.y == 0) {
		g_mutex = 0;
		cykArray = data.getCYKArray();
		inputStringLength = data.getInputCount();
		rulesNonTermsArray = data.getRulesNonTermsArray();
		nonTermsCount = data.getNonTermsCount();
		//printf("Dim %d\n", gridDim.x);
	}


	__syncthreads();

	if (action == 0) { //////////////////////////////////////////////////////////////// only threads

		if (threadIdx.x <= nonTermsCount && threadIdx.y <= nonTermsCount) {

			for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)

														  //for (int j = 0; j < inputStringLength - i; j++) { // every word of given length 5, 4, 3, 2, 1... (2)

				float iter = ceilf((float)(inputStringLength - i) / (float)gridDim.x);
				//iter = 2.0f;

				for (int r = 0; r < (int)iter; r++) {

					int temp_bidx = bidx + (r * gridDim.x);

					if (temp_bidx < inputStringLength - i) {
						int j = temp_bidx; //J

						//for (int p = 0; p < rulesCount; p++) { //for each production (each rule)
						if (idx < rulesCount) {

							int p = idx;

							for (int k = 0; k < i; k++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

														  //TODO correct split points!
								int first = cykArray[k][j];
								int second = cykArray[i - k - 1][j + k + 1];

								//decode nonterminals (find out if bits are on a given positions)
								int base = 1;
								int bitMaskFirst = base << rulesArray[0][p];
								int bitMaskSecond = base << rulesArray[1][p];
								if (first & bitMaskFirst && second & bitMaskSecond) {

									int shiftValue = rulesArray[2][p];
									int bitValue = base << shiftValue;
									//TODO - tutaj może być problem
									atomicOr(&cykArray[i][j], bitValue);
								}

							}
						}

					}
				}

				//break; //only first line
				if (idx == 0 && idy == 0) {
					//printf("%d | ", g_mutex);
					atomicAdd((int *)&g_mutex, 1);
					//only when all blocks add 1 to g_mutex
					//will g_mutex equal to goalVal
					while (g_mutex != (gridDim.x * i)) {
						//Do nothing here
					}

				}
				__syncthreads();

			}
		}

		// only thread 0 is used for synchronization


	} if (action == 2) { //////////////////////////////////////////////////////////////// only threads

		if (threadIdx.x <= nonTermsCount && threadIdx.y <= nonTermsCount) {

			//for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)

														  //for (int j = 0; j < inputStringLength - i; j++) { // every word of given length 5, 4, 3, 2, 1... (2)
			/*if (threadIdx.x == 0 && threadIdx.y == 0 && blockIdx.y == 0 && blockIdx.x == 0) {
				printf("%d\n", rulesCount);
			}*/
			int i = rulesCount;
				float iter = ceilf((float)(inputStringLength - i) / (float)gridDim.x);
				//iter = 2.0f;

				//for (int r = 0; r < (int)iter; r++) {

				int temp_bidx = bidx;// +(r * gridDim.x);

					if (temp_bidx < inputStringLength - i) {
						int j = temp_bidx; //J

						/*if (threadIdx.x == 0 && threadIdx.y == 0 && blockIdx.y == 0 && blockIdx.x == 0) {
							printf("%d\n", gridDim.x);
						}*/

										   //for (int p = 0; p < rulesCount; p++) { //for each production (each rule)
						if (idx < 19) { //TODO wtf?

							int p = idx;

							for (int k = 0; k < i; k++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

														  //TODO correct split points!
								int first = cykArray[k][j];
								int second = cykArray[i - k - 1][j + k + 1];

								//decode nonterminals (find out if bits are on a given positions)
								int base = 1;
								int bitMaskFirst = base << rulesArray[0][p];
								int bitMaskSecond = base << rulesArray[1][p];
								if (first & bitMaskFirst && second & bitMaskSecond) {

									int shiftValue = rulesArray[2][p];
									int bitValue = base << shiftValue;
									//TODO - tutaj może być problem
									atomicOr(&cykArray[i][j], bitValue);
								}

							}
						}

					}
				//}

				//break; //only first line
				//if (idx == 0 && idy == 0) {
				//	//printf("%d | ", g_mutex);
				//	atomicAdd((int *)&g_mutex, 1);
				//	//only when all blocks add 1 to g_mutex
				//	//will g_mutex equal to goalVal
				//	while (g_mutex != (gridDim.x * i)) {
				//		//Do nothing here
				//	}

				//}
				__syncthreads();

			//}
		}

		// only thread 0 is used for synchronization


	} else if (action == 1) { //////////////////////////////////////////////////////////////// blocks + threads


		int numberOfProductions = rulesArray[blockIdx.y][1];


		if (true) {

			//if (idx == 0 && idy == 0) {
			//	printf("HEHE %d\n", rulesArray[blockIdx.y][1]);
			//}
			
			for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)

														  //for (int j = 0; j < inputStringLength - i; j++) { // every word of given length 5, 4, 3, 2, 1... (2)

				float iter = ceilf((float)(inputStringLength - i) / (float)gridDim.x);
				//iter = 2.0f;

				for (int r = 0; r < (int)iter; r++) {

					int temp_bidx = bidx + (r * gridDim.x);

					if (temp_bidx < inputStringLength - i) {
						int j = temp_bidx; //J

										   //for (int p = 0; p < rulesCount; p++) { //for each production (each rule)
						if (idx < numberOfProductions) {

							int p = idx;

							for (int k = 0; k < i; k++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

														  //TODO correct split points!
								int first = cykArray[k][j];
								int second = cykArray[i - k - 1][j + k + 1];

								//decode nonterminals (find out if bits are on a given positions)
								int base = 1;
								int bitMaskFirst = base << rulesArray[blockIdx.y][(p + 1) * 2];
								int bitMaskSecond = base << rulesArray[blockIdx.y][(p + 1) * 2 + 1];
								if (first & bitMaskFirst && second & bitMaskSecond) {

									int shiftValue = rulesArray[blockIdx.y][0];
									int bitValue = base << shiftValue;
									//TODO - tutaj może być problem
									atomicOr(&cykArray[i][j], bitValue);
								}

							}
						}

					}
				}

				//break; //only first line
				if (idx == 0 && idy == 0) {
					//printf("%d | ", g_mutex);
					atomicAdd((int *)&g_mutex, 1);
					//only when all blocks add 1 to g_mutex
					//will g_mutex equal to goalVal
					while (g_mutex != (gridDim.y * gridDim.x * i)) {
						//Do nothing here
					}

				}
				__syncthreads();

			}
		}

		// only thread 0 is used for synchronization


	}

	__syncthreads();
	//&& rulesCount == inputStringLength
	if (threadIdx.x == 0 && threadIdx.y == 0 && blockIdx.y == 0 && blockIdx.x == 0 ) {
		if (action != 2 || rulesCount == inputStringLength -1) {
			for (int i = 0; i < nonTermsCount; i++) {
				for (int j = 0; j < nonTermsCount; j++) {
					//cout << rulesNonTermsArray[i][j] << " | ";

					printf("%d | ", rulesNonTermsArray[i][j]);
				}
				//cout << endl;

				printf("\n");
			}

			for (int j = 1; j < inputStringLength; j++) {
				for (int i = 0; i < inputStringLength - j; i++) {
					printf("%d | ", cykArray[j][i]);
				}
				printf("\n");
			}

			int* result = data.getResult();
			printf("RESUUUULt: %d | ", result[0]);
			result[0] = 1337;

			/*for (int i = 0; i < 3; i++) {
			for (int j = 0; j < rulesCount; j++) {
			printf("%d | ", rulesArray[i][j]);
			}
			printf("\n");
			}*/
		}
		
	}

	__syncthreads();

	return;
}


