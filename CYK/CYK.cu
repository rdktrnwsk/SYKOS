#include "hip/hip_runtime.h"
﻿#include "CYK.cuh"

template<int action>
__global__ void cykAlgorithm(DeviceCYKData data, hiprandState * randGlobal)
{
	/*if (action == 1 || action == 2) {
		printf("%d", threadIdx.x);
	}*/

	__shared__ int** cykArray;
	__shared__ int inputStringLength;
	__shared__ int** rulesNonTermsArray;
	__shared__ int nonTermsCount;

	if (threadIdx.x == 0 && threadIdx.y == 0) {
		cykArray = data.getCYKArray();
		inputStringLength = data.getInputCount();
		rulesNonTermsArray = data.getRulesNonTermsArray();
		nonTermsCount = data.getNonTermsCount();
	}

	__syncthreads();

	if (action == 1) {
		if (threadIdx.x == 0) {
			int idx = threadIdx.x;

			for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)
				
				for (int j = 0; j < inputStringLength - i; j++) { // every word of given length 5, 4, 3, 2, 1... (2)

					for (int k = 0; k < i; k++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

												  //TODO correct split points!
						int first = cykArray[k][j];
						int second = cykArray[i - k - 1][j + k + 1];

						//decode nonterminals (find out if bits are on a given positions)
						int base = 1;
						for (int l = 0; l < nonTermsCount; l++) {
						
							int bitMaskFirst = base << l;

							// all possibilities connected with rules
							for (int m = 0; m < nonTermsCount; m++) {
								int bitMaskSecond = base << m;

								// if rule with 'l' index and 'm' index is created and ready to be found if corrrect X ->lm (does X exist in a grammar?)
								if (first & bitMaskFirst && second & bitMaskSecond) {

									//rule exists
									if (rulesNonTermsArray[l][m] != -1) {
										int shiftValue = rulesNonTermsArray[l][m];
										int bitValue = base << shiftValue;

										cykArray[i][j] |= bitValue;
									}

								}

							}

						} // l loop end

					}

				}

				//break; //only first line

			}
		} 
	} else if (action == 2) { //////////////////////////////////////////////////////////////// each entry 1D threads (last loop 1)

		if (threadIdx.x <= 32) {
			int idx = threadIdx.x;

			for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)

				for (int j = 0; j < inputStringLength - i; j++) { // every word of given length 5, 4, 3, 2, 1... (2)

					for (int k = 0; k < i; k++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

												  //TODO correct split points!
						int first = cykArray[k][j];
						int second = cykArray[i - k - 1][j + k + 1];

						//decode nonterminals (find out if bits are on a given positions)
						int base = 1;
						//for (int l = 0; l < nonTermsCount; l++) {
						int l = idx;
						int bitMaskFirst = base << l;
						//all possibilities connected with rules

						for (int m = 0; m < nonTermsCount; m++) {
							int bitMaskSecond = base << m;

							// if rule with 'l' index and 'm' index is created and ready to be found if corrrect X ->lm (does X exist in a grammar?)
							if (first & bitMaskFirst && second & bitMaskSecond) {
								//cout << bitMaskFirst << ", " << bitMaskSecond << " | ";

								//rule exists
								if (rulesNonTermsArray[l][m] != -1) {
									int shiftValue = rulesNonTermsArray[l][m];
									int bitValue = base << shiftValue;

									//TODO - tutaj może być problem
									atomicOr(&cykArray[i][j], bitValue);

									//cykArray[i][j] |= bitValue;
								}

							}


						}

						//} // l loop end

						//cout << first << " | " << second << endl;

						//combinations of productions

						// for each production (rulesNonTerminals)

					}

				}

				//break; //only first line

			}
		}
	} else if (action == 3) { //////////////////////////////////////////////////////////////// each entry 2D threads (last loop 1)

	 if (threadIdx.x <= nonTermsCount && threadIdx.y <= nonTermsCount) {
		 int idx = threadIdx.x;
		 int idy = threadIdx.y;

		 for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)

			 for (int j = 0; j < inputStringLength - i; j++) { // every word of given length 5, 4, 3, 2, 1... (2)

				 for (int k = 0; k < i; k++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

											   //TODO correct split points!
					 int first = cykArray[k][j];
					 int second = cykArray[i - k - 1][j + k + 1];

					 //decode nonterminals (find out if bits are on a given positions)
					 int base = 1;
					 //for (int l = 0; l < nonTermsCount; l++) {
					 int l = idx;
					 int bitMaskFirst = base << l;
					 //all possibilities connected with rules

					 //for (int m = 0; m < nonTermsCount; m++) {

						 int m = idy;

						 int bitMaskSecond = base << m;

						 // if rule with 'l' index and 'm' index is created and ready to be found if corrrect X ->lm (does X exist in a grammar?)
						 if (first & bitMaskFirst && second & bitMaskSecond) {
							 //cout << bitMaskFirst << ", " << bitMaskSecond << " | ";

							 //rule exists
							 if (rulesNonTermsArray[l][m] != -1) {
								 int shiftValue = rulesNonTermsArray[l][m];
								 int bitValue = base << shiftValue;

								 //TODO - tutaj może być problem
								 atomicOr(&cykArray[i][j], bitValue);

								 //cykArray[i][j] |= bitValue;
							 }

						 }


					// }

					 //} // l loop end

					 //cout << first << " | " << second << endl;

					 //combinations of productions

					 // for each production (rulesNonTerminals)

				 }

			 }

			 //break; //only first line

		 }
	 }
 }

	
	
	__syncthreads();
	
	if (threadIdx.x == 0 && threadIdx.y == 0) {

		for (int i = 0; i < nonTermsCount; i++) {
			for (int j = 0; j < nonTermsCount; j++) {
				//cout << rulesNonTermsArray[i][j] << " | ";

				printf("%d | ", rulesNonTermsArray[i][j]);
			}
			//cout << endl;

			printf("\n");
		}

		for (int j = 1; j < inputStringLength; j++) {
			for (int i = 0; i < inputStringLength - j; i++) {
				printf("%d | ", cykArray[j][i]);
			}
			printf("\n");
		}

		int* result = data.getResult();
		printf("RESUUUULt: %d | ", result[0]);
		result[0] = 1337;
	}

	__syncthreads();

	return;
}

/*                                                                                      GLOBAL FUNCTION                                                                     */

__device__ volatile int g_mutex;

template<int action>
__global__ void cykAlgorithmCooperative(DeviceCYKData data, hiprandState * randGlobal)
{
	__shared__ int** cykArray;
	__shared__ int inputStringLength;
	__shared__ int** rulesNonTermsArray;
	__shared__ int nonTermsCount;

	int bidx = blockIdx.x;
	int idx = threadIdx.x;
	int idy = threadIdx.y;

	if (threadIdx.x == 0 && threadIdx.y == 0) {
		g_mutex = 0;
		cykArray = data.getCYKArray();
		inputStringLength = data.getInputCount();
		rulesNonTermsArray = data.getRulesNonTermsArray();
		nonTermsCount = data.getNonTermsCount();
		//printf("HEHEHEHEHEHHE:%d | ", bidx);
	}


	__syncthreads();

	if (action == 0) { //////////////////////////////////////////////////////////////// entry = block, block used are decremented, threads 2D

		if (threadIdx.x <= nonTermsCount && threadIdx.y <= nonTermsCount) {
		

			for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)

				//for (int j = 0; j < inputStringLength - i; j++) { // every word of given length 5, 4, 3, 2, 1... (2)
				if (bidx < inputStringLength - i) {
					int j = bidx;

					for (int k = 0; k < i; k++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

												  //TODO correct split points!
						int first = cykArray[k][j];
						int second = cykArray[i - k - 1][j + k + 1];

						//decode nonterminals (find out if bits are on a given positions)
						int base = 1;
						//for (int l = 0; l < nonTermsCount; l++) {
						int l = idx;
						int bitMaskFirst = base << l;
						//all possibilities connected with rules

						//for (int m = 0; m < nonTermsCount; m++) {

						int m = idy;

						int bitMaskSecond = base << m;

						// if rule with 'l' index and 'm' index is created and ready to be found if corrrect X ->lm (does X exist in a grammar?)
						if (first & bitMaskFirst && second & bitMaskSecond) {
							//cout << bitMaskFirst << ", " << bitMaskSecond << " | ";

							//rule exists
							if (rulesNonTermsArray[l][m] != -1) {
								int shiftValue = rulesNonTermsArray[l][m];
								int bitValue = base << shiftValue;

								//TODO - tutaj może być problem
								atomicOr(&cykArray[i][j], bitValue);

								//cykArray[i][j] |= bitValue;
							}

						}


						// }

						//} // l loop end

						//cout << first << " | " << second << endl;

						//combinations of productions

						// for each production (rulesNonTerminals)

					}

				}

				//break; //only first line
				if (idx == 0 && idy == 0) {
					atomicAdd((int *)&g_mutex, 1);
					//only when all blocks add 1 to g_mutex
					//will g_mutex equal to goalVal
					while (g_mutex != (10 * i)) {
						//Do nothing here
					}
				}
				__syncthreads();

			}
		}

		// only thread 0 is used for synchronization


	}

	


	__syncthreads();

	if (threadIdx.x == 0 && threadIdx.y == 0 && bidx == 0) {

		for (int i = 0; i < nonTermsCount; i++) {
			for (int j = 0; j < nonTermsCount; j++) {
				//cout << rulesNonTermsArray[i][j] << " | ";

				printf("%d | ", rulesNonTermsArray[i][j]);
			}
			//cout << endl;

			printf("\n");
		}

		for (int j = 1; j < inputStringLength; j++) {
			for (int i = 0; i < inputStringLength - j; i++) {
				printf("%d | ", cykArray[j][i]);
			}
			printf("\n");
		}

		int* result = data.getResult();
		printf("RESUUUULt: %d | ", result[0]);
		result[0] = 1337;
	}

	__syncthreads();

	return;
}


