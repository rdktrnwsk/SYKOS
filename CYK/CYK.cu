#include "hip/hip_runtime.h"
﻿#include "CYK.cuh"

template<int action>
__global__ void cykAlgorithm(DeviceCYKData data, hiprandState * randGlobal)
{
	/*if (action == 1 || action == 2) {
		printf("%d", threadIdx.x);
	}*/

	__shared__ int** cykArray;
	__shared__ int inputStringLength;
	__shared__ int** rulesNonTermsArray;
	__shared__ int nonTermsCount;

	if (threadIdx.x == 0 && threadIdx.y == 0) {
		cykArray = data.getCYKArray();
		inputStringLength = data.getInputCount();
		rulesNonTermsArray = data.getRulesNonTermsArray();
		nonTermsCount = data.getNonTermsCount();
	}

	__syncthreads();

	if (action == 1) {
		if (threadIdx.x == 0) {
			int idx = threadIdx.x;

			for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)
				
				for (int j = 0; j < inputStringLength - i; j++) { // every word of given length 5, 4, 3, 2, 1... (2)

					for (int k = 0; k < i; k++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

												  //TODO correct split points!
						int first = cykArray[k][j];
						int second = cykArray[i - k - 1][j + k + 1];

						//decode nonterminals (find out if bits are on a given positions)
						int base = 1;
						for (int l = 0; l < nonTermsCount; l++) {
						
							int bitMaskFirst = base << l;

							// all possibilities connected with rules
							for (int m = 0; m < nonTermsCount; m++) {
								int bitMaskSecond = base << m;

								// if rule with 'l' index and 'm' index is created and ready to be found if corrrect X ->lm (does X exist in a grammar?)
								if (first & bitMaskFirst && second & bitMaskSecond) {

									//rule exists
									if (rulesNonTermsArray[l][m] != -1) {
										int shiftValue = rulesNonTermsArray[l][m];
										int bitValue = base << shiftValue;

										cykArray[i][j] |= bitValue;
									}

								}

							}

						} // l loop end

					}

				}

				//break; //only first line

			}
		} 
	} else if (action == 2) { //////////////////////////////////////////////////////////////// each entry 1D threads (last loop 1)

		if (threadIdx.x <= 32) {
			int idx = threadIdx.x;

			for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)

				for (int j = 0; j < inputStringLength - i; j++) { // every word of given length 5, 4, 3, 2, 1... (2)

					for (int k = 0; k < i; k++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

												  //TODO correct split points!
						int first = cykArray[k][j];
						int second = cykArray[i - k - 1][j + k + 1];

						//decode nonterminals (find out if bits are on a given positions)
						int base = 1;
						//for (int l = 0; l < nonTermsCount; l++) {
						int l = idx;
						int bitMaskFirst = base << l;
						//all possibilities connected with rules

						for (int m = 0; m < nonTermsCount; m++) {
							int bitMaskSecond = base << m;

							// if rule with 'l' index and 'm' index is created and ready to be found if corrrect X ->lm (does X exist in a grammar?)
							if (first & bitMaskFirst && second & bitMaskSecond) {
								//cout << bitMaskFirst << ", " << bitMaskSecond << " | ";

								//rule exists
								if (rulesNonTermsArray[l][m] != -1) {
									int shiftValue = rulesNonTermsArray[l][m];
									int bitValue = base << shiftValue;

									//TODO - tutaj może być problem
									atomicOr(&cykArray[i][j], bitValue);

									//cykArray[i][j] |= bitValue;
								}

							}


						}

						//} // l loop end

						//cout << first << " | " << second << endl;

						//combinations of productions

						// for each production (rulesNonTerminals)

					}

				}

				//break; //only first line

			}
		}
	} else if (action == 3) { //////////////////////////////////////////////////////////////// each entry 2D threads (last loop 1)

	 if (threadIdx.x <= nonTermsCount && threadIdx.y <= nonTermsCount) {
		 int idx = threadIdx.x;
		 int idy = threadIdx.y;

		 for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)

			 for (int j = 0; j < inputStringLength - i; j++) { // every word of given length 5, 4, 3, 2, 1... (2)

				 for (int k = 0; k < i; k++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

											   //TODO correct split points!
					 int first = cykArray[k][j];
					 int second = cykArray[i - k - 1][j + k + 1];

					 //decode nonterminals (find out if bits are on a given positions)
					 int base = 1;
					 //for (int l = 0; l < nonTermsCount; l++) {
					 int l = idx;
					 int bitMaskFirst = base << l;
					 //all possibilities connected with rules

					 //for (int m = 0; m < nonTermsCount; m++) {

						 int m = idy;

						 int bitMaskSecond = base << m;

						 // if rule with 'l' index and 'm' index is created and ready to be found if corrrect X ->lm (does X exist in a grammar?)
						 if (first & bitMaskFirst && second & bitMaskSecond) {
							 //cout << bitMaskFirst << ", " << bitMaskSecond << " | ";

							 //rule exists
							 if (rulesNonTermsArray[l][m] != -1) {
								 int shiftValue = rulesNonTermsArray[l][m];
								 int bitValue = base << shiftValue;

								 //TODO - tutaj może być problem
								 atomicOr(&cykArray[i][j], bitValue);

								 //cykArray[i][j] |= bitValue;
							 }

						 }


					// }

					 //} // l loop end

					 //cout << first << " | " << second << endl;

					 //combinations of productions

					 // for each production (rulesNonTerminals)

				 }

			 }

			 //break; //only first line

		 }
	 }
 }	else if (action == 4) { //////////////////////////////////////////////////////////////// magisterka rozwiazanie pierwsze

	 if (threadIdx.x < inputStringLength) {
		 int idx = threadIdx.x;

		 for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)

			 //for (int j = 0; j < inputStringLength - i; j++) { // every word of given length 5, 4, 3, 2, 1... (2)

			 if (threadIdx.x < inputStringLength - i) {
				 int j = threadIdx.x;

				 for (int k = 0; k < i; k++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

											   //TODO correct split points!
					 int first = cykArray[k][j];
					 int second = cykArray[i - k - 1][j + k + 1];

					 //decode nonterminals (find out if bits are on a given positions)
					 int base = 1;
					 for (int l = 0; l < nonTermsCount; l++) {

						 int bitMaskFirst = base << l;

						 // all possibilities connected with rules
						 for (int m = 0; m < nonTermsCount; m++) {
							 int bitMaskSecond = base << m;

							 // if rule with 'l' index and 'm' index is created and ready to be found if corrrect X ->lm (does X exist in a grammar?)
							 if (first & bitMaskFirst && second & bitMaskSecond) {

								 //rule exists
								 if (rulesNonTermsArray[l][m] != -1) {
									 int shiftValue = rulesNonTermsArray[l][m];
									 int bitValue = base << shiftValue;

									 cykArray[i][j] |= bitValue;
								 }

							 }

						 }

					 } // l loop end

				 }

			 }

			 __syncthreads();

			 //break; //only first line

		 }
	 }
	}
 else if (action == 5) { //////////////////////////////////////////////////////////////// magisterka rozwiazanie drugie

	 int idx = threadIdx.x;
	 int idy = threadIdx.y;


	 if (1 == 1) { 

		 for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)

			 /*int pidx = i - idy - 1;
			 int pidy = idx + idy + 1;*/


			 //for (int j = 0; j < inputStringLength - i; j++) { // every word of given length 5, 4, 3, 2, 1... (2)

			 float iter = ceilf((float)(inputStringLength - i) / (float)blockDim.x);
			 //iter = 2.0f;

			 for (int r = 0; r < (int)iter; r++) {

				 int temp_idx = idx + (r * blockDim.x);

				 float iter2 = ceilf((float)(i) / (float)blockDim.y);

				 for (int s = 0; s < (int)iter2; s++) {

					 int temp_idy = idy + (s * blockDim.y);

				 if (temp_idx < inputStringLength - i && temp_idy < i) {
					 int j = temp_idx;
					 int k = temp_idy;


					 //for (int k = 0; k < i; k++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

					 

						 //TODO correct split points!
						 int first = cykArray[k][j];
						 int second = cykArray[i - k - 1][j + k + 1];

						 //decode nonterminals (find out if bits are on a given positions)
						 int base = 1;
						 for (int l = 0; l < nonTermsCount; l++) {

							 int bitMaskFirst = base << l;

							 // all possibilities connected with rules
							 for (int m = 0; m < nonTermsCount; m++) {
								 int bitMaskSecond = base << m;

								 // if rule with 'l' index and 'm' index is created and ready to be found if corrrect X ->lm (does X exist in a grammar?)
								 if (first & bitMaskFirst && second & bitMaskSecond) {

									 //rule exists
									 if (rulesNonTermsArray[l][m] != -1) {
										 int shiftValue = rulesNonTermsArray[l][m];
										 int bitValue = base << shiftValue;

										 cykArray[i][j] |= bitValue;
									 }

								 }

							 }

						 } // l loop end
				 

					 //} // k loop

					 }
				 } // end s loop

		 } // r loop

			 __syncthreads();

			 //break; //only first line

		 }
	 }
 }

	
	
	__syncthreads();
	
	//if (threadIdx.x == 0 && threadIdx.y == 0) {

	//	for (int i = 0; i < nonTermsCount; i++) {
	//		for (int j = 0; j < nonTermsCount; j++) {
	//			//cout << rulesNonTermsArray[i][j] << " | ";

	//			printf("%d | ", rulesNonTermsArray[i][j]);
	//		}
	//		//cout << endl;

	//		printf("\n");
	//	}

	//	for (int j = 1; j < inputStringLength; j++) {
	//		for (int i = 0; i < inputStringLength - j; i++) {
	//			printf("%d | ", cykArray[j][i]);
	//		}
	//		printf("\n");
	//	}

	//	int* result = data.getResult();
	//	printf("RESUUUULt: %d | ", result[0]);
	//	result[0] = 1337;
	//}

	__syncthreads();

	return;
}

/*                                                                                      GLOBAL FUNCTION                                                                     */

__device__ volatile int g_mutex;

template<int action>
__global__ void cykAlgorithmCooperative(DeviceCYKData data, hiprandState * randGlobal, volatile int* arrayIn, volatile int* arrayOut)
{
	__shared__ int** cykArray;
	__shared__ int inputStringLength;
	__shared__ int** rulesNonTermsArray;
	__shared__ int nonTermsCount;

	int bidx = blockIdx.x;
	int idx = threadIdx.x;
	int idy = threadIdx.y;

	if (threadIdx.x == 0 && threadIdx.y == 0) {
		g_mutex = 0;
		cykArray = data.getCYKArray();
		inputStringLength = data.getInputCount();
		rulesNonTermsArray = data.getRulesNonTermsArray();
		nonTermsCount = data.getNonTermsCount();
		//printf("Dim %d\n", gridDim.x);
	}


	__syncthreads();

	if (action == 0) { //////////////////////////////////////////////////////////////// entry = block, block used are decremented, threads 2D

		if (threadIdx.x <= nonTermsCount && threadIdx.y <= nonTermsCount) {
		
			for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)

				//for (int j = 0; j < inputStringLength - i; j++) { // every word of given length 5, 4, 3, 2, 1... (2)

				float iter = ceilf((float)(inputStringLength - i) / (float)gridDim.x);
				//iter = 2.0f;

				for (int r = 0; r < (int)iter ; r++) {

					int temp_bidx = bidx + (r * gridDim.x);

					if (temp_bidx < inputStringLength - i) {
						int j = temp_bidx;

						for (int k = 0; k < i; k++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

													  //TODO correct split points!
							int first = cykArray[k][j];
							int second = cykArray[i - k - 1][j + k + 1];

							//decode nonterminals (find out if bits are on a given positions)
							int base = 1;
							//for (int l = 0; l < nonTermsCount; l++) {
							int l = idx;
							int bitMaskFirst = base << l;
							//all possibilities connected with rules

							//for (int m = 0; m < nonTermsCount; m++) {

							int m = idy;

							int bitMaskSecond = base << m;

							// if rule with 'l' index and 'm' index is created and ready to be found if corrrect X ->lm (does X exist in a grammar?)
							if (first & bitMaskFirst && second & bitMaskSecond) {
								//cout << bitMaskFirst << ", " << bitMaskSecond << " | ";

								//rule exists
								if (rulesNonTermsArray[l][m] != -1) {
									int shiftValue = rulesNonTermsArray[l][m];
									int bitValue = base << shiftValue;

									//TODO - tutaj może być problem
									atomicOr(&cykArray[i][j], bitValue);

									//cykArray[i][j] |= bitValue;
								}

							}


							// }

							//} // l loop end

							//cout << first << " | " << second << endl;

							//combinations of productions

							// for each production (rulesNonTerminals)

						}

					}
				}

				//break; //only first line
				if (idx == 0 && idy == 0) {
					//printf("%d | ", g_mutex);
					atomicAdd((int *)&g_mutex, 1);
					//only when all blocks add 1 to g_mutex
					//will g_mutex equal to goalVal
					while (g_mutex != (gridDim.x * i)) {
						//Do nothing here
					}
					
				}
				__syncthreads();

			}
		}

		// only thread 0 is used for synchronization


	} else if (action == 1) { //////////////////////////////////////////////////////////////// entry = block, block used are decremented, threads 2D + FAST BARIER SYNCHRONISATION

		if (threadIdx.x <= nonTermsCount && threadIdx.y <= nonTermsCount) {


			for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)

				__syncthreads();
														  //for (int j = 0; j < inputStringLength - i; j++) { // every word of given length 5, 4, 3, 2, 1... (2)

				float iter = ceilf((float)(inputStringLength - i) / (float)gridDim.x);
				//iter = 2.0f;

				for (int j = 0; j < (int)iter; j++) {

					int temp_bidx = bidx + (j * gridDim.x);

					if (temp_bidx < inputStringLength - i) {
						int j = temp_bidx;

						for (int k = 0; k < i; k++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

													  //TODO correct split points!
							int first = cykArray[k][j];
							int second = cykArray[i - k - 1][j + k + 1];

							//decode nonterminals (find out if bits are on a given positions)
							int base = 1;
							//for (int l = 0; l < nonTermsCount; l++) {
							int l = idx;
							int bitMaskFirst = base << l;
							//all possibilities connected with rules

							//for (int m = 0; m < nonTermsCount; m++) {

							int m = idy;

							int bitMaskSecond = base << m;

							// if rule with 'l' index and 'm' index is created and ready to be found if corrrect X ->lm (does X exist in a grammar?)
							if (first & bitMaskFirst && second & bitMaskSecond) {
								//cout << bitMaskFirst << ", " << bitMaskSecond << " | ";

								//rule exists
								if (rulesNonTermsArray[l][m] != -1) {
									int shiftValue = rulesNonTermsArray[l][m];
									int bitValue = base << shiftValue;

									//TODO - tutaj może być problem
									atomicOr(&cykArray[i][j], bitValue);

									//cykArray[i][j] |= bitValue;
								}

							}


							// }

							//} // l loop end

							//cout << first << " | " << second << endl;

							//combinations of productions

							// for each production (rulesNonTerminals)

						}

					}
				}

				if (idx == 0 && idy == 0) {
					arrayIn[bidx] = i;
				}

				if (bidx == 1) {

					if (idx < gridDim.x && idy == 0) {
						while (arrayIn[idx] != i) {

						}
					}

					__syncthreads();

					if (idx < gridDim.x && idy == 0) {
						arrayOut[idx] = i;
					}

				}

				//break; //only first line
				if (idx == 0 && idy == 0) {
					while (arrayOut[bidx] != i) {

					}
				}
				__syncthreads();

			}
		}

		// only thread 0 is used for synchronization


	}

	


	__syncthreads();

	//if (threadIdx.x == 0 && threadIdx.y == 0 && bidx == 0) {
	//	
	//	for (int i = 0; i < nonTermsCount; i++) {
	//		for (int j = 0; j < nonTermsCount; j++) {
	//			//cout << rulesNonTermsArray[i][j] << " | ";

	//			printf("%d | ", rulesNonTermsArray[i][j]);
	//		}
	//		//cout << endl;

	//		printf("\n");
	//	}

	//	for (int j = 1; j < inputStringLength; j++) {
	//		for (int i = 0; i < inputStringLength - j; i++) {
	//			printf("%d | ", cykArray[j][i]);
	//		}
	//		printf("\n");
	//	}

	//	int* result = data.getResult();
	//	printf("RESUUUULt: %d | ", result[0]);
	//	result[0] = 1337;
	//}

	__syncthreads();

	return;
}


