#include "hip/hip_runtime.h"
﻿#include "CYK.cuh"

template<int action>
__global__ void cykAlgorithm(DeviceCYKData data, hiprandState * randGlobal) {

	__shared__ int** cykArray;
	__shared__ int inputStringLength;
	__shared__ int** rulesNonTermsArray;
	__shared__ int nonTermsCount;
	__shared__ int cellWidth;

	if (threadIdx.x == 0 && threadIdx.y == 0) {
		cykArray = data.getCYKArray();
		inputStringLength = data.getInputCount();
		rulesNonTermsArray = data.getRulesNonTermsArray();
		nonTermsCount = data.getNonTermsCount();
		cellWidth = (int)(ceilf(((float)nonTermsCount / 32.0f)));
	}

	__syncthreads();

	if (action == 0) {
		if (threadIdx.x == 0) {
			int idx = threadIdx.x;

			for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)

				for (int j = 0; j < inputStringLength - i; j++) { // every word of given length 5, 4, 3, 2, 1... (2)

					for (int k = 0; k < i; k++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

												  //TODO correct split points!
						int first = cykArray[k][j];
						int second = cykArray[i - k - 1][j + k + 1];

						//decode nonterminals (find out if bits are on a given positions)
						int base = 1;
						for (int l = 0; l < nonTermsCount; l++) {

							int bitMaskFirst = base << l;


							if (first & bitMaskFirst) {

								// all possibilities connected with rules
								for (int m = 0; m < nonTermsCount; m++) {
									int bitMaskSecond = base << m;

									// if rule with 'l' index and 'm' index is created and ready to be found if corrrect X ->lm (does X exist in a grammar?)
									if (second & bitMaskSecond) {

										//rule exists
										if (rulesNonTermsArray[l][m] != -1) {
											int shiftValue = rulesNonTermsArray[l][m];
											int bitValue = base << shiftValue;

											cykArray[i][j] |= bitValue;
										}

									}

								}
							}

						} // l loop end

					}

				}

				//break; //only first line

			}
		}
	} else if (action == 1) {

		int idx = threadIdx.x;

		if (idx < nonTermsCount) {
			
			for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)
				
				for (int j = 0; j < inputStringLength - i; j++) { // every word of given length 5, 4, 3, 2, 1... (2)

					for (int k = 0; k < i; k++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

												  //TODO correct split points!
						int first = cykArray[k][j];
						int second = cykArray[i - k - 1][j + k + 1];

						//decode nonterminals (find out if bits are on a given positions)
						int base = 1;
						for (int l = 0; l < nonTermsCount; l++) {
						
							int bitMaskFirst = base << l;

							if (first & bitMaskFirst) {
								// all possibilities connected with rules
								//for (int m = 0; m < nonTermsCount; m++) {
								int m = idx;

								int bitMaskSecond = base << m;

								// if rule with 'l' index and 'm' index is created and ready to be found if corrrect X ->lm (does X exist in a grammar?)
								//if (first & bitMaskFirst && second & bitMaskSecond) {
								if (second & bitMaskSecond) {

									//rule exists
									if (rulesNonTermsArray[l][m] != -1) {
										int shiftValue = rulesNonTermsArray[l][m];
										int bitValue = base << shiftValue;

										cykArray[i][j] |= bitValue;
									}

								}

								//}
							}

						} // l loop end

					}

				}

				//break; //only first line

			}
		} 
	} else if (action == 2) { //////////////////////////////////////////////////////////////// each entry 1D threads (last loop 1)

		int idx = threadIdx.x;

		if (idx < nonTermsCount) {
			for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)

				for (int j = 0; j < inputStringLength - i; j++) { // every word of given length 5, 4, 3, 2, 1... (2)

					for (int k = 0; k < i; k++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

												  //TODO correct split points!
						int first = cykArray[k][j];
						int second = cykArray[i - k - 1][j + k + 1];

						//decode nonterminals (find out if bits are on a given positions)
						int base = 1;
						//for (int m = 0; m < nonTermsCount; m++) {
						int m = idx;
						int bitMaskFirst = base << m;
						//all possibilities connected with rules
						if (first & bitMaskFirst) {

					
							for (int n = 0; n < nonTermsCount; n++) {
								int bitMaskSecond = base << n;

								// if rule with 'm' index and 'n' index is created and ready to be found if corrrect X ->mn (does X exist in a grammar?)
								if (second & bitMaskSecond) {
									//cout << bitMaskFirst << ", " << bitMaskSecond << " | ";

									//rule exists
									if (rulesNonTermsArray[m][n] != -1) {
										int shiftValue = rulesNonTermsArray[m][n];
										int bitValue = base << shiftValue;

										//TODO - tutaj może być problem
										atomicOr(&cykArray[i][j], bitValue);
										//cykArray[i][j] |= bitValue;
									}

								}

							}
						}


					}

				}

			}
		}

		
	} else if (action == 3) { //////////////////////////////////////////////////////////////// each entry 2D threads (last loop 1)

	 if (threadIdx.x < nonTermsCount && threadIdx.y < nonTermsCount) {
		 int idx = threadIdx.x;
		 int idy = threadIdx.y;

		 //for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)

			// for (int j = 0; j < inputStringLength - i; j++) { // every word of given length 5, 4, 3, 2, 1... (2)

			//	 for (int k = 0; k < i; k++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

			//								   //TODO correct split points!
			//		 int first = cykArray[k][j];
			//		 int second = cykArray[i - k - 1][j + k + 1];

			//		 //decode nonterminals (find out if bits are on a given positions)
			//		 int base = 1;

			//		 int l = idx;
			//		 int bitMaskFirst = base << l;



			//			 int m = idy;

			//			 int bitMaskSecond = base << m;

			//			 // if rule with 'l' index and 'm' index is created and ready to be found if corrrect X ->lm (does X exist in a grammar?)
			//			 if (first & bitMaskFirst && second & bitMaskSecond) {
			//				 //cout << bitMaskFirst << ", " << bitMaskSecond << " | ";

			//				 //rule exists
			//				 if (rulesNonTermsArray[l][m] != -1) {
			//					 int shiftValue = rulesNonTermsArray[l][m];
			//					 int bitValue = base << shiftValue;

			//					 //TODO - tutaj może być problem
			//					 atomicOr(&cykArray[i][j], bitValue);

			//					 //cykArray[i][j] |= bitValue;
			//				 }

			//			 }

			//	 }

			// }

		 //}
		 for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)

			 for (int j = 0; j < (inputStringLength - i) * cellWidth; j += cellWidth) { // every word <of given length: 5 words, 4 words, 3 words, 2, 1...> (2)

				 for (int k = 0; k < i; k++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

											   //decode nonterminals (find out if bits are on a given positions)
					 int base = 1;
					 float iterM = ceilf((float)(nonTermsCount) / (float)blockDim.x);
					 //for (int m = 0; m < nonTermsCount; m++) {
					 for (int md = 0; md < (int)iterM; md++) {

						int temp_idx = idx + (md * blockDim.x);
					 
						int m = temp_idx;

						 int offset = (int)(m / 32);
						 int first = cykArray[k][j + offset];
						 int bitMaskFirst = (base << (m - (offset * 32)));

						 if (first & bitMaskFirst) {

							 float iterN = ceilf((float)(nonTermsCount) / (float)blockDim.y);
							 // for (int n = 0; n < nonTermsCount; n++) {
							 for (int nd = 0; nd < (int)iterN; nd++) {

								 int temp_idy = idy + (nd * blockDim.y);
							 //all possibilities connected with rules
							
								int n = temp_idy;

								 int offset2 = (int)(n / 32); // shift by 32 is the next cell
								 int second = cykArray[i - k - 1][(((j / cellWidth) + k + 1) * cellWidth) + offset2];
								 int bitMaskSecond = (base << (n - (offset2 * 32)));

								 if (second & bitMaskSecond) {

									 //rule exists
									 if (rulesNonTermsArray[m][n] != -1) {
										 int shiftValue = rulesNonTermsArray[m][n];
										 //int bitValue = base << shiftValue;

										 int offset = (int)(shiftValue / 32); // shift by 32 is the next cell
										 int base = 1;
										 int bitValue = base << (shiftValue - (offset * 32));

										 cykArray[i][j + offset] |= bitValue;
									 }

								 }

							 } // end n loop
						 }

					 } // end m loop

				 } // end k loop

			 } // end j loop

		 } // end i loop
	 }
 }	else if (action == 4) { //////////////////////////////////////////////////////////////// magisterka rozwiazanie pierwsze

	 //int idx = threadIdx.x;

	 ////if (idx < inputStringLength) {
		//
		// for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)

		//	 //for (int j = 0; j < inputStringLength - i; j++) { // every word of given length 5, 4, 3, 2, 1... (2)

		//	 if (threadIdx.x < inputStringLength - i) {
		//		 int j = threadIdx.x;

		//		 for (int k = 0; k < i; k++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

		//									   //TODO correct split points!
		//			 int first = cykArray[k][j];
		//			 int second = cykArray[i - k - 1][j + k + 1];

		//			 //decode nonterminals (find out if bits are on a given positions)
		//			 int base = 1;
		//			 for (int l = 0; l < nonTermsCount; l++) {

		//				 int bitMaskFirst = base << l;
		//				 if (first & bitMaskFirst) {

		//					 // all possibilities connected with rules
		//					 for (int m = 0; m < nonTermsCount; m++) {
		//						 int bitMaskSecond = base << m;

		//						 // if rule with 'l' index and 'm' index is created and ready to be found if corrrect X ->lm (does X exist in a grammar?)
		//						 if (second & bitMaskSecond) {

		//							 //rule exists
		//							 if (rulesNonTermsArray[l][m] != -1) {
		//								 int shiftValue = rulesNonTermsArray[l][m];
		//								 int bitValue = base << shiftValue;

		//								 cykArray[i][j] |= bitValue;
		//							 }

		//						 }

		//					 } // m loop
		//				 }

		//			 } // l loop end

		//		 } // k loop

		//	 }

		//	 __syncthreads();

		//	 //break; //only first line

		//	}
		////}
	 int idx = threadIdx.x;

	 int base = 1;
	 for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)

		 float iterJ = ceilf((float)(inputStringLength - i) / (float)blockDim.x);
		 for (int jd = 0; jd < (int)iterJ; jd++) { // every word <of given length: 5 words, 4 words, 3 words, 2, 1...> (2)
												   ////TODO!!!!
			 int temp_idx = idx + (jd * blockDim.x);

			 if (temp_idx < inputStringLength - i) {

				 for (int k = 0; k < i; k++) {
	
					 //printf("%d - %d \n", temp_idx, temp_idy);

					 int j = temp_idx * cellWidth;

					 for (int m = 0; m < nonTermsCount; m++) {

						 int offset = (int)(m / 32);
						 int first = cykArray[k][j + offset];
						 int bitMaskFirst = (base << (m - (offset * 32)));

						 //if (first & bitMaskFirst) {
						 if (first & bitMaskFirst) {

							 for (int n = 0; n < nonTermsCount; n++) {

								 int offset2 = (int)(n / 32); // shift by 32 is the next cell
								 int second = cykArray[i - k - 1][(((j / cellWidth) + k + 1) * cellWidth) + offset2];
								 int bitMaskSecond = (base << (n - (offset2 * 32)));

								 if (second & bitMaskSecond) {

									 //rule exists
									 if (rulesNonTermsArray[m][n] != -1) {

										 int shiftValue = rulesNonTermsArray[m][n];
										 //int bitValue = base << shiftValue;

										 int offset = (int)(shiftValue / 32); // shift by 32 is the next cell
										 int base = 1;
										 int bitValue = base << (shiftValue - (offset * 32));

										 //cykArray[i][j + offset] |= bitValue;

										 atomicOr(&cykArray[i][j + offset], bitValue);
									 }

								 }

							 } // end n loop
						 }

					 } // end m loop

				 } // end k loop

			 } 

		 } // end j loop

		 __syncthreads();

	 } // end i loop
	}
 else if (action == 5) { //////////////////////////////////////////////////////////////// magisterka rozwiazanie drugie

	 int idx = threadIdx.x;
	 int idy = threadIdx.y;

		 //for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)

			// /*int pidx = i - idy - 1;
			// int pidy = idx + idy + 1;*/

			// //for (int j = 0; j < inputStringLength - i; j++) { // every word of given length 5, 4, 3, 2, 1... (2)

			// float iter = ceilf((float)(inputStringLength - i) / (float)blockDim.x);
			// //iter = 2.0f;

			// for (int r = 0; r < (int)iter; r++) {

			//	 int temp_idx = idx + (r * blockDim.x);

			//	 float iter2 = ceilf((float)(i) / (float)blockDim.y);

			//	 for (int s = 0; s < (int)iter2; s++) {

			//		 int temp_idy = idy + (s * blockDim.y);

			//	 if (temp_idx < inputStringLength - i && temp_idy < i) {
			//		 int j = temp_idx;
			//		 int k = temp_idy;

			//		 //for (int k = 0; k < i; k++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

			//			 //TODO correct split points!
			//			 int first = cykArray[k][j];
			//			 int second = cykArray[i - k - 1][j + k + 1];

			//			 //decode nonterminals (find out if bits are on a given positions)
			//			 int base = 1;
			//			 for (int l = 0; l < nonTermsCount; l++) {

			//				 int bitMaskFirst = base << l;

			//				 // all possibilities connected with rules
			//				 for (int m = 0; m < nonTermsCount; m++) {
			//					 int bitMaskSecond = base << m;

			//					 // if rule with 'l' index and 'm' index is created and ready to be found if corrrect X ->lm (does X exist in a grammar?)
			//					 if (first & bitMaskFirst && second & bitMaskSecond) {

			//						 //rule exists
			//						 if (rulesNonTermsArray[l][m] != -1) {
			//							 int shiftValue = rulesNonTermsArray[l][m];
			//							 int bitValue = base << shiftValue;

			//							 cykArray[i][j] |= bitValue;
			//						 }

			//					 }

			//				 }

			//			 } // l loop end
			//	 

			//		 //} // k loop

			//		 }
			//	 } // end s loop

		 //} // r loop

			// __syncthreads();

			// //break; //only first line

		 //} // l loop
		 
	 //if (idx < inputStringLength && idy < inputStringLength) {
	 if (true) {
	 
	 int base = 1;
	 for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)

		 float iterJ = ceilf((float)(inputStringLength - i) / (float)blockDim.x);
		 for (int jd = 0; jd < (int)iterJ; jd++) { // every word <of given length: 5 words, 4 words, 3 words, 2, 1...> (2)
																////TODO!!!!
			 int temp_idx = idx + (jd * blockDim.x);

			 float iterK = ceilf((float)(i) / (float)blockDim.y);
			 for (int kd = 0; kd < (int)iterK; kd++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

				 int temp_idy = idy + (kd * blockDim.y);

				 if (temp_idx < inputStringLength - i && temp_idy < i) {

					 //printf("%d - %d \n", temp_idx, temp_idy);

					 int j = temp_idx * cellWidth;
					 int k = temp_idy;
					
					 for (int m = 0; m < nonTermsCount; m++) {

						 int offset = (int)(m / 32);
						 int first = cykArray[k][j + offset];
						 int bitMaskFirst = (base << (m - (offset * 32)));

						 //if (first & bitMaskFirst) {
						 if (first & bitMaskFirst) {
							 
							 for (int n = 0; n < nonTermsCount; n++) {

								 int offset2 = (int)(n / 32); // shift by 32 is the next cell
								 int second = cykArray[i - k - 1][(((j / cellWidth) + k + 1) * cellWidth) + offset2];
								 int bitMaskSecond = (base << (n - (offset2 * 32)));

								 if (second & bitMaskSecond) {
									 
									 //rule exists
									 if (rulesNonTermsArray[m][n] != -1) {
										 
										 int shiftValue = rulesNonTermsArray[m][n];
										 //int bitValue = base << shiftValue;

										 int offset = (int)(shiftValue / 32); // shift by 32 is the next cell
										 int base = 1;
										 int bitValue = base << (shiftValue - (offset * 32));

										 //cykArray[i][j + offset] |= bitValue;

										 atomicOr(&cykArray[i][j + offset], bitValue);
									 }

								 }

							 } // end n loop
						 }

					 } // end m loop

				 }

			 } // end k loop

		 } // end j loop

		 __syncthreads();

	 } // end i loop


 }
	 } else if (action == 101) { //////////////////////////////////////////////////////////////// magisterka rozwiazanie drugie

		 int idx = threadIdx.x;
		 int idy = threadIdx.y;

			int base = 1;
			for (int i = 1; i < inputStringLength; i++) {

				float iterJ = ceilf((float)(inputStringLength - i) / (float)blockDim.x);
				for (int jd = 0; jd < (int)iterJ; jd++) { 
														
					int temp_idx = idx + (jd * blockDim.x);

					float iterK = ceilf((float)(i) / (float)blockDim.y);
					for (int kd = 0; kd < (int)iterK; kd++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

						int temp_idy = idy + (kd * blockDim.y);

						if (temp_idx < inputStringLength - i && temp_idy < i) {

							//printf("%d - %d \n", temp_idx, temp_idy);

							int j = temp_idx * cellWidth;
							int k = temp_idy;

							for (int m = 0; m < nonTermsCount; m++) {

								int offset = (int)(m / 32);
								int first = cykArray[k][j + offset];
								int bitMaskFirst = (base << (m - (offset * 32)));

								//if (first & bitMaskFirst) {
								if (first & bitMaskFirst) {

									for (int n = 0; n < nonTermsCount; n++) {

										int offset2 = (int)(n / 32); // shift by 32 is the next cell
										int second = cykArray[i - k - 1][(((j / cellWidth) + k + 1) * cellWidth) + offset2];
										int bitMaskSecond = (base << (n - (offset2 * 32)));

										if (second & bitMaskSecond) {

											//rule exists
											if (rulesNonTermsArray[m][n] != -1) {

												int shiftValue = rulesNonTermsArray[m][n];
												//int bitValue = base << shiftValue;

												int offset = (int)(shiftValue / 32); // shift by 32 is the next cell
												int base = 1;
												int bitValue = base << (shiftValue - (offset * 32));

												//cykArray[i][j + offset] |= bitValue;

												atomicOr(&cykArray[i][j + offset], bitValue);
											}

										}

									} // end n loop
								}

							} // end m loop

						}

					} // end k loop

				} // end j loop

				__syncthreads();

			} // end i loop

	 }

	
	__syncthreads();
	
	if (threadIdx.x == 0 && threadIdx.y == 0) {

		/*for (int i = 0; i < nonTermsCount; i++) {
			for (int j = 0; j < nonTermsCount; j++) {

				printf("%d | ", rulesNonTermsArray[i][j]);
			}

			printf("\n");
		}*/

		/*for (int j = 1; j < inputStringLength; j++) {
			for (int i = 0; i < inputStringLength - j; i++) {
				printf("%d | ", cykArray[j][i]);
			}
			printf("\n");
		}*/

		int* result = data.getResult();
		//printf("RESUUUULt: %d | ", result[0]);
		result[0] = cykArray[inputStringLength - 1][0];
		//printf("JEST? %d \n", cykArray[inputStringLength -1][0]);
	}

	__syncthreads();

	return;
}

/*                                                                                      GLOBAL FUNCTION                                                                     */

__device__ volatile int g_mutex;

template<int action>
__global__ void cykAlgorithmCooperative(DeviceCYKData data, hiprandState * randGlobal, volatile int* arrayIn, volatile int* arrayOut, int additionalVariable) {
	__shared__ int** cykArray;
	__shared__ int inputStringLength;
	__shared__ int** rulesNonTermsArray;
	__shared__ int nonTermsCount;
	__shared__ int cellWidth;

	int bidx = blockIdx.x;
	int bidy = blockIdx.y;
	int idx = threadIdx.x;
	int idy = threadIdx.y;

	if (threadIdx.x == 0 && threadIdx.y == 0) {
		g_mutex = 0;
		cykArray = data.getCYKArray();
		inputStringLength = data.getInputCount();
		rulesNonTermsArray = data.getRulesNonTermsArray();
		nonTermsCount = data.getNonTermsCount();
		//printf("Dim %d\n", gridDim.x);
		cellWidth = (int)(ceilf(((float)nonTermsCount / 32.0f)));
	}


	__syncthreads();

	if (action == 0) { //////////////////////////////////////////////////////////////// entry = block, block used are decremented, threads 2D

		if (threadIdx.x < nonTermsCount && threadIdx.y < nonTermsCount ) {
		
			for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)
				
				//for (int j = 0; j < inputStringLength - i; j++) { // every word of given length 5, 4, 3, 2, 1... (2)

				float iter = ceilf((float)(inputStringLength - i) / (float)gridDim.x);
				//iter = 2.0f;
				//if (iter < 1.0f) iter = 1.0f;
				

				for (int r = 0; r < (int)iter ; r++) {

					int temp_bidx = bidx + (r * gridDim.x);

					
					if (temp_bidx < inputStringLength - i) {
						int j = temp_bidx;

						

						for (int k = 0; k < i; k++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

													  //TODO correct split points!
							//int first = cykArray[k][j];
							//int second = cykArray[i - k - 1][j + k + 1];

							////decode nonterminals (find out if bits are on a given positions)
							//int base = 1;
							////for (int l = 0; l < nonTermsCount; l++) {
							//int l = idx;
							//int bitMaskFirst = base << l;
							////all possibilities connected with rules

							////for (int m = 0; m < nonTermsCount; m++) {

							//int m = idy;

							//int bitMaskSecond = base << m;

							//// if rule with 'l' index and 'm' index is created and ready to be found if corrrect X ->lm (does X exist in a grammar?)
							//if (first & bitMaskFirst && second & bitMaskSecond) {
							//	//cout << bitMaskFirst << ", " << bitMaskSecond << " | ";

							//	//rule exists
							//	if (rulesNonTermsArray[l][m] != -1) {
							//		int shiftValue = rulesNonTermsArray[l][m];
							//		int bitValue = base << shiftValue;

							//		//TODO - tutaj może być problem
							//		atomicOr(&cykArray[i][j], bitValue);

							//		//cykArray[i][j] |= bitValue;
							//	}

							//}
							int base = 1;
							float iterM = ceilf((float)(nonTermsCount) / (float)blockDim.x);
							//for (int m = 0; m < nonTermsCount; m++) {
							for (int md = 0; md < (int)iterM; md++) {

								int temp_idx = idx + (md * blockDim.x);

								int m = temp_idx;

								int offset = (int)(m / 32);
								int first = cykArray[k][j + offset];
								int bitMaskFirst = (base << (m - (offset * 32)));

								if (first & bitMaskFirst) {

									float iterN = ceilf((float)(nonTermsCount) / (float)blockDim.y);
									// for (int n = 0; n < nonTermsCount; n++) {
									for (int nd = 0; nd < (int)iterN; nd++) {

										int temp_idy = idy + (nd * blockDim.y);
										//all possibilities connected with rules

										int n = temp_idy;

										int offset2 = (int)(n / 32); // shift by 32 is the next cell
										int second = cykArray[i - k - 1][(((j / cellWidth) + k + 1) * cellWidth) + offset2];
										int bitMaskSecond = (base << (n - (offset2 * 32)));

										if (second & bitMaskSecond) {

											//rule exists
											if (rulesNonTermsArray[m][n] != -1) {
												int shiftValue = rulesNonTermsArray[m][n];
												//int bitValue = base << shiftValue;

												int offset = (int)(shiftValue / 32); // shift by 32 is the next cell
												int base = 1;
												int bitValue = base << (shiftValue - (offset * 32));

												//cykArray[i][j + offset] |= bitValue;
												atomicOr(&cykArray[i][j + offset], bitValue);
											}

										}

									} // end n loop
								}

							} // end m loop

						}

					}
				}

				//break; //only first line
				if (idx == 0 && idy == 0) {
					//printf("%d | ", g_mutex);
					atomicAdd((int *)&g_mutex, 1);
					//only when all blocks add 1 to g_mutex
					//will g_mutex equal to goalVal
					while (g_mutex != (gridDim.x * i)) {
						//Do nothing here
					}
					
				}
				__syncthreads();

			}
		}

		// only thread 0 is used for synchronization

		//int base = 1;
	 //for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)

		// float iterJ = ceilf((float)(inputStringLength - i) / (float)blockDim.x);
		// for (int jd = 0; jd < (int)iterJ; jd++) { // every word <of given length: 5 words, 4 words, 3 words, 2, 1...> (2)
		//														////TODO!!!!
		//	 int temp_idx = idx + (jd * blockDim.x);

		//	 float iterK = ceilf((float)(i) / (float)blockDim.y);
		//	 for (int kd = 0; kd < (int)iterK; kd++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

		//		 int temp_idy = idy + (kd * blockDim.y);

		//		 if (temp_idx < inputStringLength - i && temp_idy < i) {

		//			 //printf("%d - %d \n", temp_idx, temp_idy);

		//			 int j = temp_idx * cellWidth;
		//			 int k = temp_idy;

		//			 

		//			 for (int m = 0; m < nonTermsCount; m++) {

		//				 int offset = (int)(m / 32);
		//				 int first = cykArray[k][j + offset];
		//				 int bitMaskFirst = (base << (m - (offset * 32)));

		//				 //if (first & bitMaskFirst) {
		//				 if (first & bitMaskFirst) {
		//					 
		//					 for (int n = 0; n < nonTermsCount; n++) {

		//						 int offset2 = (int)(n / 32); // shift by 32 is the next cell
		//						 int second = cykArray[i - k - 1][(((j / cellWidth) + k + 1) * cellWidth) + offset2];
		//						 int bitMaskSecond = (base << (n - (offset2 * 32)));

		//						 if (second & bitMaskSecond) {
		//							 
		//							 //rule exists
		//							 if (rulesNonTermsArray[m][n] != -1) {
		//								 
		//								 int shiftValue = rulesNonTermsArray[m][n];
		//								 //int bitValue = base << shiftValue;

		//								 int offset = (int)(shiftValue / 32); // shift by 32 is the next cell
		//								 int base = 1;
		//								 int bitValue = base << (shiftValue - (offset * 32));

		//								 cykArray[i][j + offset] |= bitValue;
		//							 }

		//						 }

		//					 } // end n loop
		//				 }

		//			 } // end m loop

		//		 }

		//	 } // end k loop

		// } // end j loop

		// __syncthreads();

	 //} // end i loop


	} else if (action == 1) { //////////////////////////////////////////////////////////////// entry = block, block used are decremented, threads 2D + FAST BARIER SYNCHRONISATION

		if (threadIdx.x <= nonTermsCount && threadIdx.y <= nonTermsCount) {


			for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)

				__syncthreads();
														  //for (int j = 0; j < inputStringLength - i; j++) { // every word of given length 5, 4, 3, 2, 1... (2)

				float iter = ceilf((float)(inputStringLength - i) / (float)gridDim.x);
				//iter = 2.0f;

				for (int j = 0; j < (int)iter; j++) {

					int temp_bidx = bidx + (j * gridDim.x);

					if (temp_bidx < inputStringLength - i) {
						int j = temp_bidx;

						for (int k = 0; k < i; k++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

													  //TODO correct split points!
							int first = cykArray[k][j];
							int second = cykArray[i - k - 1][j + k + 1];

							//decode nonterminals (find out if bits are on a given positions)
							int base = 1;
							//for (int l = 0; l < nonTermsCount; l++) {
							int l = idx;
							int bitMaskFirst = base << l;
							//all possibilities connected with rules

							//for (int m = 0; m < nonTermsCount; m++) {

							int m = idy;

							int bitMaskSecond = base << m;

							// if rule with 'l' index and 'm' index is created and ready to be found if corrrect X ->lm (does X exist in a grammar?)
							if (first & bitMaskFirst && second & bitMaskSecond) {
								//cout << bitMaskFirst << ", " << bitMaskSecond << " | ";

								//rule exists
								if (rulesNonTermsArray[l][m] != -1) {
									int shiftValue = rulesNonTermsArray[l][m];
									int bitValue = base << shiftValue;

									//TODO - tutaj może być problem
									atomicOr(&cykArray[i][j], bitValue);

									//cykArray[i][j] |= bitValue;
								}

							}


							// }

							//} // l loop end

							//cout << first << " | " << second << endl;

							//combinations of productions

							// for each production (rulesNonTerminals)

						}

					}
				}

				if (idx == 0 && idy == 0) {
					arrayIn[bidx] = i;
				}

				if (bidx == 1) {

					if (idx < gridDim.x && idy == 0) {
						while (arrayIn[idx] != i) {

						}
					}

					__syncthreads();

					if (idx < gridDim.x && idy == 0) {
						arrayOut[idx] = i;
					}

				}

				//break; //only first line
				if (idx == 0 && idy == 0) {
					while (arrayOut[bidx] != i) {

					}
				}
				__syncthreads();

			}
		}

		// only thread 0 is used for synchronization


	} else if (action == 2) { //////////////////////////////////////////////////////////////// entry = block, block used are decremented, threads 2D + FAST BARIER SYNCHRONISATION

		if (threadIdx.x <= nonTermsCount && threadIdx.y <= nonTermsCount) {

			for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)

														  //for (int j = 0; j < inputStringLength - i; j++) { // every word of given length 5, 4, 3, 2, 1... (2)

				float iter = ceilf((float)(inputStringLength - i) / (float)gridDim.x);
				//iter = 2.0f;

				for (int r = 0; r < (int)iter; r++) {

					int temp_bidx = bidx + (r * gridDim.x);

					if (temp_bidx < inputStringLength - i) {
						int j = temp_bidx;

						int idz = threadIdx.z;

						float iter2 = ceilf((float)(i) / (float)blockDim.z);

						for (int s = 0; s < (int)iter2; s++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

							int temp_idz = idz + (s * blockDim.z);

							if (temp_idz < i) {

								int k = temp_idz;

													  //TODO correct split points!
							int first = cykArray[k][j];
							int second = cykArray[i - k - 1][j + k + 1];

							//decode nonterminals (find out if bits are on a given positions)
							int base = 1;
							//for (int l = 0; l < nonTermsCount; l++) {
							int l = idx;
							int bitMaskFirst = base << l;
							//all possibilities connected with rules

							//for (int m = 0; m < nonTermsCount; m++) {

							int m = idy;

							int bitMaskSecond = base << m;

							// if rule with 'l' index and 'm' index is created and ready to be found if corrrect X ->lm (does X exist in a grammar?)
							if (first & bitMaskFirst && second & bitMaskSecond) {
								//cout << bitMaskFirst << ", " << bitMaskSecond << " | ";

								//rule exists
								if (rulesNonTermsArray[l][m] != -1) {
									int shiftValue = rulesNonTermsArray[l][m];
									int bitValue = base << shiftValue;

									//TODO - tutaj może być problem
									atomicOr(&cykArray[i][j], bitValue);

									//cykArray[i][j] |= bitValue;
								}

							}

						} //end s loop
					}

					}
				}

				//break; //only first line
				if (idx == 0 && idy == 0) {
					//printf("%d | ", g_mutex);
					atomicAdd((int *)&g_mutex, 1);
					//only when all blocks add 1 to g_mutex
					//will g_mutex equal to goalVal
					while (g_mutex != (gridDim.x * i)) {
						//Do nothing here
					}

				}
				__syncthreads();

			}
		}

	 } else if (action == 3) { //////////////////////////////////////////////////////////////// entry = block, block used are decremented, threads 2D + FAST BARIER SYNCHRONISATION

		if (threadIdx.x <= nonTermsCount) {

			for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)

														  //for (int j = 0; j < inputStringLength - i; j++) { // every word of given length 5, 4, 3, 2, 1... (2)

				float iter = ceilf((float)(inputStringLength - i) / (float)gridDim.x);
				//iter = 2.0f;

				for (int r = 0; r < (int)iter; r++) {

					int temp_bidx = bidx + (r * gridDim.x);

					if (temp_bidx < inputStringLength - i) {
						int j = temp_bidx;

						int idy = threadIdx.y;

						float iter2 = ceilf((float)(i) / (float)blockDim.y);

						for (int s = 0; s < (int)iter2; s++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

							int temp_idy = idy + (s * blockDim.y);

							if (temp_idy < i) {

								int k = temp_idy;

								//TODO correct split points!
								int first = cykArray[k][j];
								int second = cykArray[i - k - 1][j + k + 1];

								//decode nonterminals (find out if bits are on a given positions)
								int base = 1;
								//for (int l = 0; l < nonTermsCount; l++) {
								int l = idx;
								int bitMaskFirst = base << l;
								//all possibilities connected with rules

								for (int m = 0; m < nonTermsCount; m++) {
									int bitMaskSecond = base << m;

									// if rule with 'l' index and 'm' index is created and ready to be found if corrrect X ->lm (does X exist in a grammar?)
									if (first & bitMaskFirst && second & bitMaskSecond) {
										//cout << bitMaskFirst << ", " << bitMaskSecond << " | ";

										//rule exists
										if (rulesNonTermsArray[l][m] != -1) {
											int shiftValue = rulesNonTermsArray[l][m];
											int bitValue = base << shiftValue;

											//TODO - tutaj może być problem
											atomicOr(&cykArray[i][j], bitValue);

											//cykArray[i][j] |= bitValue;
										}

									}


								}

							} //end s loop
						}

					}
				}

				//break; //only first line
				if (idx == 0 && idy == 0) {
					//printf("%d | ", g_mutex);
					atomicAdd((int *)&g_mutex, 1);
					//only when all blocks add 1 to g_mutex
					//will g_mutex equal to goalVal
					while (g_mutex != (gridDim.x * i)) {
						//Do nothing here
					}

				}
				__syncthreads();

			}
		}

	}
	 else if (action == 4) { //////////////////////////////////////////////////////////////// entry = block, block used are decremented, threads 2D + FAST BARIER SYNCHRONISATION

		 if (threadIdx.x < nonTermsCount) {

			 for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)

														   //for (int j = 0; j < inputStringLength - i; j++) { // every word of given length 5, 4, 3, 2, 1... (2)

				 float iter = ceilf((float)(inputStringLength - i) / (float)gridDim.x);
				 //iter = 2.0f;

				 for (int r = 0; r < (int)iter; r++) {

					 int temp_bidx = bidx + (r * gridDim.x);

					 if (temp_bidx < inputStringLength - i) {
						 int j = temp_bidx * cellWidth;

						 int idy = threadIdx.y;

						 float iter2 = ceilf((float)(i) / (float)blockDim.y);

						 for (int s = 0; s < (int)iter2; s++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

							 int temp_idy = idy + (s * blockDim.y);

							 if (temp_idy < i) {

								 int k = temp_idy;

								 //TODO correct split points!
								 int first = cykArray[k][j];
								 int second = cykArray[i - k - 1][j + k + 1];

								 //decode nonterminals (find out if bits are on a given positions)
								 int base = 1;
								 //for (int l = 0; l < nonTermsCount; l++) {
								 //int l = idx;
								 //int bitMaskFirst = base << l;
								 ////all possibilities connected with rules

								 //for (int m = 0; m < nonTermsCount; m++) {
									// int bitMaskSecond = base << m;

									// // if rule with 'l' index and 'm' index is created and ready to be found if corrrect X ->lm (does X exist in a grammar?)
									// if (first & bitMaskFirst && second & bitMaskSecond) {
									//	 //cout << bitMaskFirst << ", " << bitMaskSecond << " | ";

									//	 //rule exists
									//	 if (rulesNonTermsArray[l][m] != -1) {
									//		 int shiftValue = rulesNonTermsArray[l][m];
									//		 int bitValue = base << shiftValue;

									//		 //TODO - tutaj może być problem
									//		 atomicOr(&cykArray[i][j], bitValue);

									//		 //cykArray[i][j] |= bitValue;
									//	 }

									// }


								 //}
								 

								 float iterM = ceilf((float)(nonTermsCount) / (float)blockDim.x);
								 for (int md = 0; md < (int)iterM; md++) {

									 int temp_idx = idy + (md * blockDim.x); 

									 if (temp_idx < nonTermsCount) {

										 int m = temp_idx;

										 int offset = (int)(m / 32);
										 int first = cykArray[k][j + offset];
										 int bitMaskFirst = (base << (m - (offset * 32)));

										 //if (first & bitMaskFirst) {
										 if (first & bitMaskFirst) {

											 for (int n = 0; n < nonTermsCount; n++) {

												 int offset2 = (int)(n / 32); // shift by 32 is the next cell
												 int second = cykArray[i - k - 1][(((j / cellWidth) + k + 1) * cellWidth) + offset2];
												 int bitMaskSecond = (base << (n - (offset2 * 32)));

												 if (second & bitMaskSecond) {

													 //rule exists
													 if (rulesNonTermsArray[m][n] != -1) {

														 int shiftValue = rulesNonTermsArray[m][n];
														 //int bitValue = base << shiftValue;

														 int offset = (int)(shiftValue / 32); // shift by 32 is the next cell
														 int base = 1;
														 int bitValue = base << (shiftValue - (offset * 32));

														 cykArray[i][j + offset] |= bitValue;
													 }

												 }

											 } // end n loop
										 }
									 }

									 

								 } // end m loop

							 } //end s loop
						 }

					 }
				 }

				 //break; //only first line
				 if (idx == 0 && idy == 0) {
					 //printf("%d | ", g_mutex);
					 atomicAdd((int *)&g_mutex, 1);
					 //only when all blocks add 1 to g_mutex
					 //will g_mutex equal to goalVal
					 while (g_mutex != (gridDim.x * i)) {
						 //Do nothing here
					 }

				 }
				 __syncthreads();

			 }
		 }

		 //start
		 int base = 1;
		 for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)

			 float iterJ = ceilf((float)(inputStringLength - i) / (float)blockDim.x);
			 for (int jd = 0; jd < (int)iterJ; jd++) { // every word <of given length: 5 words, 4 words, 3 words, 2, 1...> (2)
													   ////TODO!!!!
				 int temp_idx = idx + (jd * blockDim.x);

				 float iterK = ceilf((float)(i) / (float)blockDim.y);
				 for (int kd = 0; kd < (int)iterK; kd++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

					 int temp_idy = idy + (kd * blockDim.y);

					 if (temp_idx < inputStringLength - i && temp_idy < i) {

						 //printf("%d - %d \n", temp_idx, temp_idy);

						 int j = temp_idx * cellWidth;
						 int k = temp_idy;


						 for (int m = 0; m < nonTermsCount; m++) {

							 int offset = (int)(m / 32);
							 int first = cykArray[k][j + offset];
							 int bitMaskFirst = (base << (m - (offset * 32)));

							 //if (first & bitMaskFirst) {
							 if (first & bitMaskFirst) {

								 for (int n = 0; n < nonTermsCount; n++) {

									 int offset2 = (int)(n / 32); // shift by 32 is the next cell
									 int second = cykArray[i - k - 1][(((j / cellWidth) + k + 1) * cellWidth) + offset2];
									 int bitMaskSecond = (base << (n - (offset2 * 32)));

									 if (second & bitMaskSecond) {

										 //rule exists
										 if (rulesNonTermsArray[m][n] != -1) {

											 int shiftValue = rulesNonTermsArray[m][n];
											 //int bitValue = base << shiftValue;

											 int offset = (int)(shiftValue / 32); // shift by 32 is the next cell
											 int base = 1;
											 int bitValue = base << (shiftValue - (offset * 32));

											 cykArray[i][j + offset] |= bitValue;
										 }

									 }

								 } // end n loop
							 }

						 } // end m loop

					 }

				 } // end k loop

			 } // end j loop

			 __syncthreads();

		 } // end i loop
		 //end

	 } else if (action == 5) { //////////////////////////////////////////////////////////////// entry = block, block used are decremented, threads 2D

				 int base = 1;

				 int i = additionalVariable;

					 float iterJ = ceilf((float)(inputStringLength - i) / (float)blockDim.x);
					 for (int jd = 0; jd < (int)iterJ; jd++) { // every word <of given length: 5 words, 4 words, 3 words, 2, 1...> (2)
															   ////TODO!!!!
						 int temp_idx = idx + (jd * blockDim.x);

						 float iterK = ceilf((float)(i) / (float)blockDim.y);
						 for (int kd = 0; kd < (int)iterK; kd++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

							 int temp_idy = idy + (kd * blockDim.y);

							 if (temp_idx < inputStringLength - i && temp_idy < i) {

								 //printf("%d - %d \n", temp_idx, temp_idy);

								 int j = temp_idx * cellWidth;
								 int k = temp_idy;

								 for (int m = 0; m < nonTermsCount; m++) {

									 int offset = (int)(m / 32);
									 int first = cykArray[k][j + offset];
									 int bitMaskFirst = (base << (m - (offset * 32)));

									 //if (first & bitMaskFirst) {
									 if (first & bitMaskFirst) {

										 for (int n = 0; n < nonTermsCount; n++) {

											 int offset2 = (int)(n / 32); // shift by 32 is the next cell
											 int second = cykArray[i - k - 1][(((j / cellWidth) + k + 1) * cellWidth) + offset2];
											 int bitMaskSecond = (base << (n - (offset2 * 32)));

											 if (second & bitMaskSecond) {

												 //rule exists
												 if (rulesNonTermsArray[m][n] != -1) {

													 int shiftValue = rulesNonTermsArray[m][n];
													 //int bitValue = base << shiftValue;

													 int offset = (int)(shiftValue / 32); // shift by 32 is the next cell
													 int base = 1;
													 int bitValue = base << (shiftValue - (offset * 32));

													 //cykArray[i][j + offset] |= bitValue;

													 atomicOr(&cykArray[i][j + offset], bitValue);
												 }

											 }

										 } // end n loop
									 }

								 } // end m loop

							 }

						 } // end k loop

					 } // end j loop

	 }
	 else if (action == 6) { //////////////////////////////////////////////////////////////// entry = block, block used are decremented, threads 2D

		 int base = 1;

		 int i = additionalVariable;

		 float iterJ = ceilf((float)(inputStringLength - i) / (float)gridDim.x);
		 for (int jd = 0; jd < (int)iterJ; jd++) { // every word <of given length: 5 words, 4 words, 3 words, 2, 1...> (2)
												   ////TODO!!!!
			 int temp_bidx = bidx + (jd * gridDim.x);

			 float iterK = ceilf((float)(i) / (float)blockDim.x);
			 for (int kd = 0; kd < (int)iterK; kd++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

				 int temp_idx = idx + (kd * blockDim.x);

				 if (temp_bidx < inputStringLength - i && temp_idx < i) {

					 //printf("%d - %d \n", temp_idx, temp_idy);

					 int j = temp_bidx * cellWidth;
					 int k = temp_idx;

					 for (int m = 0; m < nonTermsCount; m++) {

						 int offset = (int)(m / 32);
						 int first = cykArray[k][j + offset];
						 int bitMaskFirst = (base << (m - (offset * 32)));

						 //if (first & bitMaskFirst) {
						 if (first & bitMaskFirst) {

							 for (int n = 0; n < nonTermsCount; n++) {

								 int offset2 = (int)(n / 32); // shift by 32 is the next cell
								 int second = cykArray[i - k - 1][(((j / cellWidth) + k + 1) * cellWidth) + offset2];
								 int bitMaskSecond = (base << (n - (offset2 * 32)));

								 if (second & bitMaskSecond) {

									 //rule exists
									 if (rulesNonTermsArray[m][n] != -1) {

										 int shiftValue = rulesNonTermsArray[m][n];
										 //int bitValue = base << shiftValue;

										 int offset = (int)(shiftValue / 32); // shift by 32 is the next cell
										 int base = 1;
										 int bitValue = base << (shiftValue - (offset * 32));

										 //cykArray[i][j + offset] |= bitValue;

										 atomicOr(&cykArray[i][j + offset], bitValue);
									 }

								 }

							 } // end n loop
						 }

					 } // end m loop

				 }

			 } // end k loop

		 } // end j loop

	 }
	 else if (action == 7) { //////////////////////////////////////////////////////////////// entry = block, block used are decremented, threads 2D

		 int base = 1;

		 int i = additionalVariable;

		 float iterJ = ceilf((float)(inputStringLength - i) / (float)gridDim.x);
		 for (int jd = 0; jd < (int)iterJ; jd++) { // every word <of given length: 5 words, 4 words, 3 words, 2, 1...> (2)
												   ////TODO!!!!
			 int temp_bidx = bidx + (jd * gridDim.x);

			 float iterK = ceilf((float)(i) / (float)blockDim.x);
			 for (int kd = 0; kd < (int)iterK; kd++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

				 int temp_idx = idx + (kd * blockDim.x);

				 if (temp_bidx < inputStringLength - i && temp_idx < i) {

					 //printf("%d - %d \n", temp_idx, temp_idy);

					 int j = temp_bidx * cellWidth;
					 int k = temp_idx;


					 float iterM = ceilf((float)(nonTermsCount) / (float)blockDim.y);
					 for (int km = 0; km < (int)iterM; km++) {

						 int temp_idy = idy + (km * blockDim.y);

						 if (temp_idy < nonTermsCount) {

							 int m = temp_idy;


							 int offset = (int)(m / 32);
							 int first = cykArray[k][j + offset];
							 int bitMaskFirst = (base << (m - (offset * 32)));

							 //if (first & bitMaskFirst) {
							 if (first & bitMaskFirst) {

								 for (int n = 0; n < nonTermsCount; n++) {

									 int offset2 = (int)(n / 32); // shift by 32 is the next cell
									 int second = cykArray[i - k - 1][(((j / cellWidth) + k + 1) * cellWidth) + offset2];
									 int bitMaskSecond = (base << (n - (offset2 * 32)));

									 if (second & bitMaskSecond) {

										 //rule exists
										 if (rulesNonTermsArray[m][n] != -1) {

											 int shiftValue = rulesNonTermsArray[m][n];
											 //int bitValue = base << shiftValue;

											 int offset = (int)(shiftValue / 32); // shift by 32 is the next cell
											 int base = 1;
											 int bitValue = base << (shiftValue - (offset * 32));

											 //cykArray[i][j + offset] |= bitValue;

											 atomicOr(&cykArray[i][j + offset], bitValue);
										 }

									 }

								 } // end n loop
							 }

						 } // end m loop
					 }

				 }

			 } // end k loop

		 } // end j loop

	 }
	 else if (action == 8) { //////////////////////////////////////////////////////////////// entry = block, block used are decremented, threads 2D

		 int base = 1;

		 int i = additionalVariable;

		 float iterJ = ceilf((float)(inputStringLength - i) / (float)gridDim.x);
		 for (int jd = 0; jd < (int)iterJ; jd++) { // every word <of given length: 5 words, 4 words, 3 words, 2, 1...> (2)
												   ////TODO!!!!
			 int temp_bidx = bidx + (jd * gridDim.x);

			 float iterK = ceilf((float)(i) / (float)gridDim.y);
			 for (int kd = 0; kd < (int)iterK; kd++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

				 int temp_bidy = bidy + (kd * gridDim.y);

				 if (temp_bidx < inputStringLength - i && temp_bidy < i) {

					 //printf("%d - %d \n", temp_idx, temp_idy);

					 int j = temp_bidx * cellWidth;
					 int k = temp_bidy;


					 float iterM = ceilf((float)(nonTermsCount) / (float)blockDim.x);
					 for (int km = 0; km < (int)iterM; km++) {

						 int temp_idx = idx + (km * blockDim.x);

						 if (temp_idx < nonTermsCount) {

							 int m = temp_idx;


							 int offset = (int)(m / 32);
							 int first = cykArray[k][j + offset];
							 int bitMaskFirst = (base << (m - (offset * 32)));

							 //if (first & bitMaskFirst) {
							 if (first & bitMaskFirst) {

								 for (int n = 0; n < nonTermsCount; n++) {

									 int offset2 = (int)(n / 32); // shift by 32 is the next cell
									 int second = cykArray[i - k - 1][(((j / cellWidth) + k + 1) * cellWidth) + offset2];
									 int bitMaskSecond = (base << (n - (offset2 * 32)));

									 if (second & bitMaskSecond) {

										 //rule exists
										 if (rulesNonTermsArray[m][n] != -1) {

											 int shiftValue = rulesNonTermsArray[m][n];
											 //int bitValue = base << shiftValue;

											 int offset = (int)(shiftValue / 32); // shift by 32 is the next cell
											 int base = 1;
											 int bitValue = base << (shiftValue - (offset * 32));

											 //cykArray[i][j + offset] |= bitValue;

											 atomicOr(&cykArray[i][j + offset], bitValue);
										 }

									 }

								 } // end n loop
							 }

						 } // end m loop
					 }

				 }

			 } // end k loop

		 } // end j loop

	 }
	 else if (action == 9) { //////////////////////////////////////////////////////////////// entry = block, block used are decremented, threads 2D

		 int base = 1;

		 int i = additionalVariable;

		 float iterJ = ceilf((float)(inputStringLength - i) / (float)gridDim.x);
		 for (int jd = 0; jd < (int)iterJ; jd++) { // every word <of given length: 5 words, 4 words, 3 words, 2, 1...> (2)
												   ////TODO!!!!
			 int temp_bidx = bidx + (jd * gridDim.x);

			 float iterK = ceilf((float)(i) / (float)gridDim.y);
			 for (int kd = 0; kd < (int)iterK; kd++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

				 int temp_bidy = bidy + (kd * gridDim.y);

				 if (temp_bidx < inputStringLength - i && temp_bidy < i) {

					 //printf("%d - %d \n", temp_idx, temp_idy);

					 int j = temp_bidx * cellWidth;
					 int k = temp_bidy;


					 float iterM = ceilf((float)(nonTermsCount) / (float)blockDim.x);
					 for (int km = 0; km < (int)iterM; km++) {

						 int temp_idx = idx + (km * blockDim.x);

						 if (temp_idx < nonTermsCount) {

							 int m = temp_idx;


							 int offset = (int)(m / 32);
							 int first = cykArray[k][j + offset];
							 int bitMaskFirst = (base << (m - (offset * 32)));

							 //if (first & bitMaskFirst) {
							 if (first & bitMaskFirst) {


								 float iterN = ceilf((float)(nonTermsCount) / (float)blockDim.y);
								 for (int nd = 0; nd < nonTermsCount; nd++) {

									 int temp_idy = idy + (nd * blockDim.y);

									 if (temp_idy < nonTermsCount) {

										 int n = temp_idy;

										 int offset2 = (int)(n / 32); // shift by 32 is the next cell
										 int second = cykArray[i - k - 1][(((j / cellWidth) + k + 1) * cellWidth) + offset2];
										 int bitMaskSecond = (base << (n - (offset2 * 32)));

										 if (second & bitMaskSecond) {

											 //rule exists
											 if (rulesNonTermsArray[m][n] != -1) {

												 int shiftValue = rulesNonTermsArray[m][n];
												 //int bitValue = base << shiftValue;

												 int offset = (int)(shiftValue / 32); // shift by 32 is the next cell
												 int base = 1;
												 int bitValue = base << (shiftValue - (offset * 32));

												 //cykArray[i][j + offset] |= bitValue;

												 atomicOr(&cykArray[i][j + offset], bitValue);
											 }

										 }

									 } // end n loop
								 }
							 }

						 } // end m loop
					 }

				 }

			 } // end k loop

		 } // end j loop

	 }

	

	__syncthreads();

	if (threadIdx.x == 0 && threadIdx.y == 0 && bidx == 0) {
		
		//for (int i = 0; i < nonTermsCount; i++) {
		//	for (int j = 0; j < nonTermsCount; j++) {
		//		//cout << rulesNonTermsArray[i][j] << " | ";

		//		printf("%d | ", rulesNonTermsArray[i][j]);
		//	}
		//	//cout << endl;

		//	printf("\n");
		//}

		//for (int j = 1; j < inputStringLength; j++) {
		//	for (int i = 0; i < inputStringLength - j; i++) {
		//		printf("%d | ", cykArray[j][i]);
		//	}
		//	printf("\n");
		//}

		//int* result = data.getResult();
		//printf("RESUUUULt: %d | ", result[0]);
		//result[0] = 1337;
		int* result = data.getResult();
		//printf("RESUUUULt: %d | ", result[0]);
		result[0] = cykArray[inputStringLength - 1][0];
	}

	__syncthreads();

	return;
}



template<int action>
__global__ void cykAlgorithmRules(DeviceCYKData data, hiprandState * randGlobal, volatile int * arrayIn, volatile int * arrayOut, int** rulesArray, int rulesCount, int additionalVariable)
{
	__shared__ int** cykArray;
	__shared__ int inputStringLength;
	__shared__ int** rulesNonTermsArray;
	__shared__ int nonTermsCount;
	__shared__ int cellWidth;

	int bidx = blockIdx.x;
	int bidy = blockIdx.y;
	int bidz = blockIdx.z;
	int idx = threadIdx.x;
	int idy = threadIdx.y;
	int idz = threadIdx.z;

	if (threadIdx.x == 0 && threadIdx.y == 0) {
		g_mutex = 0;
		cykArray = data.getCYKArray();
		inputStringLength = data.getInputCount();
		rulesNonTermsArray = data.getRulesNonTermsArray();
		nonTermsCount = data.getNonTermsCount();
		//printf("Dim %d\n", gridDim.x);
		cellWidth = (int)(ceilf(((float)nonTermsCount / 32.0f)));
	}


	__syncthreads();

	if (action == 0) { //////////////////////////////////////////////////////////////// only threads

		if (threadIdx.x <= nonTermsCount && threadIdx.y <= nonTermsCount) {

			for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)

														
				float iter = ceilf((float)(inputStringLength - i) / (float)gridDim.x);
				//iter = 2.0f;

				for (int r = 0; r < (int)iter; r++) {

					int temp_bidx = bidx + (r * gridDim.x);

					if (temp_bidx < inputStringLength - i) {
						int j = temp_bidx; //J

						//for (int p = 0; p < rulesCount; p++) { //for each production (each rule)
						if (idx < rulesCount) {

							int p = idx;

							for (int k = 0; k < i; k++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

														  //TODO correct split points!
								int first = cykArray[k][j];
								int second = cykArray[i - k - 1][j + k + 1];

								//decode nonterminals (find out if bits are on a given positions)
								int base = 1;
								int bitMaskFirst = base << rulesArray[0][p];
								int bitMaskSecond = base << rulesArray[1][p];
								if (first & bitMaskFirst && second & bitMaskSecond) {

									int shiftValue = rulesArray[2][p];
									int bitValue = base << shiftValue;
									//TODO - tutaj może być problem
									atomicOr(&cykArray[i][j], bitValue);
								}

							}
						}

					}
				}

				//break; //only first line
				if (idx == 0 && idy == 0) {
					//printf("%d | ", g_mutex);
					atomicAdd((int *)&g_mutex, 1);
					//only when all blocks add 1 to g_mutex
					//will g_mutex equal to goalVal
					while (g_mutex != (gridDim.x * i)) {
						//Do nothing here
					}

				}
				__syncthreads();

			}
		}

		// only thread 0 is used for synchronization


	} if (action == 2) { //////////////////////////////////////////////////////////////// only threads


			int i = additionalVariable;

			//float iter = ceilf((float)(inputStringLength - i) / (float)gridDim.x);
			////iter = 2.0f;

			////for (int r = 0; r < (int)iter; r++) {


			//int temp_bidx = bidx;// +(r * gridDim.x);

			//if (temp_bidx < inputStringLength - i) {
			//	int j = temp_bidx; //J
			float iterJ = ceilf((float)(inputStringLength - i) / (float)gridDim.x);

			for (int jd = 0; jd < (int)iterJ; jd++) {
				
				int temp_bidx = bidx + (jd * gridDim.x);

				if (temp_bidx < inputStringLength - i) {

					int j = temp_bidx * cellWidth;

					float iterP = ceilf((float)(rulesCount) / (float)blockDim.x);

					for (int pd = 0; pd < (int)iterP; pd++) {

						int temp_idx = idx + (pd * blockDim.x);
						
						if (temp_idx < rulesCount) { //TODO wtf?

							int p = temp_idx;

							for (int k = 0; k < i; k++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

														  //TODO correct split points!
								//int first = cykArray[k][j];
								//int second = cykArray[i - k - 1][j + k + 1];

								////decode nonterminals (find out if bits are on a given positions)
								//int base = 1;
								int rule1 = rulesArray[0][p];
								int rule2 = rulesArray[1][p];

								int offset = (int)(rule1 / 32);
								int first = cykArray[k][j + offset];

								int offset2 = (int)(rule2 / 32);
								int second = cykArray[i - k - 1][(((j / cellWidth) + k + 1) * cellWidth) + offset2];

								int base = 1;

								/*int bitMaskFirst = base << rulesArray[0][p];
								int bitMaskSecond = base << rulesArray[1][p];*/
								

								int bitMaskFirst = (base << (rule1 - (offset * 32)));
								int bitMaskSecond = (base << (rule2 - (offset2 * 32)));


								if (first & bitMaskFirst && second & bitMaskSecond) {

									//int shiftValue = rulesArray[2][p];
									//int bitValue = base << shiftValue;
									////TODO - tutaj może być problem
									//atomicOr(&cykArray[i][j], bitValue);

									int shiftValue = rulesArray[2][p];
									int offset = (int)(shiftValue / 32);
									int bitValue = base << (shiftValue - (offset * 32));

									atomicOr(&cykArray[i][j + offset], bitValue);
								}

							}
						}
					}

				}
			}

			__syncthreads();


	} else if (action == 1) { //////////////////////////////////////////////////////////////// blocks + threads

		int i = additionalVariable;

		float iterJ = ceilf((float)(inputStringLength - i) / (float)gridDim.x);

		for (int jd = 0; jd < (int)iterJ; jd++) {

			int temp_bidx = bidx + (jd * gridDim.x);

			if (temp_bidx < inputStringLength - i) {

				int j = temp_bidx * cellWidth;

				float iterP = ceilf((float)(rulesCount) / (float)blockDim.x);

				for (int pd = 0; pd < (int)iterP; pd++) {

					int temp_idx = idx + (pd * blockDim.x);

					if (temp_idx < rulesCount) { //TODO wtf?

						int p = temp_idx;

						float iterk = ceilf((float)(i) / (float)gridDim.y);

						for (int kd = 0; kd < (int)iterk; kd++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

							int temp_bidy = bidy + (kd * gridDim.y);

							if (temp_bidy < i) {

								int k = temp_bidy;

								int rule1 = rulesArray[0][p];
								int rule2 = rulesArray[1][p];

								int offset = (int)(rule1 / 32);
								int first = cykArray[k][j + offset];

								int offset2 = (int)(rule2 / 32);
								int second = cykArray[i - k - 1][(((j / cellWidth) + k + 1) * cellWidth) + offset2];

								int base = 1;

								int bitMaskFirst = (base << (rule1 - (offset * 32)));
								int bitMaskSecond = (base << (rule2 - (offset2 * 32)));


								if (first & bitMaskFirst && second & bitMaskSecond) {

									int shiftValue = rulesArray[2][p];
									int offset = (int)(shiftValue / 32);
									int bitValue = base << (shiftValue - (offset * 32));

									atomicOr(&cykArray[i][j + offset], bitValue);
								}
							}
						}
					}
				}

			}
		}

		__syncthreads();

	} else if (action == 3) { //////////////////////////////////////////////////////////////// blocks + threads

							// bidx - i loop, bidy - each left symbol, idx - left symbol connected rules
							// block dim y - number of unique left symbols

		int numberOfProductions = rulesArray[bidy][1];

		int i =  additionalVariable;

			float iterJ = ceilf((float)(inputStringLength - i) / (float)gridDim.x);

			for (int jd = 0; jd < (int)iterJ; jd++) {

				int temp_bidx = bidx + (jd * gridDim.x);

				if (temp_bidx < inputStringLength - i) {

					int j = temp_bidx * cellWidth;

					//for (int p = 0; p < rulesCount; p++) { //for each production (each rule)

					float iterP = ceilf((float)(numberOfProductions) / (float)(blockDim.x));

						for (int pd = 0; pd < (int)iterP; pd++) {

							int temp_idx = idx + (pd * blockDim.x);

							if (temp_idx < numberOfProductions) {

								int p = temp_idx;

								for (int k = 0; k < i; k++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

									int rule1 = rulesArray[blockIdx.y][(p + 1) * 2];
									int rule2 = rulesArray[blockIdx.y][(p + 1) * 2 + 1];

									int offset = (int)(rule1 / 32);
									int first = cykArray[k][j + offset];

									int offset2 = (int)(rule2 / 32);
									int second = cykArray[i - k - 1][(((j / cellWidth) + k + 1) * cellWidth) + offset2];

									//decode nonterminals (find out if bits are on a given positions)
									int base = 1;
									int bitMaskFirst = (base << (rule1 - (offset * 32)));
									int bitMaskSecond = (base << (rule2 - (offset2 * 32)));

									if (first & bitMaskFirst && second & bitMaskSecond) {

										int shiftValue = rulesArray[blockIdx.y][0];
										int offset = (int)(shiftValue / 32);
										int bitValue = base << (shiftValue - (offset * 32));

										atomicOr(&cykArray[i][j + offset], bitValue);

									}

								}
							} //pd if

						} // pd loop

				} //jd if 

			} // jd loop

			__syncthreads();


	} else if (action == 4) { //////////////////////////////////////////////////////////////// blocks + threads

							// bidx - i loop, bidy - each left symbol, idx - left symbol connected rules
							// block dim y - number of unique left symbols

		int numberOfProductions = rulesArray[bidy][1];

		int i = additionalVariable;

		float iterJ = ceilf((float)(inputStringLength - i) / (float)gridDim.x);

		for (int jd = 0; jd < (int)iterJ; jd++) {

			int temp_bidx = bidx + (jd * gridDim.x);

			if (temp_bidx < inputStringLength - i) {

				int j = temp_bidx * cellWidth;

				//for (int p = 0; p < rulesCount; p++) { //for each production (each rule)

				float iterP = ceilf((float)(numberOfProductions) / (float)(blockDim.x));

				for (int pd = 0; pd < (int)iterP; pd++) {

					int temp_idx = idx + (pd * blockDim.x);

					if (temp_idx < numberOfProductions) {

						int p = temp_idx;

						float iterK = ceilf((float)(i) / (float)(blockDim.y));

						//for (int k = 0; k < i; k++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)
						for (int kd = 0; kd < (int)iterK; kd++) {

							int temp_idy = idy + (kd * blockDim.y);
							
							if (temp_idy < i) {

								int k = temp_idy;

								int rule1 = rulesArray[blockIdx.y][(p + 1) * 2];
								int rule2 = rulesArray[blockIdx.y][(p + 1) * 2 + 1];

								int offset = (int)(rule1 / 32);
								int first = cykArray[k][j + offset];

								int offset2 = (int)(rule2 / 32);
								int second = cykArray[i - k - 1][(((j / cellWidth) + k + 1) * cellWidth) + offset2];

								//decode nonterminals (find out if bits are on a given positions)
								int base = 1;
								int bitMaskFirst = (base << (rule1 - (offset * 32)));
								int bitMaskSecond = (base << (rule2 - (offset2 * 32)));

								if (first & bitMaskFirst && second & bitMaskSecond) {

									int shiftValue = rulesArray[blockIdx.y][0];
									int offset = (int)(shiftValue / 32);
									int bitValue = base << (shiftValue - (offset * 32));

									atomicOr(&cykArray[i][j + offset], bitValue);

								}
							} //kd if

						} // kd loop

					} //pd if

				} // pd loop

			} //jd if 

		} // jd loop

		__syncthreads();


	} else if (action == 5) { //////////////////////////////////////////////////////////////// blocks + threads

							// bidx - i loop, bidy - each left symbol, idx - left symbol connected rules
							// block dim y - number of unique left symbols

		int numberOfProductions = rulesArray[bidy][1];

		int i = additionalVariable;

		float iterJ = ceilf((float)(inputStringLength - i) / (float)gridDim.x);

		for (int jd = 0; jd < (int)iterJ; jd++) {

			int temp_bidx = bidx + (jd * gridDim.x);

			if (temp_bidx < inputStringLength - i) {

				int j = temp_bidx * cellWidth;

				//for (int p = 0; p < rulesCount; p++) { //for each production (each rule)

				float iterP = ceilf((float)(numberOfProductions) / (float)(blockDim.x));

				for (int pd = 0; pd < (int)iterP; pd++) {

					int temp_idx = idx + (pd * blockDim.x);

					if (temp_idx < numberOfProductions) {

						int p = temp_idx;

						float iterK = ceilf((float)(i) / (float)(gridDim.z));

						//for (int k = 0; k < i; k++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)
						for (int kd = 0; kd < (int)iterK; kd++) {

							int temp_idy = bidz + (kd * gridDim.z);

							if (temp_idy < i) {

								int k = temp_idy;

								int rule1 = rulesArray[blockIdx.y][(p + 1) * 2];
								int rule2 = rulesArray[blockIdx.y][(p + 1) * 2 + 1];

								int offset = (int)(rule1 / 32);
								int first = cykArray[k][j + offset];

								int offset2 = (int)(rule2 / 32);
								int second = cykArray[i - k - 1][(((j / cellWidth) + k + 1) * cellWidth) + offset2];

								//decode nonterminals (find out if bits are on a given positions)
								int base = 1;
								int bitMaskFirst = (base << (rule1 - (offset * 32)));
								int bitMaskSecond = (base << (rule2 - (offset2 * 32)));

								if (first & bitMaskFirst && second & bitMaskSecond) {

									int shiftValue = rulesArray[blockIdx.y][0];
									int offset = (int)(shiftValue / 32);
									int bitValue = base << (shiftValue - (offset * 32));

									atomicOr(&cykArray[i][j + offset], bitValue);

								}
							} //kd if

						} // kd loop

					} //pd if

				} // pd loop

			} //jd if 

		} // jd loop

		__syncthreads();


	}
	else if (action == 6) { //////////////////////////////////////////////////////////////// blocks + threads

		int i = additionalVariable;

		float iterJ = ceilf((float)(inputStringLength - i) / (float)gridDim.x);

		for (int jd = 0; jd < (int)iterJ; jd++) {

			int temp_bidx = bidx + (jd * gridDim.x);

			if (temp_bidx < inputStringLength - i) {

				int j = temp_bidx * cellWidth;

				float iterP = ceilf((float)(rulesCount) / (float)blockDim.x);

				for (int pd = 0; pd < (int)iterP; pd++) {

					int temp_idx = idx + (pd * blockDim.x);

					if (temp_idx < rulesCount) { //TODO wtf?

						int p = temp_idx;

						float iterk = ceilf((float)(i) / (float)blockDim.y);

						for (int kd = 0; kd < (int)iterk; kd++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

							int temp_idy = idy + (kd * blockDim.y);

							if (temp_idy < i) {

								int k = temp_idy;

								int rule1 = rulesArray[0][p];
								int rule2 = rulesArray[1][p];

								int offset = (int)(rule1 / 32);
								int first = cykArray[k][j + offset];

								int offset2 = (int)(rule2 / 32);
								int second = cykArray[i - k - 1][(((j / cellWidth) + k + 1) * cellWidth) + offset2];

								int base = 1;

								int bitMaskFirst = (base << (rule1 - (offset * 32)));
								int bitMaskSecond = (base << (rule2 - (offset2 * 32)));


								if (first & bitMaskFirst && second & bitMaskSecond) {

									int shiftValue = rulesArray[2][p];
									int offset = (int)(shiftValue / 32);
									int bitValue = base << (shiftValue - (offset * 32));

									atomicOr(&cykArray[i][j + offset], bitValue);
								}
							}
						}
					}
				}

			}
		}

		__syncthreads();

	}
	else if (action == 7) { //////////////////////////////////////////////////////////////// blocks + threads

		int i = additionalVariable;

		float iterJ = ceilf((float)(inputStringLength - i) / (float)gridDim.x);

		for (int jd = 0; jd < (int)iterJ; jd++) {

			int temp_bidx = bidx + (jd * gridDim.x);

			if (temp_bidx < inputStringLength - i) {

				int j = temp_bidx * cellWidth;

				float iterP = ceilf((float)(rulesCount) / (float)gridDim.y);

				for (int pd = 0; pd < (int)iterP; pd++) {

					int temp_bidy = bidy + (pd * gridDim.y);

					if (temp_bidy < rulesCount) { //TODO wtf?

						int p = temp_bidy;

						float iterk = ceilf((float)(i) / (float)blockDim.y);

						for (int kd = 0; kd < (int)iterk; kd++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

							int temp_idy = idy + (kd * blockDim.y);

							if (temp_idy < i) {

								int k = temp_idy;

								int rule1 = rulesArray[0][p];
								int rule2 = rulesArray[1][p];

								int offset = (int)(rule1 / 32);
								int first = cykArray[k][j + offset];

								int offset2 = (int)(rule2 / 32);
								int second = cykArray[i - k - 1][(((j / cellWidth) + k + 1) * cellWidth) + offset2];

								int base = 1;

								int bitMaskFirst = (base << (rule1 - (offset * 32)));
								int bitMaskSecond = (base << (rule2 - (offset2 * 32)));


								if (first & bitMaskFirst && second & bitMaskSecond) {

									int shiftValue = rulesArray[2][p];
									int offset = (int)(shiftValue / 32);
									int bitValue = base << (shiftValue - (offset * 32));

									atomicOr(&cykArray[i][j + offset], bitValue);
								}
							}
						}
					}
				}

			}
		}

		__syncthreads();

	}

	__syncthreads();
	//&& rulesCount == inputStringLength
	if (threadIdx.x == 0 && threadIdx.y == 0 && blockIdx.y == 0 && blockIdx.x == 0 ) {
		if (action != 2 || additionalVariable == inputStringLength -1) {
			//for (int i = 0; i < nonTermsCount; i++) {
			//	for (int j = 0; j < nonTermsCount; j++) {
			//		//cout << rulesNonTermsArray[i][j] << " | ";

			//		printf("%d | ", rulesNonTermsArray[i][j]);
			//	}
			//	//cout << endl;

			//	printf("\n");
			//}

			//for (int j = 1; j < inputStringLength; j++) {
			//	for (int i = 0; i < inputStringLength - j; i++) {
			//		for (int c = 0; c < cellWidth; c++) {
			//			//cout << cykArray[j][i + c] << " - ";

			//			printf("%d -  ", cykArray[j][i + c]);
			//		}
			//		printf(" | ");
			//	}
			//	printf("\n");
			//}

			//int* result = data.getResult();
			//printf("RESUUUULt: %d | ", result[0]);
			//result[0] = cykArray[j][i];

			int* result = data.getResult();
			//printf("RESUUUULt: %d | ", result[0]);
			result[0] = cykArray[inputStringLength - 1][0];

		}
		
	}

	__syncthreads();

	return;
}


