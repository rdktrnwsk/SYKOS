#include "hip/hip_runtime.h"
#include "Ogolne.cuh"

__global__ void randInit(hiprandState* state, int seed) {
    int threadIndex = threadIdx.x;
    hiprand_init(seed, threadIndex, 0, &state[threadIndex]);
} 
