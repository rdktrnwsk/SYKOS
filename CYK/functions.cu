#include "Ogolne.h"
#ifndef FUNCTIONS_CU
#define FUNCTIONS_CU

void initialPermutations(int** hostArray, int populationSize, int instanceSize) {

	const int maxInt = MAXINT;

	//initial permutations
	for(int i = 0; i < populationSize; i++) {	

		for(int j = 0; j < instanceSize; j++) {
			hostArray[i][j] = j;
		}		

		//hostArray[i][instanceSize] = maxInt; //poczatkowa inicjalizacja
		//hostArray[i][instanceSize + 1] = 0;

		// Fisher�Yates
		for(int j = instanceSize - 1; j > 0; j--) {
			int x = rand() % (j + 1);
			int tmp = hostArray[i][x];
			hostArray[i][x] = hostArray[i][j];
			hostArray[i][j] = tmp;
		}
	}

}

void createCuda2DArrayInt(int**& hostPtr, int**& devicePtr, int** data, int rows, int columns) {

	hostPtr = (int**)malloc((rows) * sizeof(int*));

	for(int i = 0; i < rows; i++) {
		hipMalloc((void**)&hostPtr[i], columns  * sizeof(int));
		if(data != NULL) {
			hipMemcpy(hostPtr[i], &data[i][0], columns * sizeof(int), hipMemcpyHostToDevice);
		} 
	}

	hipMalloc((void ***)&devicePtr, rows * sizeof(int*));
	hipMemcpy(devicePtr, hostPtr, rows * sizeof(int*), hipMemcpyHostToDevice);
}

void createCuda2DArrayFloat(float**& hostPtr, float**& devicePtr, float** data, int rows, int columns) {

	hostPtr = (float**)malloc((rows) * sizeof(float*));

	for(int i = 0; i < rows; i++) {
		hipMalloc((void**)&hostPtr[i], columns * sizeof(float));
		if(data != NULL) {
			hipMemcpy(hostPtr[i], &data[i][0], columns * sizeof(float), hipMemcpyHostToDevice);
		} 
	}

	hipMalloc((void ***)&devicePtr, rows * sizeof(float*));
	hipMemcpy(devicePtr, hostPtr, rows * sizeof(float*), hipMemcpyHostToDevice);
}

void destroyCuda2DArray(int**& hostPtr, int**& devicePtr, int rows) {

	for (int i = 0; i < rows; i++) {
		hipFree(hostPtr[i]);
	}
	hipFree(devicePtr);
	free(hostPtr);												
}


int testRouteArrayReversed(int** deviceArray, int rows, int columns, char* message) {
		
	bool* test = (bool*)malloc(rows * sizeof(bool));
	for(int i = 0 ; i < rows ; i++){
		test[i] = false;
	}

	int** hostArray = (int**)malloc(rows * sizeof(int*));
	for(int i = 0; i < rows; i++) {
		hostArray[i] = (int*)malloc(columns * sizeof(int));					
	}

	for(int i = 0; i < rows; i++) {
		hipMemcpy(hostArray[i], deviceArray[i], columns * sizeof(int), hipMemcpyDeviceToHost);
	}

	for(int i = 0 ; i < columns; i++) {
		/*printf( "\n\n");
		for(int k = 0; k < rows; k++) {
					printf( "%d -> ", hostArray[k][i]);
				}*/
		for(int j = 0 ; j < rows; j++) {
			/*if(test[hostArray[j][i]] == false) {
				test[hostArray[j][i]] = true;
			} else {
				fprintf(stderr, "\nRow: %d, Col: %d, Val: %d, Message: %s\n", j, i, hostArray[j][i], message);
				for(int k = 0; k < rows; k++) {
					fprintf(stderr, "%d -> ", hostArray[k][i]);
				}
				return 1;
			}*/
			printf("%d -> ", hostArray[j][i]);

		}
		return 1;
		printf("___________________________________\n\n");
		for(int i = 0 ; i < rows ; i++){
			test[i] = false;
		}
	}

				
	for(int i = 0; i < rows; i++) {
		free(hostArray[i]);				
	}
	free(hostArray);
	free(test);
	return 1;
}

int testRouteArray(int** deviceArray, int rows, int columns, char* message) {
		
	bool* test = (bool*)malloc(columns * sizeof(bool));
	for(int i = 0 ; i < columns ; i++){
		test[i] = false;
	}

	int** hostArray = (int**)malloc(rows * sizeof(int*));
	for(int i = 0; i < rows; i++) {
		hostArray[i] = (int*)malloc(columns * sizeof(int));					
	}

	for(int i = 0; i < rows; i++) {
		hipMemcpy(hostArray[i], deviceArray[i], columns * sizeof(int), hipMemcpyDeviceToHost);
	}


	for(int i = 0 ; i < rows; i++) {
		for(int j = 0 ; j < columns; j++) {
			if(test[hostArray[i][j]] == false) {
				test[hostArray[i][j]] = true;
			} else {
				fprintf(stderr, "\nRow: %d, Col: %d, Val: %d, Message: %s\n", i, j, hostArray[i][j], message);
				for(int k = 0; k < columns; k++) {
					fprintf(stderr, "%d -> ", hostArray[i][k]);
				}
				getchar();
				return 1;
			}

		}
		for(int i = 0 ; i < columns ; i++) {
			test[i] = false;
		}
	}

	free(test);

	for(int i = 0; i < rows; i++)  {
		free(hostArray[i]);				
	}
	free(hostArray);
}


#endif