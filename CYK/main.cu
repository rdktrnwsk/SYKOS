#include "hip/hip_runtime.h"
﻿#include "Utility.h"
#include "Ogolne.cuh"
#include "Ogolne.h"
#include "functions.cuh"
#include "Cultural.cuh"
#include "CYK.cuh"
#include <hip/hip_cooperative_groups.h>


int main(int argc, char** argv)
{
	printf("working\n\n\n");

	//printf("Grammar: %s\n", argv[1]);
	//printf("Input: %s\n", argv[2]);

	char* termsArray;
	int termsCount;
	char* nonTermsArray;
	int nonTermsCount;
	int* rulesTermsArray;
	int rulesTermsCount;
	int** rulesNonTermsArray;
	int rulesNonTermsCount;
	int** onlyRulesArray = NULL;
	int onlyRulesCount;

	char name[50] = "grammar.txt";

	int iterations = 3;
	//char* grammarFiles[5] = { "example1.bcfg", "g1_001.bcfg", "grammar5.txt", "grammar7.txt", "grammar7.txt" };
	char* grammarFiles[3] = { "grammar5.txt", "grammar7.txt", "grammar7.txt" };
	string inputStrings[3] = { "cababcabdcffabeedcababcabfffabeeecffabeeebdabfabebeb", "eeababkabdknteeababkabdtoc", "fdablsmteeababkabdtrfdablsmteeababkabdtrfdablsmabjrimabjrhmteeababkabdtreeababkabdhmabjreababhmabjreabab" };

	int cpuVersion = 0;


	for (int x = 0; x < iterations; x++) {

		string inputString = inputStrings[x];
		cout << "Grammar: " << grammarFiles[x] << endl << "Input string: " << inputString << endl;
		
		//readGrammar(argv[1], termsArray, termsCount, nonTermsArray, nonTermsCount, rulesTermsArray, rulesTermsCount, rulesNonTermsArray, rulesNonTermsCount, onlyRulesArray, onlyRulesCount);
		readGrammar(grammarFiles[x], termsArray, termsCount, nonTermsArray, nonTermsCount, rulesTermsArray, rulesTermsCount, rulesNonTermsArray, rulesNonTermsCount, onlyRulesArray, onlyRulesCount);
		
		// WSJ one
		/*readGrammarExtended(grammarFiles[0], termsArray, termsCount, nonTermsArray, nonTermsCount, rulesTermsArray, rulesTermsCount, rulesNonTermsArray, rulesNonTermsCount, onlyRulesArray, onlyRulesCount);
		for (int i = 0; i < nonTermsCount; i++) {
		for (int j = 0; j < nonTermsCount; j++) {
		cout << rulesNonTermsArray[i][j] << " | ";
		}
		cout << endl;
		}

		for (int i = 0; i < termsCount; i++) {
		cout << termsArray[i] << " | ";
		}
		cout << endl;

		for (int i = 0; i < termsCount; i++) {

		cout << rulesTermsArray[i] << " | ";
		}
		cout << endl;

		for (int i = 0; i < nonTermsCount; i++) {

		cout << nonTermsArray[i] << " | ";
		}
		cout << endl;


		getchar();*/
		

		/*ALGORITHM START*/

		// 1. First part

		// create CYK array of input string length

		int inputStringLength = inputString.length();

		int cellWidth = ceil(((float)nonTermsCount / 32.0f));
		//cellWidth = 1;

		int** cykArray = new int*[inputStringLength];
		for (int i = 0; i < inputStringLength; i++) {
			cykArray[i] = new int[inputStringLength * cellWidth]; // columns multiplied
		}
		// make array clear
		for (int i = 0; i < inputStringLength; i++) {
			for (int j = 0; j < inputStringLength * cellWidth; j++) {
				cykArray[i][j] = 0;
			}
		}

		// first phase, terminal rules array, for every input string character
		for (int i = 0; i < inputStringLength * cellWidth; i += cellWidth) {

			// find character (terminal index)
			int terminalIndex = -1;
			for (int j = 0; j < termsCount; j++) {
				if (inputString[i / cellWidth] == termsArray[j]) {
					terminalIndex = j;
					break;
				}
			}

			// TODO find out, if there is a possibility that one term is connected with many nonterms
			//for (int j = 0; j < rulesTermsCount; j++) {
			/*if (terminalIndex == rulesTermsArray[terminalIndex]) {
			cykArray[0][i] = j;
			}*/



			if (rulesTermsArray[terminalIndex] >= 0) {
				int shiftValue = rulesTermsArray[terminalIndex];

				int offset = (int)(shiftValue / 32); // shift by 32 is the next cell

				int base = 1;
				int bitValue = base << (shiftValue - (offset * 32));

				cykArray[0][i + offset] |= bitValue;
			}
		}

		// printing arrays and first row result - PRINTING PART

		// print rules 2D array
		/*for (int i = 0; i < nonTermsCount; i++) {
		for (int j = 0; j < nonTermsCount; j++) {
		cout << rulesNonTermsArray[i][j] << " | ";
		}
		cout << endl;
		}

		for (int i = 0; i < termsCount; i++) {
			cout << termsArray[i] << " | ";
		}
		cout << endl;

		for (int i = 0; i < termsCount; i++) {

			cout << rulesTermsArray[i] << " | ";
		}
		cout << endl;

		for (int i = 0; i < nonTermsCount; i++) {

			cout << nonTermsArray[i] << " | ";
		}
		cout << endl;

		for (int i = 0; i < inputStringLength * cellWidth; i++) {

			cout << cykArray[0][i] << " | ";
		}
		cout << endl;*/


		// 2. Second part

		// code input string to number
		int* inputNumber = new int[inputStringLength];
		for (int i = 0; i < inputStringLength; i++) {

			for (int j = 0; j < termsCount; j++) {
				if (inputString[i] == termsArray[j]) {
					inputNumber[i] = j;
					break;
				}
			}
		}

		// print coded input string
		//for (int i = 0; i < inputStringLength; i++) {

		//	cout << inputNumber[i] << " | ";
		//}
		//cout << endl;

		LARGE_INTEGER frequency;        // ticks per second
		LARGE_INTEGER t1, t2;           // ticks
		double elapsedTime;

		// get ticks per second
		QueryPerformanceFrequency(&frequency);
		// start timer
		QueryPerformanceCounter(&t1);

		if (cpuVersion == 0) {
			for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)

				for (int j = 0; j < (inputStringLength - i) * cellWidth; j += cellWidth) { // every word <of given length: 5 words, 4 words, 3 words, 2, 1...> (2)

					for (int k = 0; k < i; k++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

						//decode nonterminals (find out if bits are on a given positions)
						int base = 1;
						for (int m = 0; m < nonTermsCount; m++) {

							int offset = (int)(m / 32);
							int first = cykArray[k][j + offset];
							int bitMaskFirst = (base << (m - (offset * 32)));

							if (first & bitMaskFirst) {

								//all possibilities connected with rules
								for (int n = 0; n < nonTermsCount; n++) {

									int offset2 = (int)(n / 32); // shift by 32 is the next cell
									int second = cykArray[i - k - 1][(((j / cellWidth) + k + 1) * cellWidth) + offset2];
									int bitMaskSecond = (base << (n - (offset2 * 32)));

									if (second & bitMaskSecond) {

										//rule exists
										if (rulesNonTermsArray[m][n] != -1) {
											int shiftValue = rulesNonTermsArray[m][n];
											//int bitValue = base << shiftValue;

											int offset = (int)(shiftValue / 32); // shift by 32 is the next cell
											//cout << "ok - " << i << " _ " <<  j << " | " << m << " - " << n << " - " << offset  << " - " << shiftValue << " - bitFirst " << bitMaskFirst << " value1 " << first <<" - bitSeconf " << bitMaskSecond << " val2 " << second << " WTF _  " << ((j + k + 1) * cellWidth) + offset2 << "A standardowo: " << j + k + 1 << endl;

											int bitValue = base << (shiftValue - (offset * 32));

											cykArray[i][j + offset] |= bitValue;
										}

									}

								} // end n loop

							} //first condition

						} // end m loop

					} // end k loop

				} // end j loop

			} // end i loop
		} else if (cpuVersion == 1) {
			
			for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)

				for (int j = 0; j < (inputStringLength - i) * cellWidth; j += cellWidth) { // every word of given length 5, 4, 3, 2, 1... (2)

					for (int r = 0; r < onlyRulesCount; r++) { //for each production (each rule)

						for (int k = 0; k < i; k++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

							int offset = (int)(onlyRulesArray[0][r] / 32);
							int first = cykArray[k][j + offset];
							
							int offset2 = (int)(onlyRulesArray[1][r] / 32); // shift by 32 is the next cell
							int second = cykArray[i - k - 1][(((j / cellWidth) + k + 1) * cellWidth) + offset2];

							//decode nonterminals (find out if bits are on a given positions)
							int base = 1;
							//int bitMaskFirst = base << onlyRulesArray[0][r];
							int bitMaskFirst = (base << (onlyRulesArray[0][r] - (offset * 32)));
							//int bitMaskSecond = base << onlyRulesArray[1][r];
							int bitMaskSecond = (base << (onlyRulesArray[1][r] - (offset2 * 32)));
							if (first & bitMaskFirst && second & bitMaskSecond) {

								int shiftValue = onlyRulesArray[2][r];
								int offset = (int)(shiftValue / 32);
								int bitValue = base << (shiftValue - (offset * 32));
								//TODO - tutaj może być problem

								cykArray[i][j + offset] |= bitValue;
								
							}

						} // k loop

					} // r loop 

				} //end j loop

			} // end i loop

		}


		// stop timer
		QueryPerformanceCounter(&t2);

		elapsedTime = (t2.QuadPart - t1.QuadPart) * 1000.0 / frequency.QuadPart;
		std::cout << "CPU time: " << elapsedTime / 1000.0 << " ms\n";
		cout << "CPU result: " << cykArray[inputStringLength -1][0] << endl;

/*
		for (int j = 1; j < inputStringLength; j++) {
		for (int i = 0; i < inputStringLength - j; i++) {
		for (int c = 0; c < cellWidth; c++) {
		cout << cykArray[j][i + c] << " - ";
		}
		cout <<  " | ";
		}
		cout << endl;
		}*/

		/******************************************************************CUDA PART*********************************************************************/

		// variables
		int deviceNumber = 1;

		//select device number
		hipSetDevice(deviceNumber);

		// time measure
		hipEvent_t cudaStartTime, cudaStopTime;
		hipEventCreate(&cudaStartTime);
		hipEventCreate(&cudaStopTime);
		//default stream (time)
		hipStream_t defStream;
		hipStreamCreate(&defStream);



		// GPU Array Copy
		int** h_rulesNonTermsArray; //redundant
		int** d_rulesNonTermsArray;
		createCuda2DArrayInt(h_rulesNonTermsArray, d_rulesNonTermsArray, rulesNonTermsArray, nonTermsCount, nonTermsCount);

		// initial
		int threadsNumber = nonTermsCount; //TODO zmieniaj to odpowiednio - teraz jest to liczba nieterminali

										   // TODO remove that
		hiprandState * randState;
		hipMalloc(&randState, threadsNumber * sizeof(hiprandState)); //warning! look size
		randInit << <1, threadsNumber >> > (randState, time(NULL)); //ustawienie ziaren

		int instanceSize = 5;


		CulturalData culturalData(instanceSize + 2, threadsNumber);
		// make array clear
		for (int i = 2; i < inputStringLength; i++) {
			for (int j = 0; j < inputStringLength * cellWidth; j++) {
				cykArray[i][j] = 0;
			}
		}

		CYKData cykData(cykArray, inputStringLength, d_rulesNonTermsArray, nonTermsCount);


		int blockNumber = 16;
		int* h_array_in;
		int* array_in;
		int* h_array_out;
		int* array_out;
		int** h_onlyRulesArray;
		int** d_onlyRulesArray;
		int nonTermsWithRulesCount;
		int** devicePtr;
		int** hostPtr;

		int algorithmChoice = 36; // TODO REPLACE IT
// ----------------------
		if (algorithmChoice >= 20 && algorithmChoice <= 29) {
			h_array_in = (int*)malloc(sizeof(int) * blockNumber);
			for (int i = 0; i < blockNumber; i++) { h_array_in[i] = 0; }
			hipMalloc((void**)&array_in, sizeof(int) * blockNumber);
			hipMemcpy(array_in, h_array_in, sizeof(int) * blockNumber, hipMemcpyHostToDevice);

			h_array_out = (int*)malloc(sizeof(int) * blockNumber);
			for (int i = 0; i < blockNumber; i++) { h_array_out[i] = 0; }
			hipMalloc((void**)&array_out, sizeof(int) * blockNumber);
			hipMemcpy(array_out, h_array_out, sizeof(int) * blockNumber, hipMemcpyHostToDevice);
		}
		else if (algorithmChoice >= 30 && algorithmChoice <= 39) {

			createCuda2DArrayInt(h_onlyRulesArray, d_onlyRulesArray, onlyRulesArray, 3, onlyRulesCount); // standard array (3 rows)

			// fancy array (every row each possible symbol (existing), first cell symbol index, second cell number of rules [ech 2 cell pair])
			int* nonTermsWithRules = new int[nonTermsCount];
			for (int i = 0; i < nonTermsCount; i++) {
				nonTermsWithRules[i] = 0;
			}
			for (int i = 0; i < onlyRulesCount; i++) {
				//cout << onlyRulesCount << " : " << i << ": " << onlyRulesArray[2][i] << endl;
				nonTermsWithRules[onlyRulesArray[2][i]]++;
			}
			// to create array (row) of proper size
			nonTermsWithRulesCount = 0;
			for (int i = 0; i < nonTermsCount; i++) {
				if (nonTermsWithRules[i] > 0) {
					nonTermsWithRulesCount++;
				}
			}

			int** onlyRulesArraySplitted = new int*[nonTermsWithRulesCount];
			int* nonTermsToRules = new int[nonTermsCount]; // to project indexes
			int projectionNumber = 0;
			for (int i = 0; i < nonTermsCount; i++) {
				// create rows of given length (2 + 2 * productions)
				if (nonTermsWithRules[i] > 0) {

					onlyRulesArraySplitted[projectionNumber] = new int[(nonTermsWithRules[i] * 2) + 2]; // additional 2 positions, first for the left nonterminal, second for row length
																										// initial setup
					onlyRulesArraySplitted[projectionNumber][1] = 0;
					onlyRulesArraySplitted[projectionNumber][0] = -1; //initial value - empty row

					nonTermsToRules[i] = projectionNumber;
					projectionNumber++;
				}
				else {
					nonTermsToRules[i] = -1;
				}

			}


			for (int i = 0; i < onlyRulesCount; i++) {

				int leftSymbol = onlyRulesArray[2][i]; //left symbol numeric value
				int leftSymbolPr = nonTermsToRules[leftSymbol]; // left symbol projection to new array row
				int offset = onlyRulesArraySplitted[leftSymbolPr][1]++; //get current productions number and increment it!

				onlyRulesArraySplitted[leftSymbolPr][2 + (offset * 2)] = onlyRulesArray[0][i];
				onlyRulesArraySplitted[leftSymbolPr][3 + (offset * 2)] = onlyRulesArray[1][i];
				onlyRulesArraySplitted[leftSymbolPr][0] = leftSymbol;
			}

			//for (int i = 0; i < nonTermsWithRulesCount; i++) { //rows

			//	for (int j = 0; j < onlyRulesArraySplitted[i][1] * 2 + 2; j+=2) {
			//		cout << onlyRulesArraySplitted[i][j] << " - " << onlyRulesArraySplitted[i][j + 1] << " | ";
			//	}
			//	cout << endl;

			//}

			//create device array copy
			hostPtr = (int**)malloc((nonTermsWithRulesCount) * sizeof(int*));

			for (int i = 0; i < nonTermsWithRulesCount; i++) {
				int columns = (onlyRulesArraySplitted[i][1] * 2 + 2);
				hipMalloc((void**)&hostPtr[i], columns * sizeof(int));
				hipMemcpy(hostPtr[i], &onlyRulesArraySplitted[i][0], columns * sizeof(int), hipMemcpyHostToDevice);
			}

			hipMalloc((void ***)&devicePtr, nonTermsWithRulesCount * sizeof(int*));
			hipMemcpy(devicePtr, hostPtr, nonTermsWithRulesCount * sizeof(int*), hipMemcpyHostToDevice);

		}
		hipEventRecord(cudaStartTime, defStream); //start counting time
/////////////////////////////////////////////////////////////////////////////////////////////////
		algorithmChoice = 37;
		if (algorithmChoice == 10) {
			// TODO pamiętaj o wejściowej liczbie wątków
			// no restrictions
			cykAlgorithm<0> << <1, 1, 0, culturalData.getStream() >> >(cykData, randState);
		}
		else if (algorithmChoice == 11) {
			// noTermsCount < 1024
			cykAlgorithm<1> << <1, nonTermsCount, 0, culturalData.getStream() >> >(cykData, randState);
		}
		else if (algorithmChoice == 12) {
			// noTermsCount < 1024
			cykAlgorithm<2> << <1, nonTermsCount, 0, culturalData.getStream() >> >(cykData, randState);
		}
		else if (algorithmChoice == 13) {
			// noTermsCount < 32
			dim3 dimBlock(32, 32, 1);
			cykAlgorithm<3> << <1, dimBlock, 0, culturalData.getStream() >> >(cykData, randState);
		}
		else if (algorithmChoice == 14) {
			// dimension equals input string length
			// inputStringLength < 1024
			dim3 dimBlock(inputStringLength, 1, 1); //j - cells
			cykAlgorithm<4> << <1, dimBlock, 0, culturalData.getStream() >> >(cykData, randState);
		}
		else if (algorithmChoice == 15) {
			// block dimensions - max 1024 -> 32x32
			int tDimX = 32;
			int tDimY = 32;
			dim3 dimBlock(tDimX, tDimY, 1); //j - cells, k -> split points
			cykAlgorithm<5> << <1, dimBlock, 0, culturalData.getStream() >> >(cykData, randState);
		}
		else if (algorithmChoice == 101) {
			blockNumber = 16;
			dim3 dimBlock5(onlyRulesCount, 1, 1);

			//with local synchronisation
			for (int i = 1; i < inputStringLength; i++) {

				cykAlgorithmRules<2> << <inputStringLength - i, dimBlock5, 0 >> >(cykData, randState, array_in, array_out, d_onlyRulesArray, onlyRulesCount, i);
				//hipDeviceSynchronize();
				hipError_t cudaState;
				if (i < inputStringLength - 1) {
					cudaState = hipDeviceSynchronize();

					if (cudaState != hipSuccess) {
						fprintf(stderr, "\ncudaGetLastError: %s\n", hipGetErrorString(cudaState));
						hipGetLastError();
					}
				}
			}
		}
		else if (algorithmChoice == 20) {

			// TODO values -> now word cant be larger than 32 letters!!!! (bcs m is up to 32)
			dim3 dimBlock(32, 32, 1);
			int blockNumber = 8; // TODO and number of blocks with connection to number of threads

			cykAlgorithmCooperative<0> << <blockNumber, dimBlock, 0, culturalData.getStream() >> >(cykData, randState, array_in, array_out, 0);

		}
		else if (algorithmChoice == 21) {

			//TODO to samo co wyzej
			dim3 dimBlock(32, 32, 1);
			int blockNumber = 8;

			cykAlgorithmCooperative<1> << <blockNumber, dimBlock, 0, culturalData.getStream() >> >(cykData, randState, array_in, array_out, 0);

		}
		else if (algorithmChoice == 22) {

			//TODO to samo co wyzej
			dim3 dimBlock(16, 16, 1);
			int blockNumber = 8;

			cykAlgorithmCooperative<2> << <blockNumber, dimBlock, 0, culturalData.getStream() >> >(cykData, randState, array_in, array_out, 0);

		}
		else if (algorithmChoice == 23) {

			//TODO to samo co wyzej
			// x - m
			// y - k
			dim3 dimBlock(threadsNumber, 16, 1);
			int blockNumber = 8;

			cykAlgorithmCooperative<3> << <blockNumber, dimBlock, 0, culturalData.getStream() >> >(cykData, randState, array_in, array_out, 0);

		}
		else if (algorithmChoice == 24) {

			//TODO to samo co wyzej
			// x - m
			// y - k
			dim3 dimBlock(16, 16, 1);
			int blockNumber = 8;

			cykAlgorithmCooperative<4> << <blockNumber, dimBlock, 0, culturalData.getStream() >> >(cykData, randState, array_in, array_out, 0);

		} else if (algorithmChoice == 25) {

			

			dim3 dimBlock(32, 32, 1);

			//with local synchronisation
			for (int i = 1; i < inputStringLength; i++) {
				cykAlgorithmCooperative<5> <<<1, dimBlock, 0, culturalData.getStream() >>>(cykData, randState, array_in, array_out, i);

				//hipDeviceSynchronize();
				hipError_t cudaState;
				if (i < inputStringLength - 1) {
					cudaState = hipDeviceSynchronize();

					if (cudaState != hipSuccess) {
						fprintf(stderr, "\ncudaGetLastError: %s\n", hipGetErrorString(cudaState));
						hipGetLastError();
					}
				}
			}

		}
		else if (algorithmChoice == 26) {

			dim3 dimBlock(32, 1, 1);

			//with local synchronisation
			for (int i = 1; i < inputStringLength; i++) {
				cykAlgorithmCooperative<6> <<<inputStringLength - i, dimBlock, 0, culturalData.getStream() >> >(cykData, randState, array_in, array_out, i);

				//hipDeviceSynchronize();
				hipError_t cudaState;
				if (i < inputStringLength - 1) {
					cudaState = hipDeviceSynchronize();

					if (cudaState != hipSuccess) {
						fprintf(stderr, "\ncudaGetLastError: %s\n", hipGetErrorString(cudaState));
						hipGetLastError();
					}
				}
			}

		}
		else if (algorithmChoice == 27) {

			dim3 dimBlock(32, 32, 1);

			//with local synchronisation
			for (int i = 1; i < inputStringLength; i++) {
				cykAlgorithmCooperative<7> << <inputStringLength - i, dimBlock, 0, culturalData.getStream() >> >(cykData, randState, array_in, array_out, i);

				//hipDeviceSynchronize();
				hipError_t cudaState;
				if (i < inputStringLength - 1) {
					cudaState = hipDeviceSynchronize();

					if (cudaState != hipSuccess) {
						fprintf(stderr, "\ncudaGetLastError: %s\n", hipGetErrorString(cudaState));
						hipGetLastError();
					}
				}
			}

		}
		else if (algorithmChoice == 28) {

			dim3 dimBlock(128, 1, 1);

			//with local synchronisation

			for (int i = 1; i < inputStringLength; i++) {

				dim3 dimGrid(inputStringLength - i, 32, 1);

				cykAlgorithmCooperative<8> << <dimGrid, dimBlock, 0, culturalData.getStream() >> >(cykData, randState, array_in, array_out, i);

				//hipDeviceSynchronize();
				hipError_t cudaState;
				if (i < inputStringLength - 1) {
					cudaState = hipDeviceSynchronize();

					if (cudaState != hipSuccess) {
						fprintf(stderr, "\ncudaGetLastError: %s\n", hipGetErrorString(cudaState));
						hipGetLastError();
					}
				}
			}

		}
		else if (algorithmChoice == 29) {

			dim3 dimBlock(16, 2, 1);

			//with local synchronisation

			for (int i = 1; i < inputStringLength; i++) {

				dim3 dimGrid(inputStringLength - i, 32, 1);

				cykAlgorithmCooperative<9> << <dimGrid, dimBlock, 0, culturalData.getStream() >> >(cykData, randState, array_in, array_out, i);

				//hipDeviceSynchronize();
				hipError_t cudaState;
				if (i < inputStringLength - 1) {
					cudaState = hipDeviceSynchronize();

					if (cudaState != hipSuccess) {
						fprintf(stderr, "\ncudaGetLastError: %s\n", hipGetErrorString(cudaState));
						hipGetLastError();
					}
				}
			}

		}

		else if (algorithmChoice == 30) {

			blockNumber = 4;
			dim3 dimBlock5(onlyRulesCount, 1, 1); // every rule  = thread, blocks j loop
			cykAlgorithmRules<0> << <blockNumber, dimBlock5, 0, culturalData.getStream() >> >(cykData, randState, array_in, array_out, d_onlyRulesArray, onlyRulesCount, 0);

		}
		else if (algorithmChoice == 31) { //remember 32!!!!!!!

			dim3 dimBlock5(onlyRulesCount, 1, 1);

			//with local synchronisation
			for (int i = 1; i < inputStringLength; i++) {

				cykAlgorithmRules<2> << <inputStringLength - i, dimBlock5, 0 >> >(cykData, randState, array_in, array_out, d_onlyRulesArray, onlyRulesCount, i);
				hipError_t cudaState;
				if (i < inputStringLength - 1) {
					cudaState = hipDeviceSynchronize();

					if (cudaState != hipSuccess) {
						fprintf(stderr, "\ncudaGetLastError: %s\n", hipGetErrorString(cudaState));
						hipGetLastError();
					}
				}
			}
		}
		else if (algorithmChoice == 32) {

			dim3 dimBlock5(onlyRulesCount, 1, 1);

			//with local synchronisation
			for (int i = 1; i < inputStringLength; i++) {
				dim3 dimGrid(inputStringLength - i, 2, 1);
				cykAlgorithmRules<2> << <dimGrid, dimBlock5, 0 >> >(cykData, randState, array_in, array_out, d_onlyRulesArray, onlyRulesCount, i);
				hipError_t cudaState;
				if (i < inputStringLength - 1) {
					cudaState = hipDeviceSynchronize();

					if (cudaState != hipSuccess) {
						fprintf(stderr, "\ncudaGetLastError: %s\n", hipGetErrorString(cudaState));
						hipGetLastError();
					}
				}
			}
		}
		else if (algorithmChoice == 33) {
			//blockNumber = nonTermsWithRulesCount;
			dim3 dimBlock(64, 1, 1); //TODO change number of threads, number of rules -> x
			
														//with local synchronisation
			for (int i = 1; i < inputStringLength; i++) {
				dim3 dimGrid(inputStringLength - i, nonTermsWithRulesCount, 1); // y - left symbol, x - j loop
				cykAlgorithmRules<3> <<<dimGrid, dimBlock, 0, culturalData.getStream() >> >(cykData, randState, array_in, array_out, devicePtr, nonTermsWithRulesCount, i);

				//hipDeviceSynchronize();
				hipError_t cudaState;
				if (i < inputStringLength - 1) {
					cudaState = hipDeviceSynchronize();

					if (cudaState != hipSuccess) {
						fprintf(stderr, "\ncudaGetLastError: %s\n", hipGetErrorString(cudaState));
						hipGetLastError();
					}
				}
			}

			
		} else if (algorithmChoice == 34) {
			//blockNumber = nonTermsWithRulesCount;
			dim3 dimBlock(32, 4, 1); //TODO change number of threads, number of rules -> x

									 //with local synchronisation
			for (int i = 1; i < inputStringLength; i++) {
				dim3 dimGrid(inputStringLength - i, nonTermsWithRulesCount, 1); // y - left symbol, x - j loop
				cykAlgorithmRules<4> << <dimGrid, dimBlock, 0, culturalData.getStream() >> >(cykData, randState, array_in, array_out, devicePtr, nonTermsWithRulesCount, i);

				//hipDeviceSynchronize();
				hipError_t cudaState;
				if (i < inputStringLength - 1) {
					cudaState = hipDeviceSynchronize();

					if (cudaState != hipSuccess) {
						fprintf(stderr, "\ncudaGetLastError: %s\n", hipGetErrorString(cudaState));
						hipGetLastError();
					}
				}
			}


		} else if (algorithmChoice == 35) {
			//blockNumber = nonTermsWithRulesCount;
			dim3 dimBlock(32, 1, 1); //TODO change number of threads, number of rules -> x

									  //with local synchronisation
			for (int i = 1; i < inputStringLength; i++) {
				dim3 dimGrid(inputStringLength - i, nonTermsWithRulesCount, 8); // y - left symbol, x - j loop
				cykAlgorithmRules<5> << <dimGrid, dimBlock, 0, culturalData.getStream() >> >(cykData, randState, array_in, array_out, devicePtr, nonTermsWithRulesCount, i);

				//hipDeviceSynchronize();
				hipError_t cudaState;
				if (i < inputStringLength - 1) {
					cudaState = hipDeviceSynchronize();

					if (cudaState != hipSuccess) {
						fprintf(stderr, "\ncudaGetLastError: %s\n", hipGetErrorString(cudaState));
						hipGetLastError();
					}
				}
			}


		} else if (algorithmChoice == 36) {
			dim3 dimBlock5(32, 32, 1);

			//with local synchronisation
			for (int i = 1; i < inputStringLength; i++) {
				dim3 dimGrid(inputStringLength - i, 1, 1);
				cykAlgorithmRules<2> << <dimGrid, dimBlock5, 0 >> >(cykData, randState, array_in, array_out, d_onlyRulesArray, onlyRulesCount, i);
				hipError_t cudaState;
				if (i < inputStringLength - 1) {
					cudaState = hipDeviceSynchronize();

					if (cudaState != hipSuccess) {
						fprintf(stderr, "\ncudaGetLastError: %s\n", hipGetErrorString(cudaState));
						hipGetLastError();
					}
				}
			}
		}
		else if (algorithmChoice == 37) {
			dim3 dimBlock5(1, 32, 1);

			//with local synchronisation
			for (int i = 1; i < inputStringLength; i++) {
				dim3 dimGrid(inputStringLength - i, onlyRulesCount, 1);
				cykAlgorithmRules<2> << <dimGrid, dimBlock5, 0 >> >(cykData, randState, array_in, array_out, d_onlyRulesArray, onlyRulesCount, i);
				hipError_t cudaState;
				if (i < inputStringLength - 1) {
					cudaState = hipDeviceSynchronize();

					if (cudaState != hipSuccess) {
						fprintf(stderr, "\ncudaGetLastError: %s\n", hipGetErrorString(cudaState));
						hipGetLastError();
					}
				}
			}
		}
		else {

			/*void* params1[2];
			params1[0] = (void*)&cykData;
			params1[1] = (void*)&randState;*/

			//hipLaunchCooperativeKernel((void*)cykAlgorithmCooperative<0>, 1, dimBlock, params1, 0, culturalData.getStream());
			//hipStreamSynchronize(culturalData.getStream());

			//TODO Linux
			/*hipError_t cudaState2;
			cudaState2 = hipLaunchCooperativeKernel((void*)cykTest, 1, dimBlock, 0, 0, culturalData.getStream());

			cout << hipGetErrorString(cudaState2);*/


			// Reversed Loop CYK part


			//getchar();

			blockNumber = nonTermsWithRulesCount;
			dim3 dimBlock6(16, 1, 1); //TODO change number of threads
			dim3 dimBl(2, nonTermsWithRulesCount, 1);
			//cykAlgorithmRules<1> << <dimBl, dimBlock6, 0, culturalData.getStream() >> >(cykData, randState, array_in, array_out, devicePtr, nonTermsWithRulesCount);



		}


		


		hipError_t cudaState;
		cudaState = hipDeviceSynchronize();

		if (cudaState != hipSuccess) {
			fprintf(stderr, "\ncudaGetLastError: %s\n", hipGetErrorString(cudaState));
			hipGetLastError();
		}
		else {
			float hTimeValue = -1.0;
			hipEventRecord(cudaStopTime, defStream); //stop counting time
			hipEventSynchronize(cudaStopTime);
			hipEventElapsedTime(&hTimeValue, cudaStartTime, cudaStopTime);
			printf("CUDA time: %f\n", hTimeValue / 1000.0f);
			printf("Result: %d\n", cykData.getResultValue());

		}

		//cuda memory
		hipStreamDestroy(defStream);
		hipFree(randState);

		if (algorithmChoice >= 20 && algorithmChoice <= 29) {
			hipFree(h_array_in);
			hipFree(h_array_out);
			hipFree(array_out);
			hipFree(array_in);
		}
		if (algorithmChoice >= 30 && algorithmChoice <= 39) {
			for (int i = 0; i < nonTermsWithRulesCount; i++) {
				hipFree(hostPtr[i]);
			}
			free(hostPtr);
			hipFree(devicePtr);
		}

		for (int i = 0; i < nonTermsCount; i++) {
			hipFree(h_rulesNonTermsArray[i]);
		}
		free(h_rulesNonTermsArray);
		hipFree(d_rulesNonTermsArray);

		getchar();
	}

	

	getchar();
	return 0;

}