#include "Utility.h"
#include "Ogolne.cuh"
#include "Ogolne.h"
#include "functions.cuh"
#include "Cultural.cuh"
#include "CYK.cuh"


int main(int argc, char** argv)
{
	printf("working\n\n\n");

	printf("Grammar: %s\n", argv[1]);
	printf("Input: %s\n", argv[2]);

	char* termsArray;
	int termsCount;
	char* nonTermsArray;
	int nonTermsCount;
	int* rulesTermsArray;
	int rulesTermsCount;
	int** rulesNonTermsArray;
	int rulesNonTermsCount;

	char name[50] = "grammar.txt";

	readGrammar(argv[1], termsArray, termsCount, nonTermsArray, nonTermsCount, rulesTermsArray, rulesTermsCount, rulesNonTermsArray, rulesNonTermsCount);

	//for (int i = 0; i < terms; i++) {
	//	for (int j = 0; j < rules; j++) {

	//		//cout << rulesArray[i][j] << " | ";

	//	}
	//	cout << rulesTerms[i] << " | ";
	//	//cout << endl;
	//}

	for (int i = 0; i < nonTermsCount; i++) {
		for (int j = 0; j < nonTermsCount; j++) {
			cout << rulesNonTermsArray[i][j] << " | ";
		}
		cout << endl;
	}

	/*ALGORITHM START*/

	// 1. First part

	// create CYK array of input string length
	//string inputString = "abcabdcabe"; //example input string
	string inputString = argv[2];
	int inputStringLength = inputString.length();
	int** cykArray = new int*[inputStringLength];
	for (int i = 0; i < inputStringLength; i++) {
		cykArray[i] = new int[inputStringLength];
	}
	// make array clear
	for (int i = 0; i < inputStringLength; i++) {
		for (int j = 0; j < inputStringLength; j++) {
			cykArray[i][j] = 0;
		}
	}

	// first phase, terminal rules array, for every input string character
	for (int i = 0; i < inputStringLength; i++) {

		// find character (terminal index)
		int terminalIndex = -1;
		for (int j = 0; j < termsCount; j++) {
			if (inputString[i] == termsArray[j]) {
				terminalIndex = j;
				break;
			}
		}

		// TODO find out, if there is a possibility that one term is connected with many nonterms
		//for (int j = 0; j < rulesTermsCount; j++) {
			/*if (terminalIndex == rulesTermsArray[terminalIndex]) {
				cykArray[0][i] = j;
			}*/
		if (rulesTermsArray[terminalIndex] >= 0) {
			int shiftValue = rulesTermsArray[terminalIndex];

			int base = 1;
			int bitValue = base << shiftValue;

			cykArray[0][i] |= bitValue;
		}
		//}
	}

	// printing arrays and first row result
	/*for (int i = 0; i < termsCount; i++) {

		cout << termsArray[i] << " | ";
	}
	cout << endl;

	for (int i = 0; i < termsCount; i++) {

		cout << rulesTermsArray[i] << " | ";
	}
	cout << endl;

	for (int i = 0; i < nonTermsCount; i++) {

		cout << nonTermsArray[i] << " | ";
	}
	cout << endl;

	for (int i = 0; i < inputStringLength; i++) {

		cout << cykArray[0][i] << " | ";
	}*/

	// 2. Second part

	// code input string to number
	int* inputNumber = new int[inputStringLength];
	for (int i = 0; i < inputStringLength; i++) {

		for (int j = 0; j < termsCount; j++) {
			if (inputString[i] == termsArray[j]) {
				inputNumber[i] = j;
				break;
			}
		}
	}

	// print coded input string
	//for (int i = 0; i < inputStringLength; i++) {

	//	cout << inputNumber[i] << " | ";
	//}
	//cout << endl;


	for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)

		for (int j = 0; j < inputStringLength - i; j++) { // every word of given length 5, 4, 3, 2, 1... (2)

			for (int k = 0; k < i; k++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

				 //TODO correct split points!
				int first = cykArray[k][j];
				int second = cykArray[i - k - 1][j + k + 1];

				//decode nonterminals (find out if bits are on a given positions)
				int base = 1;
				for (int l = 0; l < nonTermsCount; l++) {
					int bitMaskFirst = base << l;
					//all possibilities connected with rules
					for (int m = 0; m < nonTermsCount; m++) {
						int bitMaskSecond = base << m;

						// if rule with 'l' index and 'm' index is created and ready to be found if corrrect X ->lm (does X exist in a grammar?)
						if (first & bitMaskFirst && second & bitMaskSecond) {
							//cout << bitMaskFirst << ", " << bitMaskSecond << " | ";

							//rule exists
							if (rulesNonTermsArray[l][m] != -1) {
								int shiftValue = rulesNonTermsArray[l][m];
								int bitValue = base << shiftValue;

								cykArray[i][j] |= bitValue;
							}

						}


					}

				}

				//cout << first << " | " << second << endl;

				//combinations of productions

				// for each production (rulesNonTerminals)

			}

		}

		//break; //only first line

	}

	for (int j = 1; j < inputStringLength; j++) {
		for (int i = 0; i < inputStringLength - j; i++) {

			cout << cykArray[j][i] << " | ";
		}
		cout << endl;
	}


	/******************************************************************CUDA PART*********************************************************************/

	// initial
	int threadsNumber = 8;

	//
	hiprandState * randState;
	hipMalloc(&randState, threadsNumber * sizeof(hiprandState)); //warning! look size
	randInit <<<1, threadsNumber >>> (randState, time(NULL)); //ustawienie ziaren

	int instanceSize = 5;


	CulturalData culturalData(instanceSize + 2, threadsNumber);
	float** d_instanceMatrix;
	//CYKData cykData(d_instanceMatrix);


	cykAlgorithm<1><<<1, threadsNumber, 0, culturalData.getStream()>>>(randState);
	
	//cuda memory
	hipFree(randState);

	getchar();
	return 0;

}