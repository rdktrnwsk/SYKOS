﻿#include "Utility.h"
#include "Ogolne.cuh"
#include "Ogolne.h"
#include "functions.cuh"
#include "Cultural.cuh"
#include "CYK.cuh"
#include <hip/hip_cooperative_groups.h>


int main(int argc, char** argv)
{
	printf("working\n\n\n");

	printf("Grammar: %s\n", argv[1]);
	printf("Input: %s\n", argv[2]);

	char* termsArray;
	int termsCount;
	char* nonTermsArray;
	int nonTermsCount;
	int* rulesTermsArray;
	int rulesTermsCount;
	int** rulesNonTermsArray;
	int rulesNonTermsCount;

	char name[50] = "grammar.txt";

	readGrammar(argv[1], termsArray, termsCount, nonTermsArray, nonTermsCount, rulesTermsArray, rulesTermsCount, rulesNonTermsArray, rulesNonTermsCount);

	//for (int i = 0; i < terms; i++) {
	//	for (int j = 0; j < rules; j++) {

	//		//cout << rulesArray[i][j] << " | ";

	//	}
	//	cout << rulesTerms[i] << " | ";
	//	//cout << endl;
	//}

	for (int i = 0; i < nonTermsCount; i++) {
		for (int j = 0; j < nonTermsCount; j++) {
			cout << rulesNonTermsArray[i][j] << " | ";
		}
		cout << endl;
	}

	/*ALGORITHM START*/

	// 1. First part

	// create CYK array of input string length
	//string inputString = "abcabdcabe"; //example input string
	string inputString = argv[2];
	int inputStringLength = inputString.length();
	int** cykArray = new int*[inputStringLength];
	for (int i = 0; i < inputStringLength; i++) {
		cykArray[i] = new int[inputStringLength];
	}
	// make array clear
	for (int i = 0; i < inputStringLength; i++) {
		for (int j = 0; j < inputStringLength; j++) {
			cykArray[i][j] = 0;
		}
	}

	// first phase, terminal rules array, for every input string character
	for (int i = 0; i < inputStringLength; i++) {

		// find character (terminal index)
		int terminalIndex = -1;
		for (int j = 0; j < termsCount; j++) {
			if (inputString[i] == termsArray[j]) {
				terminalIndex = j;
				break;
			}
		}

		// TODO find out, if there is a possibility that one term is connected with many nonterms
		//for (int j = 0; j < rulesTermsCount; j++) {
			/*if (terminalIndex == rulesTermsArray[terminalIndex]) {
				cykArray[0][i] = j;
			}*/
		if (rulesTermsArray[terminalIndex] >= 0) {
			int shiftValue = rulesTermsArray[terminalIndex];

			int base = 1;
			int bitValue = base << shiftValue;

			cykArray[0][i] |= bitValue;
		}
		//}
	}

	// printing arrays and first row result
	for (int i = 0; i < termsCount; i++) {

		cout << termsArray[i] << " | ";
	}
	cout << endl;

	for (int i = 0; i < termsCount; i++) {

		cout << rulesTermsArray[i] << " | ";
	}
	cout << endl;

	for (int i = 0; i < nonTermsCount; i++) {

		cout << nonTermsArray[i] << " | ";
	}
	cout << endl;

	for (int i = 0; i < inputStringLength; i++) {

		cout << cykArray[0][i] << " | ";
	}

	// 2. Second part

	// code input string to number
	int* inputNumber = new int[inputStringLength];
	for (int i = 0; i < inputStringLength; i++) {

		for (int j = 0; j < termsCount; j++) {
			if (inputString[i] == termsArray[j]) {
				inputNumber[i] = j;
				break;
			}
		}
	}

	// print coded input string
	//for (int i = 0; i < inputStringLength; i++) {

	//	cout << inputNumber[i] << " | ";
	//}
	//cout << endl;


	for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)

		for (int j = 0; j < inputStringLength - i; j++) { // every word <of given length: 5 words, 4 words, 3 words, 2, 1...> (2)

			for (int k = 0; k < i; k++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

				 //TODO correct split points!
				int first = cykArray[k][j];
				int second = cykArray[i - k - 1][j + k + 1];

				//decode nonterminals (find out if bits are on a given positions)
				int base = 1;
				for (int l = 0; l < nonTermsCount; l++) {
					int bitMaskFirst = base << l;
					//all possibilities connected with rules
					for (int m = 0; m < nonTermsCount; m++) {
						int bitMaskSecond = base << m;

						// if rule with 'l' index and 'm' index is created and ready to be found if corrrect X ->lm (does X exist in a grammar?)
						if (first & bitMaskFirst && second & bitMaskSecond) {
							//cout << bitMaskFirst << ", " << bitMaskSecond << " | ";

							//rule exists
							if (rulesNonTermsArray[l][m] != -1) {
								int shiftValue = rulesNonTermsArray[l][m];
								int bitValue = base << shiftValue;

								cykArray[i][j] |= bitValue;
							}

						}


					}

				}

				//cout << first << " | " << second << endl;

				//combinations of productions

				// for each production (rulesNonTerminals)

			}

		}

		//break; //only first line

	}

	for (int j = 1; j < inputStringLength; j++) {
		for (int i = 0; i < inputStringLength - j; i++) {

			cout << cykArray[j][i] << " | ";
		}
		cout << endl;
	}


	/******************************************************************CUDA PART*********************************************************************/
	
	hipSetDevice(1);

	/*char* termsArray;
	int termsCount;
	char* nonTermsArray;
	int nonTermsCount;
	int* rulesTermsArray;
	int rulesTermsCount;
	int** rulesNonTermsArray;
	int rulesNonTermsCount;*/

	// GPU Array Copy
	int** h_rulesNonTermsArray; //redundant
	int** d_rulesNonTermsArray;
	createCuda2DArrayInt(h_rulesNonTermsArray, d_rulesNonTermsArray, rulesNonTermsArray, nonTermsCount, nonTermsCount);

	// initial
	int threadsNumber = nonTermsCount; //TODO zmieniaj to odpowiednio - teraz jest to liczba nieterminali

	//
	hiprandState * randState;
	hipMalloc(&randState, threadsNumber * sizeof(hiprandState)); //warning! look size
	randInit <<<1, threadsNumber >>> (randState, time(NULL)); //ustawienie ziaren

	int instanceSize = 5;


	CulturalData culturalData(instanceSize + 2, threadsNumber);
	// make array clear
	for (int i = 2; i < inputStringLength; i++) {
		for (int j = 0; j < inputStringLength; j++) {
			cykArray[i][j] = 0;
		}
	}
	
	CYKData cykData(cykArray, inputStringLength, d_rulesNonTermsArray, nonTermsCount);


	// time measure
	hipEvent_t cudaStartTime, cudaStopTime;
	hipEventCreate(&cudaStartTime);
	hipEventCreate(&cudaStopTime);

	//default stream (time)
	hipStream_t defStream;
	hipStreamCreate(&defStream);

	hipEventRecord(cudaStartTime, defStream); //start counting time
																			// TODO pamiętaj o wejściowej liczbie wątków
	//cykAlgorithm<1><<<1, threadsNumber, 0, culturalData.getStream()>>>(cykData, randState);

	dim3 dimBlock(threadsNumber, threadsNumber, 1);
	//cykAlgorithm<3> <<<1, dimBlock, 0, culturalData.getStream() >>>(cykData, randState);

	/*void* params1[2];
	params1[0] = (void*)&cykData;
	params1[1] = (void*)&randState;*/

	//hipLaunchCooperativeKernel((void*)cykAlgorithmCooperative<0>, 1, dimBlock, params1, 0, culturalData.getStream());
	//hipStreamSynchronize(culturalData.getStream());

	//TODO Linux
	/*hipError_t cudaState2;
	cudaState2 = hipLaunchCooperativeKernel((void*)cykTest, 1, dimBlock, 0, 0, culturalData.getStream());
	
	cout << hipGetErrorString(cudaState2);*/

	/*int* array_in = (int*)malloc(sizeof(int) * 10);
	*h_result = 420;
	hipMalloc((void**)&result, sizeof(int));
	hipMemcpy(result, h_result, sizeof(int), hipMemcpyHostToDevice);*/

	cykAlgorithmCooperative<0> <<<inputStringLength, dimBlock, 0, culturalData.getStream() >>>(cykData, randState);

	hipError_t cudaState;
	cudaState = hipDeviceSynchronize();

	if (cudaState != hipSuccess) {
		fprintf(stderr, "\ncudaGetLastError: %s\n", hipGetErrorString(cudaState));
		hipGetLastError();
	} else {
		float hTimeValue = -1.0;
		hipEventRecord(cudaStopTime, defStream); //stop counting time
		hipEventSynchronize(cudaStopTime);
		hipEventElapsedTime(&hTimeValue, cudaStartTime, cudaStopTime);
		printf("CUDA time: %f\n", hTimeValue / 1000.0f);
		printf("Result: %d\n", cykData.getResultValue());
		
	}

	//cuda memory
	hipStreamDestroy(defStream);
	hipFree(randState);

	getchar();
	return 0;

}