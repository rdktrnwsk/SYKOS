#include "hip/hip_runtime.h"
﻿#include "Utility.h"
#include "Ogolne.cuh"
#include "Ogolne.h"
#include "functions.cuh"
#include "Cultural.cuh"
#include "CYK.cuh"
#include <hip/hip_cooperative_groups.h>


int main(int argc, char** argv)
{
	printf("working\n\n\n");

	printf("Grammar: %s\n", argv[1]);
	printf("Input: %s\n", argv[2]);

	char* termsArray;
	int termsCount;
	char* nonTermsArray;
	int nonTermsCount;
	int* rulesTermsArray;
	int rulesTermsCount;
	int** rulesNonTermsArray;
	int rulesNonTermsCount;

	char name[50] = "grammar.txt";

	readGrammar(argv[1], termsArray, termsCount, nonTermsArray, nonTermsCount, rulesTermsArray, rulesTermsCount, rulesNonTermsArray, rulesNonTermsCount);

	//for (int i = 0; i < terms; i++) {
	//	for (int j = 0; j < rules; j++) {

	//		//cout << rulesArray[i][j] << " | ";

	//	}
	//	cout << rulesTerms[i] << " | ";
	//	//cout << endl;
	//}

	for (int i = 0; i < nonTermsCount; i++) {
		for (int j = 0; j < nonTermsCount; j++) {
			cout << rulesNonTermsArray[i][j] << " | ";
		}
		cout << endl;
	}

	/*ALGORITHM START*/

	// 1. First part

	// create CYK array of input string length
	//string inputString = "abcabdcabe"; //example input string
	string inputString = argv[2];
	int inputStringLength = inputString.length();
	int** cykArray = new int*[inputStringLength];
	for (int i = 0; i < inputStringLength; i++) {
		cykArray[i] = new int[inputStringLength];
	}
	// make array clear
	for (int i = 0; i < inputStringLength; i++) {
		for (int j = 0; j < inputStringLength; j++) {
			cykArray[i][j] = 0;
		}
	}

	// first phase, terminal rules array, for every input string character
	for (int i = 0; i < inputStringLength; i++) {

		// find character (terminal index)
		int terminalIndex = -1;
		for (int j = 0; j < termsCount; j++) {
			if (inputString[i] == termsArray[j]) {
				terminalIndex = j;
				break;
			}
		}

		// TODO find out, if there is a possibility that one term is connected with many nonterms
		//for (int j = 0; j < rulesTermsCount; j++) {
			/*if (terminalIndex == rulesTermsArray[terminalIndex]) {
				cykArray[0][i] = j;
			}*/
		if (rulesTermsArray[terminalIndex] >= 0) {
			int shiftValue = rulesTermsArray[terminalIndex];

			int base = 1;
			int bitValue = base << shiftValue;

			cykArray[0][i] |= bitValue;
		}
		//}
	}

	// printing arrays and first row result
	/*for (int i = 0; i < termsCount; i++) {

		cout << termsArray[i] << " | ";
	}
	cout << endl;

	for (int i = 0; i < termsCount; i++) {

		cout << rulesTermsArray[i] << " | ";
	}
	cout << endl;

	for (int i = 0; i < nonTermsCount; i++) {

		cout << nonTermsArray[i] << " | ";
	}
	cout << endl;

	for (int i = 0; i < inputStringLength; i++) {

		cout << cykArray[0][i] << " | ";
	}*/

	// 2. Second part

	// code input string to number
	int* inputNumber = new int[inputStringLength];
	for (int i = 0; i < inputStringLength; i++) {

		for (int j = 0; j < termsCount; j++) {
			if (inputString[i] == termsArray[j]) {
				inputNumber[i] = j;
				break;
			}
		}
	}

	// print coded input string
	//for (int i = 0; i < inputStringLength; i++) {

	//	cout << inputNumber[i] << " | ";
	//}
	//cout << endl;

	std::clock_t c_start = std::clock();
	// your_algorithm
	


	for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)

		for (int j = 0; j < inputStringLength - i; j++) { // every word <of given length: 5 words, 4 words, 3 words, 2, 1...> (2)

			for (int k = 0; k < i; k++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

				 //TODO correct split points!
				int first = cykArray[k][j];
				int second = cykArray[i - k - 1][j + k + 1];

				//decode nonterminals (find out if bits are on a given positions)
				int base = 1;
				for (int l = 0; l < nonTermsCount; l++) {
					int bitMaskFirst = base << l;
					//all possibilities connected with rules
					for (int m = 0; m < nonTermsCount; m++) {
						int bitMaskSecond = base << m;

						// if rule with 'l' index and 'm' index is created and ready to be found if corrrect X ->lm (does X exist in a grammar?)
						if (first & bitMaskFirst && second & bitMaskSecond) {
							//cout << bitMaskFirst << ", " << bitMaskSecond << " | ";

							//rule exists
							if (rulesNonTermsArray[l][m] != -1) {
								int shiftValue = rulesNonTermsArray[l][m];
								int bitValue = base << shiftValue;

								cykArray[i][j] |= bitValue;
							}

						}


					}

				}

				//cout << first << " | " << second << endl;

				//combinations of productions

				// for each production (rulesNonTerminals)

			}

		}

		//break; //only first line

	}

	std::clock_t c_end = std::clock();

	double time_elapsed_ms = 1000.0 * (c_end - c_start) / CLOCKS_PER_SEC;
	std::cout << "CPU time used: " << time_elapsed_ms / 1000.0 << " ms\n";

	/*for (int j = 1; j < inputStringLength; j++) {
		for (int i = 0; i < inputStringLength - j; i++) {

			cout << cykArray[j][i] << " | ";
		}
		cout << endl;
	}*/


	/******************************************************************CUDA PART*********************************************************************/
	
	hipSetDevice(1);

	/*char* termsArray;
	int termsCount;
	char* nonTermsArray;
	int nonTermsCount;
	int* rulesTermsArray;
	int rulesTermsCount;
	int** rulesNonTermsArray;
	int rulesNonTermsCount;*/

	// GPU Array Copy
	int** h_rulesNonTermsArray; //redundant
	int** d_rulesNonTermsArray;
	createCuda2DArrayInt(h_rulesNonTermsArray, d_rulesNonTermsArray, rulesNonTermsArray, nonTermsCount, nonTermsCount);

	// initial
	int threadsNumber = nonTermsCount; //TODO zmieniaj to odpowiednio - teraz jest to liczba nieterminali

	//
	hiprandState * randState;
	hipMalloc(&randState, threadsNumber * sizeof(hiprandState)); //warning! look size
	randInit <<<1, threadsNumber >>> (randState, time(NULL)); //ustawienie ziaren

	int instanceSize = 5;


	CulturalData culturalData(instanceSize + 2, threadsNumber);
	// make array clear
	for (int i = 2; i < inputStringLength; i++) {
		for (int j = 0; j < inputStringLength; j++) {
			cykArray[i][j] = 0;
		}
	}
	
	CYKData cykData(cykArray, inputStringLength, d_rulesNonTermsArray, nonTermsCount);


	// time measure
	hipEvent_t cudaStartTime, cudaStopTime;
	hipEventCreate(&cudaStartTime);
	hipEventCreate(&cudaStopTime);

	//default stream (time)
	hipStream_t defStream;
	hipStreamCreate(&defStream);

	hipEventRecord(cudaStartTime, defStream); //start counting time
																			// TODO pamiętaj o wejściowej liczbie wątków
	cykAlgorithm<1><<<1, threadsNumber, 0, culturalData.getStream()>>>(cykData, randState);

	dim3 dimBlock(threadsNumber, threadsNumber, 1);
	//cykAlgorithm<3> <<<1, dimBlock, 0, culturalData.getStream() >>>(cykData, randState);
	
	dim3 dimBlock2(32, 32, 1);
	//cykAlgorithm<5> <<<1, dimBlock2, 0, culturalData.getStream() >>>(cykData, randState);

	/*void* params1[2];
	params1[0] = (void*)&cykData;
	params1[1] = (void*)&randState;*/

	//hipLaunchCooperativeKernel((void*)cykAlgorithmCooperative<0>, 1, dimBlock, params1, 0, culturalData.getStream());
	//hipStreamSynchronize(culturalData.getStream());

	//TODO Linux
	/*hipError_t cudaState2;
	cudaState2 = hipLaunchCooperativeKernel((void*)cykTest, 1, dimBlock, 0, 0, culturalData.getStream());
	
	cout << hipGetErrorString(cudaState2);*/

	

	int blockNumber = 16;
	int* h_array_in = (int*)malloc(sizeof(int) * blockNumber);
	for (int i = 0; i < blockNumber; i++) { h_array_in[i] = 0; }
	int* array_in;
	hipMalloc((void**)&array_in, sizeof(int) * blockNumber);
	hipMemcpy(array_in, h_array_in, sizeof(int) * blockNumber, hipMemcpyHostToDevice);

	int* h_array_out = (int*)malloc(sizeof(int) * blockNumber);
	for (int i = 0; i < blockNumber; i++) { h_array_out[i] = 0; }
	int* array_out;
	hipMalloc((void**)&array_out, sizeof(int) * blockNumber);
	hipMemcpy(array_out, h_array_out, sizeof(int) * blockNumber, hipMemcpyHostToDevice);

	//cykAlgorithmCooperative<0> <<<32, dimBlock, 0, culturalData.getStream() >>>(cykData, randState, array_in, array_out);

	//cykAlgorithmCooperative<1> <<<blockNumber, dimBlock, 0, culturalData.getStream() >>>(cykData, randState, array_in, array_out);

	hipError_t cudaState;
	cudaState = hipDeviceSynchronize();

	if (cudaState != hipSuccess) {
		fprintf(stderr, "\ncudaGetLastError: %s\n", hipGetErrorString(cudaState));
		hipGetLastError();
	} else {
		float hTimeValue = -1.0;
		hipEventRecord(cudaStopTime, defStream); //stop counting time
		hipEventSynchronize(cudaStopTime);
		hipEventElapsedTime(&hTimeValue, cudaStartTime, cudaStopTime);
		printf("CUDA time: %f\n", hTimeValue / 1000.0f);
		printf("Result: %d\n", cykData.getResultValue());
		
	}

	//cuda memory
	hipStreamDestroy(defStream);
	hipFree(randState);
	hipFree(array_out);
	hipFree(array_in);
	free(h_array_in);
	free(h_array_out);

	getchar();
	return 0;

}