#include "hip/hip_runtime.h"
﻿#include "Utility.h"
#include "Ogolne.cuh"
#include "Ogolne.h"
#include "functions.cuh"
#include "Cultural.cuh"
#include "CYK.cuh"
#include <hip/hip_cooperative_groups.h>


int main(int argc, char** argv)
{
	printf("working\n\n\n");

	printf("Grammar: %s\n", argv[1]);
	printf("Input: %s\n", argv[2]);

	char* termsArray;
	int termsCount;
	char* nonTermsArray;
	int nonTermsCount;
	int* rulesTermsArray;
	int rulesTermsCount;
	int** rulesNonTermsArray;
	int rulesNonTermsCount;
	int** onlyRulesArray = NULL;
	int onlyRulesCount;

	char name[50] = "grammar.txt";

	readGrammar(argv[1], termsArray, termsCount, nonTermsArray, nonTermsCount, rulesTermsArray, rulesTermsCount, rulesNonTermsArray, rulesNonTermsCount, onlyRulesArray, onlyRulesCount);

	//for (int i = 0; i < 3; i++) {
	//	for (int j = 0; j < nonTermsCount; j++) {

	//		cout << onlyRulesArray[i][j] << " | ";

	//	}
	//	//cout << rulesTerms[i] << " | ";
	//	cout << endl;
	//}

	//getchar();

	for (int i = 0; i < nonTermsCount; i++) {
		for (int j = 0; j < nonTermsCount; j++) {
			cout << rulesNonTermsArray[i][j] << " | ";
		}
		cout << endl;
	}

	/*ALGORITHM START*/

	// 1. First part

	// create CYK array of input string length
	//string inputString = "abcabdcabe"; //example input string
	string inputString = argv[2];
	int inputStringLength = inputString.length();
	int** cykArray = new int*[inputStringLength];
	for (int i = 0; i < inputStringLength; i++) {
		cykArray[i] = new int[inputStringLength];
	}
	// make array clear
	for (int i = 0; i < inputStringLength; i++) {
		for (int j = 0; j < inputStringLength; j++) {
			cykArray[i][j] = 0;
		}
	}

	// first phase, terminal rules array, for every input string character
	for (int i = 0; i < inputStringLength; i++) {

		// find character (terminal index)
		int terminalIndex = -1;
		for (int j = 0; j < termsCount; j++) {
			if (inputString[i] == termsArray[j]) {
				terminalIndex = j;
				break;
			}
		}

		// TODO find out, if there is a possibility that one term is connected with many nonterms
		//for (int j = 0; j < rulesTermsCount; j++) {
			/*if (terminalIndex == rulesTermsArray[terminalIndex]) {
				cykArray[0][i] = j;
			}*/
		if (rulesTermsArray[terminalIndex] >= 0) {
			int shiftValue = rulesTermsArray[terminalIndex];

			int base = 1;
			int bitValue = base << shiftValue;

			cykArray[0][i] |= bitValue;
		}
		//}
	}

	// printing arrays and first row result
	/*for (int i = 0; i < termsCount; i++) {

		cout << termsArray[i] << " | ";
	}
	cout << endl;

	for (int i = 0; i < termsCount; i++) {

		cout << rulesTermsArray[i] << " | ";
	}
	cout << endl;

	for (int i = 0; i < nonTermsCount; i++) {

		cout << nonTermsArray[i] << " | ";
	}
	cout << endl;

	for (int i = 0; i < inputStringLength; i++) {

		cout << cykArray[0][i] << " | ";
	}*/

	// 2. Second part

	// code input string to number
	int* inputNumber = new int[inputStringLength];
	for (int i = 0; i < inputStringLength; i++) {

		for (int j = 0; j < termsCount; j++) {
			if (inputString[i] == termsArray[j]) {
				inputNumber[i] = j;
				break;
			}
		}
	}

	// print coded input string
	//for (int i = 0; i < inputStringLength; i++) {

	//	cout << inputNumber[i] << " | ";
	//}
	//cout << endl;

	std::clock_t c_start = std::clock();
	// your_algorithm
	


	for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)

		for (int j = 0; j < inputStringLength - i; j++) { // every word <of given length: 5 words, 4 words, 3 words, 2, 1...> (2)

			for (int k = 0; k < i; k++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

				 //TODO correct split points!
				int first = cykArray[k][j];
				int second = cykArray[i - k - 1][j + k + 1];

				//decode nonterminals (find out if bits are on a given positions)
				int base = 1;
				for (int l = 0; l < nonTermsCount; l++) {
					int bitMaskFirst = base << l;
					//all possibilities connected with rules
					for (int m = 0; m < nonTermsCount; m++) {
						int bitMaskSecond = base << m;

						// if rule with 'l' index and 'm' index is created and ready to be found if corrrect X ->lm (does X exist in a grammar?)
						if (first & bitMaskFirst && second & bitMaskSecond) {
							//cout << bitMaskFirst << ", " << bitMaskSecond << " | ";

							//rule exists
							if (rulesNonTermsArray[l][m] != -1) {
								int shiftValue = rulesNonTermsArray[l][m];
								int bitValue = base << shiftValue;

								cykArray[i][j] |= bitValue;
							}

						}


					}

				}

				//cout << first << " | " << second << endl;

				//combinations of productions

				// for each production (rulesNonTerminals)

			}

		}

		//break; //only first line

	}

	std::clock_t c_end = std::clock();

	double time_elapsed_ms = 1000.0 * (c_end - c_start) / CLOCKS_PER_SEC;
	std::cout << "CPU time used: " << time_elapsed_ms / 1000.0 << " ms\n";

	/*for (int j = 1; j < inputStringLength; j++) {
		for (int i = 0; i < inputStringLength - j; i++) {

			cout << cykArray[j][i] << " | ";
		}
		cout << endl;
	}*/


	/******************************************************************CUDA PART*********************************************************************/
	
	hipSetDevice(1);

	/*char* termsArray;
	int termsCount;
	char* nonTermsArray;
	int nonTermsCount;
	int* rulesTermsArray;
	int rulesTermsCount;
	int** rulesNonTermsArray;
	int rulesNonTermsCount;*/

	// GPU Array Copy
	int** h_rulesNonTermsArray; //redundant
	int** d_rulesNonTermsArray;
	createCuda2DArrayInt(h_rulesNonTermsArray, d_rulesNonTermsArray, rulesNonTermsArray, nonTermsCount, nonTermsCount);

	// initial
	int threadsNumber = nonTermsCount; //TODO zmieniaj to odpowiednio - teraz jest to liczba nieterminali

	//
	hiprandState * randState;
	hipMalloc(&randState, threadsNumber * sizeof(hiprandState)); //warning! look size
	randInit <<<1, threadsNumber >>> (randState, time(NULL)); //ustawienie ziaren

	int instanceSize = 5;


	CulturalData culturalData(instanceSize + 2, threadsNumber);
	// make array clear
	for (int i = 2; i < inputStringLength; i++) {
		for (int j = 0; j < inputStringLength; j++) {
			cykArray[i][j] = 0;
		}
	}
	
	CYKData cykData(cykArray, inputStringLength, d_rulesNonTermsArray, nonTermsCount);


	// time measure
	hipEvent_t cudaStartTime, cudaStopTime;
	hipEventCreate(&cudaStartTime);
	hipEventCreate(&cudaStopTime);

	//default stream (time)
	hipStream_t defStream;
	hipStreamCreate(&defStream);

	hipEventRecord(cudaStartTime, defStream); //start counting time
																			// TODO pamiętaj o wejściowej liczbie wątków
	//cykAlgorithm<1><<<1, threadsNumber, 0, culturalData.getStream()>>>(cykData, randState);

	dim3 dimBlock(threadsNumber, threadsNumber, 1);
	//cykAlgorithm<3> <<<1, dimBlock, 0, culturalData.getStream() >>>(cykData, randState);
	
	dim3 dimBlock2(32, 32, 1);
	//cykAlgorithm<5> <<<1, dimBlock2, 0, culturalData.getStream() >>>(cykData, randState);

	/*void* params1[2];
	params1[0] = (void*)&cykData;
	params1[1] = (void*)&randState;*/

	//hipLaunchCooperativeKernel((void*)cykAlgorithmCooperative<0>, 1, dimBlock, params1, 0, culturalData.getStream());
	//hipStreamSynchronize(culturalData.getStream());

	//TODO Linux
	/*hipError_t cudaState2;
	cudaState2 = hipLaunchCooperativeKernel((void*)cykTest, 1, dimBlock, 0, 0, culturalData.getStream());
	
	cout << hipGetErrorString(cudaState2);*/

	

	int blockNumber = 16;
	int* h_array_in = (int*)malloc(sizeof(int) * blockNumber);
	for (int i = 0; i < blockNumber; i++) { h_array_in[i] = 0; }
	int* array_in;
	hipMalloc((void**)&array_in, sizeof(int) * blockNumber);
	hipMemcpy(array_in, h_array_in, sizeof(int) * blockNumber, hipMemcpyHostToDevice);

	int* h_array_out = (int*)malloc(sizeof(int) * blockNumber);
	for (int i = 0; i < blockNumber; i++) { h_array_out[i] = 0; }
	int* array_out;
	hipMalloc((void**)&array_out, sizeof(int) * blockNumber);
	hipMemcpy(array_out, h_array_out, sizeof(int) * blockNumber, hipMemcpyHostToDevice);

	dim3 dimBlock3(threadsNumber, threadsNumber, 1);

	//cykAlgorithmCooperative<0> <<<32, dimBlock3, 0, culturalData.getStream() >>>(cykData, randState, array_in, array_out);

	//cykAlgorithmCooperative<1> <<<blockNumber, dimBlock3, 0, culturalData.getStream() >>>(cykData, randState, array_in, array_out);

	//dim3 dimBlock4(threadsNumber, 16, 1);
	//cykAlgorithmCooperative<3> <<<blockNumber, dimBlock4, 0, culturalData.getStream() >>>(cykData, randState, array_in, array_out);

	

	// Reversed Loop CYK part

	// create new rules array
	
	//createCuda2DArrayInt(this->h_cykArray, this->cykArray, cykArray, inputStringLength, inputStringLength);

	

	//only rules with threads
	int** h_onlyRulesArray;
	int** d_onlyRulesArray;
	createCuda2DArrayInt(h_onlyRulesArray, d_onlyRulesArray, onlyRulesArray, 3, onlyRulesCount);
	blockNumber = 1;
	//dim3 dimBlock5(onlyRulesCount, 1, 1);
	//cykAlgorithmRules<0><<<blockNumber, dimBlock5, 0, culturalData.getStream() >>>(cykData, randState, array_in, array_out, d_onlyRulesArray, onlyRulesCount);


	//only rules blocks + threads
	int* nonTermsWithRules = new int[nonTermsCount];
	for (int i = 0; i < nonTermsCount; i++) {
		nonTermsWithRules[i] = 0;
	}
	for (int i = 0; i < onlyRulesCount; i++) {
		nonTermsWithRules[onlyRulesArray[2][i]]++;
	}
	// to create array (row) of proper size
	int nonTermsWithRulesCount = 0;
	for (int i = 0; i < nonTermsCount; i++) {
		if (nonTermsWithRules[i] > 0) {
			nonTermsWithRulesCount++;
		}
	}
	
	int** onlyRulesArraySplitted = new int*[nonTermsWithRulesCount];
	int* nonTermsToRules = new int[nonTermsCount]; // to project indexes
	int projectionNumber = 0;
	for (int i = 0; i < nonTermsCount; i++) {
		// create rows of given length (2 + 2 * productions)
		if (nonTermsWithRules[i] > 0) {
			
			onlyRulesArraySplitted[projectionNumber] = new int[(nonTermsWithRules[i] * 2) + 2]; // additional 2 positions, first for the left nonterminal, second for row length
																				 // initial setup
			onlyRulesArraySplitted[projectionNumber][1] = 0;
			onlyRulesArraySplitted[projectionNumber][0] = -1; //initial value - empty row

			nonTermsToRules[i] = projectionNumber;
			projectionNumber++;
		} else {
			nonTermsToRules[i] = -1;
		}
		
	}


	for (int i = 0; i < onlyRulesCount; i++) {

		int leftSymbol = onlyRulesArray[2][i]; //left symbol numeric value
		int leftSymbolPr = nonTermsToRules[leftSymbol]; // left symbol projection to new array row
		int offset = onlyRulesArraySplitted[leftSymbolPr][1]++; //get current productions number and increment it!

		onlyRulesArraySplitted[leftSymbolPr][2 + (offset * 2)] = onlyRulesArray[0][i];
		onlyRulesArraySplitted[leftSymbolPr][3 + (offset * 2)] = onlyRulesArray[1][i];
		onlyRulesArraySplitted[leftSymbolPr][0] = leftSymbol;
	}

	for (int i = 0; i < nonTermsWithRulesCount; i++) { //rows

		for (int j = 0; j < onlyRulesArraySplitted[i][1] * 2 + 2; j+=2) {
			cout << onlyRulesArraySplitted[i][j] << " - " << onlyRulesArraySplitted[i][j + 1] << " | ";
		}
		cout << endl;

	}
	

	getchar();

	blockNumber = 1;
	dim3 dimBlock5(onlyRulesCount, 1, 1);
	cykAlgorithmRules<1> << <blockNumber, dimBlock5, 0, culturalData.getStream() >> >(cykData, randState, array_in, array_out, d_onlyRulesArray, onlyRulesCount);


	hipError_t cudaState;
	cudaState = hipDeviceSynchronize();

	if (cudaState != hipSuccess) {
		fprintf(stderr, "\ncudaGetLastError: %s\n", hipGetErrorString(cudaState));
		hipGetLastError();
	} else {
		float hTimeValue = -1.0;
		hipEventRecord(cudaStopTime, defStream); //stop counting time
		hipEventSynchronize(cudaStopTime);
		hipEventElapsedTime(&hTimeValue, cudaStartTime, cudaStopTime);
		printf("CUDA time: %f\n", hTimeValue / 1000.0f);
		printf("Result: %d\n", cykData.getResultValue());
		
	}

	//cuda memory
	hipStreamDestroy(defStream);
	hipFree(randState);
	hipFree(array_out);
	hipFree(array_in);
	free(h_array_in);
	free(h_array_out);
	/*for (int i = 0; i < blockNumber; i++) {
		free(onlyRulesArray[i]);
		free(h_onlyRulesArray[i]);
		hipFree(d_onlyRulesArray[i]);
	}
	hipFree(d_onlyRulesArray);
	free(onlyRulesArray);
	free(h_onlyRulesArray);*/

	getchar();
	return 0;

}