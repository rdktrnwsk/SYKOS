#include "hip/hip_runtime.h"
﻿#include "Utility.h"
#include "Ogolne.cuh"
#include "Ogolne.h"
#include "functions.cuh"
#include "Cultural.cuh"
#include "CYK.cuh"
#include <hip/hip_cooperative_groups.h>


int main(int argc, char** argv)
{
	printf("working\n\n\n");

	printf("Grammar: %s\n", argv[1]);
	printf("Input: %s\n", argv[2]);

	char* termsArray;
	int termsCount;
	char* nonTermsArray;
	int nonTermsCount;
	int* rulesTermsArray;
	int rulesTermsCount;
	int** rulesNonTermsArray;
	int rulesNonTermsCount;
	int** onlyRulesArray = NULL;
	int onlyRulesCount;

	char name[50] = "grammar.txt";

	readGrammar(argv[1], termsArray, termsCount, nonTermsArray, nonTermsCount, rulesTermsArray, rulesTermsCount, rulesNonTermsArray, rulesNonTermsCount, onlyRulesArray, onlyRulesCount);

	//for (int i = 0; i < 3; i++) {
	//	for (int j = 0; j < nonTermsCount; j++) {

	//		cout << onlyRulesArray[i][j] << " | ";

	//	}
	//	//cout << rulesTerms[i] << " | ";
	//	cout << endl;
	//}

	//getchar();

	for (int i = 0; i < nonTermsCount; i++) {
		for (int j = 0; j < nonTermsCount; j++) {
			cout << rulesNonTermsArray[i][j] << " | ";
		}
		cout << endl;
	}

	/*ALGORITHM START*/

	// 1. First part

	// create CYK array of input string length
	//string inputString = "abcabdcabe"; //example input string
	string inputString = argv[2];
	int inputStringLength = inputString.length();

	int cellWidth =  ceil(((float)nonTermsCount / 32.0f));
	//cellWidth = 1;
	cout << "hahahah" << cellWidth << endl;

	int** cykArray = new int*[inputStringLength];
	for (int i = 0; i < inputStringLength; i++) {
		cykArray[i] = new int[inputStringLength * cellWidth]; // columns multiplied
	}
	// make array clear
	for (int i = 0; i < inputStringLength; i++) {
		for (int j = 0; j < inputStringLength * cellWidth; j++) {
			cykArray[i][j] = 0;
		}
	}

	// first phase, terminal rules array, for every input string character
	for (int i = 0; i < inputStringLength * cellWidth; i+= cellWidth) {

		// find character (terminal index)
		int terminalIndex = -1;
		for (int j = 0; j < termsCount; j++) {
			if (inputString[i/cellWidth] == termsArray[j]) {
				terminalIndex = j;
				break;
			}
		}

		// TODO find out, if there is a possibility that one term is connected with many nonterms
		//for (int j = 0; j < rulesTermsCount; j++) {
			/*if (terminalIndex == rulesTermsArray[terminalIndex]) {
				cykArray[0][i] = j;
			}*/

		

		if (rulesTermsArray[terminalIndex] >= 0) {
			int shiftValue = rulesTermsArray[terminalIndex];

			int offset = (int)(shiftValue / 32); // shift by 32 is the next cell

			int base = 1;
			int bitValue = base << (shiftValue - (offset * 32));

			//cout << "X" << bitValue << endl;

			cykArray[0][i + offset] |= bitValue;
		}
		//}
	}

	// printing arrays and first row result
	for (int i = 0; i < termsCount; i++) {

		cout << termsArray[i] << " | ";
	}
	cout << endl;

	for (int i = 0; i < termsCount; i++) {

		cout << rulesTermsArray[i] << " | ";
	}
	cout << endl;

	for (int i = 0; i < nonTermsCount; i++) {

		cout << nonTermsArray[i] << " | ";
	}
	cout << endl;

	for (int i = 0; i < inputStringLength * cellWidth; i++) {

		cout << cykArray[0][i] << " | ";
	}
	cout << endl;
	// 2. Second part

	// code input string to number
	int* inputNumber = new int[inputStringLength];
	for (int i = 0; i < inputStringLength; i++) {

		for (int j = 0; j < termsCount; j++) {
			if (inputString[i] == termsArray[j]) {
				inputNumber[i] = j;
				break;
			}
		}
	}

	// print coded input string
	//for (int i = 0; i < inputStringLength; i++) {

	//	cout << inputNumber[i] << " | ";
	//}
	//cout << endl;

	std::clock_t c_start = std::clock();
	// your_algorithm
	
	for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)

		for (int j = 0; j < (inputStringLength - i) * cellWidth; j += cellWidth) { // every word <of given length: 5 words, 4 words, 3 words, 2, 1...> (2)

			for (int k = 0; k < i; k++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

				//for (int c = 0; c < cellWidth; c++) {

					//decode nonterminals (find out if bits are on a given positions)
					int base = 1;
					for (int m = 0; m < nonTermsCount; m++) {

						//int shiftValue = rulesTermsArray[terminalIndex];
						//int offset = (int)(shiftValue / 32); // shift by 32 is the next cell
						//int base = 1;
						//int bitValue = base << (shiftValue - (offset * 32));
						////cout << "X" << bitValue << endl;
						//cykArray[0][i + offset] |= bitValue;

						int offset = (int)(m / 32); // shift by 32 is the next cell
						int first = cykArray[k][j + offset];

						int bitMaskFirst = (base << (m - (offset * 32)));

						//all possibilities connected with rules
						for (int n = 0; n < nonTermsCount; n++) {

							int offset2 = (int)(n / 32); // shift by 32 is the next cell
							int second = cykArray[i - k - 1][(((j / cellWidth) + k + 1) * cellWidth) + offset2];

							int bitMaskSecond = (base << (n - (offset2 * 32)));

							// if rule with 'm' index and 'n' index is created and ready to be found if corrrect X ->lm (does X exist in a grammar?)
							if (first & bitMaskFirst && second & bitMaskSecond) {
								//cout << bitMaskFirst << ", " << bitMaskSecond << " | ";
								
								//rule exists
								if (rulesNonTermsArray[m][n] != -1) {
									int shiftValue = rulesNonTermsArray[m][n];
									//int bitValue = base << shiftValue;
									
									int offset = (int)(shiftValue / 32); // shift by 32 is the next cell
									//cout << "ok - " << i << " _ " <<  j << " | " << m << " - " << n << " - " << offset  << " - " << shiftValue << " - bitFirst " << bitMaskFirst << " value1 " << first <<" - bitSeconf " << bitMaskSecond << " val2 " << second << " WTF _  " << ((j + k + 1) * cellWidth) + offset2 << "A standardowo: " << j + k + 1 << endl;
									int base = 1;
									int bitValue = base << (shiftValue - (offset * 32));

									cykArray[i][j + offset] |= bitValue;
								}

							}

						}

					}

				//} // end c loop

			} // end k loop

		} // end j loop



	}

	//for (int i = 1; i < inputStringLength; i++) { // for every row (starting from second one) (word length of 2, 3, 4 etc.) (1)

	//	for (int j = 0; j < inputStringLength - i; j++) { // every word of given length 5, 4, 3, 2, 1... (2)

	//		for (int k = 0; k < i; k++) { // for each neighbour (split points number of a word) 2| 1_2 - 2_1| 3_1 - 2_2 - 1_3| 4_1 - 3_2 - 2_3 - 1_4 (3)

	//									  //TODO correct split points!
	//			int first = cykArray[k][j];
	//			int second = cykArray[i - k - 1][j + k + 1];

	//			//decode nonterminals (find out if bits are on a given positions)
	//			int base = 1;
	//			for (int l = 0; l < nonTermsCount; l++) {

	//				int bitMaskFirst = base << l;

	//				// all possibilities connected with rules
	//				for (int m = 0; m < nonTermsCount; m++) {
	//					int bitMaskSecond = base << m;

	//					// if rule with 'l' index and 'm' index is created and ready to be found if corrrect X ->lm (does X exist in a grammar?)
	//					if (first & bitMaskFirst && second & bitMaskSecond) {
	//						
	//						//rule exists
	//						if (rulesNonTermsArray[l][m] != -1) {
	//							int shiftValue = rulesNonTermsArray[l][m];
	//							int bitValue = base << shiftValue;
	//							cout << "ok - " << i << " _ " << j << " | " << l << " - " << n << " - " << "Brak" << " - " << shiftValue << endl;
	//							cykArray[i][j] |= bitValue;
	//						}

	//					}

	//				}

	//			} // l loop end

	//		}

	//	}

	//	//break; //only first line

	//}



	std::clock_t c_end = std::clock();

	double time_elapsed_ms = 1000.0 * (c_end - c_start) / CLOCKS_PER_SEC;
	std::cout << "CPU time used: " << time_elapsed_ms / 1000.0 << " ms\n";

	

	for (int j = 1; j < inputStringLength; j++) {
		for (int i = 0; i < inputStringLength - j; i++) {
			for (int c = 0; c < cellWidth; c++) {
				cout << cykArray[j][i + c] << " - ";
			}
			cout <<  " | ";
		}
		cout << endl;
	}
	getchar();

	/******************************************************************CUDA PART*********************************************************************/
	
	// variables
	int deviceNumber = 1;


	

	//select device number
	hipSetDevice(deviceNumber);

	// time measure
	hipEvent_t cudaStartTime, cudaStopTime;
	hipEventCreate(&cudaStartTime);
	hipEventCreate(&cudaStopTime);
	//default stream (time)
	hipStream_t defStream;
	hipStreamCreate(&defStream);

	/*char* termsArray;
	int termsCount;
	char* nonTermsArray;
	int nonTermsCount;
	int* rulesTermsArray;
	int rulesTermsCount;
	int** rulesNonTermsArray;
	int rulesNonTermsCount;*/

	// GPU Array Copy
	int** h_rulesNonTermsArray; //redundant
	int** d_rulesNonTermsArray;
	createCuda2DArrayInt(h_rulesNonTermsArray, d_rulesNonTermsArray, rulesNonTermsArray, nonTermsCount, nonTermsCount);

	// initial
	int threadsNumber = nonTermsCount; //TODO zmieniaj to odpowiednio - teraz jest to liczba nieterminali

	// TODO remove that
	hiprandState * randState;
	hipMalloc(&randState, threadsNumber * sizeof(hiprandState)); //warning! look size
	randInit <<<1, threadsNumber >>> (randState, time(NULL)); //ustawienie ziaren

	int instanceSize = 5;


	CulturalData culturalData(instanceSize + 2, threadsNumber);
	// make array clear
	for (int i = 2; i < inputStringLength; i++) {
		for (int j = 0; j < inputStringLength; j++) {
			cykArray[i][j] = 0;
		}
	}
	
	CYKData cykData(cykArray, inputStringLength, d_rulesNonTermsArray, nonTermsCount);


	
	hipEventRecord(cudaStartTime, defStream); //start counting time
	
	int algorithmChoice = 11;

	int blockNumber = 16;
	int* h_array_in;
	int* array_in;
	int* h_array_out;
	int* array_out;
	int** h_onlyRulesArray;
	int** d_onlyRulesArray;
	int nonTermsWithRulesCount;
	int** devicePtr;

	if (algorithmChoice >= 20 && algorithmChoice <= 29) {
		h_array_in = (int*)malloc(sizeof(int) * blockNumber);
		for (int i = 0; i < blockNumber; i++) { h_array_in[i] = 0; }
		hipMalloc((void**)&array_in, sizeof(int) * blockNumber);
		hipMemcpy(array_in, h_array_in, sizeof(int) * blockNumber, hipMemcpyHostToDevice);

		h_array_out = (int*)malloc(sizeof(int) * blockNumber);
		for (int i = 0; i < blockNumber; i++) { h_array_out[i] = 0; }
		hipMalloc((void**)&array_out, sizeof(int) * blockNumber);
		hipMemcpy(array_out, h_array_out, sizeof(int) * blockNumber, hipMemcpyHostToDevice);
	}
	else if (algorithmChoice >= 29 && algorithmChoice <= 39) {

		createCuda2DArrayInt(h_onlyRulesArray, d_onlyRulesArray, onlyRulesArray, 3, onlyRulesCount); // standard array (3 rows)

		// fancy array (every row each possible symbol (existing), first cell symbol index, second cell number of rules [ech 2 cell pair])
		int* nonTermsWithRules = new int[nonTermsCount];
		for (int i = 0; i < nonTermsCount; i++) {
			nonTermsWithRules[i] = 0;
		}
		for (int i = 0; i < onlyRulesCount; i++) {
			//cout << onlyRulesCount << " : " << i << ": " << onlyRulesArray[2][i] << endl;
			nonTermsWithRules[onlyRulesArray[2][i]]++;
		}
		// to create array (row) of proper size
		nonTermsWithRulesCount = 0;
		for (int i = 0; i < nonTermsCount; i++) {
			if (nonTermsWithRules[i] > 0) {
				nonTermsWithRulesCount++;
			}
		}

		int** onlyRulesArraySplitted = new int*[nonTermsWithRulesCount];
		int* nonTermsToRules = new int[nonTermsCount]; // to project indexes
		int projectionNumber = 0;
		for (int i = 0; i < nonTermsCount; i++) {
			// create rows of given length (2 + 2 * productions)
			if (nonTermsWithRules[i] > 0) {

				onlyRulesArraySplitted[projectionNumber] = new int[(nonTermsWithRules[i] * 2) + 2]; // additional 2 positions, first for the left nonterminal, second for row length
																									// initial setup
				onlyRulesArraySplitted[projectionNumber][1] = 0;
				onlyRulesArraySplitted[projectionNumber][0] = -1; //initial value - empty row

				nonTermsToRules[i] = projectionNumber;
				projectionNumber++;
			}
			else {
				nonTermsToRules[i] = -1;
			}

		}


		for (int i = 0; i < onlyRulesCount; i++) {

			int leftSymbol = onlyRulesArray[2][i]; //left symbol numeric value
			int leftSymbolPr = nonTermsToRules[leftSymbol]; // left symbol projection to new array row
			int offset = onlyRulesArraySplitted[leftSymbolPr][1]++; //get current productions number and increment it!

			onlyRulesArraySplitted[leftSymbolPr][2 + (offset * 2)] = onlyRulesArray[0][i];
			onlyRulesArraySplitted[leftSymbolPr][3 + (offset * 2)] = onlyRulesArray[1][i];
			onlyRulesArraySplitted[leftSymbolPr][0] = leftSymbol;
		}

		//for (int i = 0; i < nonTermsWithRulesCount; i++) { //rows

		//	for (int j = 0; j < onlyRulesArraySplitted[i][1] * 2 + 2; j+=2) {
		//		cout << onlyRulesArraySplitted[i][j] << " - " << onlyRulesArraySplitted[i][j + 1] << " | ";
		//	}
		//	cout << endl;

		//}

		//create device array copy
		int** hostPtr = (int**)malloc((nonTermsWithRulesCount) * sizeof(int*));

		for (int i = 0; i < nonTermsWithRulesCount; i++) {
			int columns = (onlyRulesArraySplitted[i][1] * 2 + 2);
			hipMalloc((void**)&hostPtr[i], columns * sizeof(int));
			hipMemcpy(hostPtr[i], &onlyRulesArraySplitted[i][0], columns * sizeof(int), hipMemcpyHostToDevice);
		}

		hipMalloc((void ***)&devicePtr, nonTermsWithRulesCount * sizeof(int*));
		hipMemcpy(devicePtr, hostPtr, nonTermsWithRulesCount * sizeof(int*), hipMemcpyHostToDevice);

	}

	if (algorithmChoice == 10) {
		// TODO pamiętaj o wejściowej liczbie wątków
		// no restrictions
		cykAlgorithm<0> << <1, 1, 0, culturalData.getStream() >> >(cykData, randState);
	}
	else if (algorithmChoice == 11) {
		// noTermsCount < 1024
		cykAlgorithm<1> << <1, nonTermsCount, 0, culturalData.getStream() >> >(cykData, randState);
	}
	else if (algorithmChoice == 12) {
		// noTermsCount < 1024
		cykAlgorithm<2> << <1, nonTermsCount, 0, culturalData.getStream() >> >(cykData, randState);
	}
	else if (algorithmChoice == 13) {
		// noTermsCount < 32
		dim3 dimBlock(nonTermsCount, nonTermsCount, 1);
		cykAlgorithm<3> <<<1, dimBlock, 0, culturalData.getStream() >>>(cykData, randState);
	}
	else if (algorithmChoice == 14) {
		// dimension equals input string length
		// inputStringLength < 1024
		dim3 dimBlock(inputStringLength, 1, 1); //j - cells
		cykAlgorithm<4> << <1, dimBlock, 0, culturalData.getStream() >> >(cykData, randState);
	}
	else if (algorithmChoice == 15) {
		// block dimensions - max 1024 -> 32x32
		int tDimX = 32;
		int tDimY = 32;
		dim3 dimBlock(tDimX, tDimY, 1); //j - cells, k -> split points
		cykAlgorithm<5> <<<1, dimBlock, 0, culturalData.getStream() >>>(cykData, randState);
	}
	else if (algorithmChoice == 20) {

		// TODO values -> now word cant be larger than 32 letters!!!! (bcs m is up to 32)
		dim3 dimBlock(32, 32, 1);
		int blockNumber = 8; // TODO and number of blocks with connection to number of threads

		cykAlgorithmCooperative<0> <<<blockNumber, dimBlock, 0, culturalData.getStream() >>>(cykData, randState, array_in, array_out);

	}
	else if (algorithmChoice == 21) {

		//TODO to samo co wyzej
		dim3 dimBlock(32, 32, 1);
		int blockNumber = 8;

		cykAlgorithmCooperative<1> <<<blockNumber, dimBlock, 0, culturalData.getStream() >>>(cykData, randState, array_in, array_out);

	}
	else if (algorithmChoice == 22) {

		//TODO to samo co wyzej
		dim3 dimBlock(32, 32, 1);
		int blockNumber = 8;

		cykAlgorithmCooperative<2> <<<blockNumber, dimBlock, 0, culturalData.getStream() >>>(cykData, randState, array_in, array_out);

	}
	else if (algorithmChoice == 23) {

		//TODO to samo co wyzej
		// x - m
		// y - k
		dim3 dimBlock(threadsNumber, 16, 1);
		int blockNumber = 16;

		cykAlgorithmCooperative<3> <<<blockNumber, dimBlock, 0, culturalData.getStream() >>>(cykData, randState, array_in, array_out);

	}
	else if (algorithmChoice == 30) {

		blockNumber = 16;
		dim3 dimBlock5(onlyRulesCount, 1, 1); // every rule  = thread, blocks j loop
		cykAlgorithmRules<0><<<blockNumber, dimBlock5, 0, culturalData.getStream() >>>(cykData, randState, array_in, array_out, d_onlyRulesArray, onlyRulesCount, 0);

	}
	else if (algorithmChoice == 31) {

		blockNumber = 16;
		dim3 dimBlock5(onlyRulesCount, 1, 1);

		//with local synchronisation
		for (int i = 1; i < inputStringLength; i++) {

			cykAlgorithmRules<2> << <inputStringLength - i, dimBlock5, 0 >> >(cykData, randState, array_in, array_out, d_onlyRulesArray, onlyRulesCount, i);
			//hipDeviceSynchronize();
			hipError_t cudaState;
			if (i < inputStringLength - 1) {
				cudaState = hipDeviceSynchronize();

				if (cudaState != hipSuccess) {
					fprintf(stderr, "\ncudaGetLastError: %s\n", hipGetErrorString(cudaState));
					hipGetLastError();
				}
			}
		}
	}
	else if (algorithmChoice == 32) {
		//blockNumber = nonTermsWithRulesCount;
		dim3 dimBlock(32, 1, 1); //TODO change number of threads
		dim3 dimBl(2, nonTermsWithRulesCount, 1); // y - left symbol, x - j loop
		cykAlgorithmRules<1> <<<dimBl, dimBlock, 0, culturalData.getStream() >>>(cykData, randState, array_in, array_out, devicePtr, nonTermsWithRulesCount, 0);

	}
	else {
		
		/*void* params1[2];
		params1[0] = (void*)&cykData;
		params1[1] = (void*)&randState;*/

		//hipLaunchCooperativeKernel((void*)cykAlgorithmCooperative<0>, 1, dimBlock, params1, 0, culturalData.getStream());
		//hipStreamSynchronize(culturalData.getStream());

		//TODO Linux
		/*hipError_t cudaState2;
		cudaState2 = hipLaunchCooperativeKernel((void*)cykTest, 1, dimBlock, 0, 0, culturalData.getStream());

		cout << hipGetErrorString(cudaState2);*/

		
		// Reversed Loop CYK part


		//getchar();

		blockNumber = nonTermsWithRulesCount;
		dim3 dimBlock6(16, 1, 1); //TODO change number of threads
		dim3 dimBl(2, nonTermsWithRulesCount, 1);
		//cykAlgorithmRules<1> << <dimBl, dimBlock6, 0, culturalData.getStream() >> >(cykData, randState, array_in, array_out, devicePtr, nonTermsWithRulesCount);


		
	}
																			
	

	

	hipError_t cudaState;
	cudaState = hipDeviceSynchronize();

	if (cudaState != hipSuccess) {
		fprintf(stderr, "\ncudaGetLastError: %s\n", hipGetErrorString(cudaState));
		hipGetLastError();
	} else {
		float hTimeValue = -1.0;
		hipEventRecord(cudaStopTime, defStream); //stop counting time
		hipEventSynchronize(cudaStopTime);
		hipEventElapsedTime(&hTimeValue, cudaStartTime, cudaStopTime);
		printf("CUDA time: %f\n", hTimeValue / 1000.0f);
		printf("Result: %d\n", cykData.getResultValue());
		
	}

	//cuda memory
	hipStreamDestroy(defStream);
	hipFree(randState);

	if (algorithmChoice >= 20 && algorithmChoice <= 29) {
		hipFree(array_out);
		hipFree(array_in);
		free(h_array_in);
		free(h_array_out);
	}
	
	/*for (int i = 0; i < blockNumber; i++) {
		free(onlyRulesArray[i]);
		free(h_onlyRulesArray[i]);
		hipFree(d_onlyRulesArray[i]);
	}
	hipFree(d_onlyRulesArray);
	free(onlyRulesArray);
	free(h_onlyRulesArray);*/

	getchar();
	return 0;

}